#include "hip/hip_runtime.h"
 typedef float G4double;
typedef float G4float;
typedef int G4int;
typedef int G4bool;
typedef long G4long;
const G4double kInfinity = 1.0E37;
const int BlockSize = 32;
const int Multiplier = 4;
const G4double twopi = 2.0*3.14159265358979323846264338327;
const G4double kMinExitingNormalCosine = 1E-3;
typedef enum {kOutside,kSurface,kInside} EInside;
typedef enum {kNormal,kReplica,kParameterised} EVolume;
typedef enum {kXAxis,kYAxis,kZAxis,kRho,kRadial3D,kPhi,kUndefined} EAxis;
typedef enum { kBox = 0 , kOrb, kTubs, kCons, kPolyCone, Solidcount } ESolid;
typedef struct
{
 G4double x,y,z;

 G4double w;

}
G4ThreeVector;
__device__
G4ThreeVector G4ThreeVector_create( G4double x, G4double y, G4double z )
{
 G4ThreeVector v =
   {x,y,z,0};
 return v;
}
__device__
G4ThreeVector G4ThreeVector_saxpy( G4double a, G4ThreeVector x, G4ThreeVector y )
{
 return G4ThreeVector_create(
  a*x.x + y.x,
  a*x.y + y.y,
  a*x.z + y.z );
}
__device__
G4ThreeVector G4ThreeVector_sum( G4ThreeVector a, G4ThreeVector b )
{
 return G4ThreeVector_create( a.x+b.x, a.y+b.y, a.z+b.z );
}
__device__
G4ThreeVector G4ThreeVector_subtract( G4ThreeVector a, G4ThreeVector b )
{
 return G4ThreeVector_create( a.x-b.x, a.y-b.y, a.z-b.z );
}
__device__
G4ThreeVector G4ThreeVector_sum_assign( G4ThreeVector *This, G4ThreeVector b )
{
 (*This).x += b.x;
 (*This).y += b.y;
 (*This).z += b.z;
 return *This;
}
__device__
G4ThreeVector G4ThreeVector_subtract_assign( G4ThreeVector *This, G4ThreeVector b )
{
 (*This).x -= b.x;
 (*This).y -= b.y;
 (*This).z -= b.z;
 return *This;
}
__device__
G4ThreeVector G4ThreeVector_mult_assign( G4ThreeVector *This, G4double m )
{
 (*This).x *= m;
 (*This).y *= m;
 (*This).z *= m;
 return *This;
}
__device__
G4ThreeVector G4ThreeVector_negation( G4ThreeVector a )
{
 return G4ThreeVector_create( -a.x, -a.y, -a.z );
}
__device__
G4double G4ThreeVector_mag2( G4ThreeVector v )
{
 return v.x*v.x + v.y*v.y + v.z*v.z;
}
__device__
G4double G4ThreeVector_mag( G4ThreeVector v )
{
 return sqrt(G4ThreeVector_mag2(v));
}

__device__
G4double G4ThreeVector_dot( G4ThreeVector a, G4ThreeVector b )
{
 return a.x*b.x + a.y*b.y + a.z*b.z;
}

__device__
G4ThreeVector G4ThreeVector_cross( G4ThreeVector a, G4ThreeVector p )
{
 return G4ThreeVector_create(
  a.y*p.z-p.y*a.z,
  a.z*p.x-p.z*a.x,
  a.x*p.y-p.x*a.y );
}

__device__
G4ThreeVector G4ThreeVector_mult( G4ThreeVector a, G4double m )
{
 return G4ThreeVector_create( a.x*m, a.y*m, a.z*m );
}

__device__
G4ThreeVector G4ThreeVector_unit( G4ThreeVector v )
{
 G4double l = G4ThreeVector_mag(v);
 if ( l > 0 )
  return G4ThreeVector_mult( v, 1.0/l );
 return v;
}

__device__
G4bool G4ThreeVector_equal( G4ThreeVector a, G4ThreeVector b )
{
 return a.x == b.x && a.y == b.y && a.z == b.z;
}

__device__
G4double G4ThreeVector_diff2( G4ThreeVector a, G4ThreeVector b )
{
 return G4ThreeVector_mag2( G4ThreeVector_subtract(a,b) );
}
__device__
G4double G4ThreeVector_coord( G4ThreeVector v, EAxis axis )
{
 switch( axis )
 {
 case kXAxis: return v.x;
 case kYAxis: return v.y;
 case kZAxis: return v.z;
 default:
  (void)0;
  return 0;
 }
}
__device__
void G4ThreeVector_set_coord( G4ThreeVector *v, EAxis axis, G4double val )
{
 switch( axis )
 {
 case kXAxis: v->x = val; break;
 case kYAxis: v->y = val; break;
 case kZAxis: v->z = val; break;
 default:
  (void)0;
  break;
 }
}
typedef struct
{
 G4ThreeVector pos, dir;
}
StubParticle;
typedef struct
{
 G4ThreeVector pos, dir;
 G4double t;
}
ParticleWithLifetime;
typedef StubParticle Particle;
__device__ void Prefix_Sum ( int * input, int * output, int length)
{
 int tid = (blockIdx.x * blockDim.x + threadIdx.x);
 int offset = 1;
 if ( tid< length)
  output[tid] = input[ tid ];
 for(int d = length>>1; d > 0; d >>=1)
 {
__syncthreads();
  if(tid<d)
  {
   int ai = offset*(2*tid + 1) - 1;
   int bi = offset*(2*tid + 2) - 1;
   output[bi] += output[ai];
  }
  offset *= 2;
 }
 if(tid == 0)
 {
  output[length - 1] = 0;
 }
 for(int d = 1; d < length ; d *= 2)
 {
  offset >>=1;
  __syncthreads();
  if(tid < d)
  {
   int ai = offset*(2*tid + 1) - 1;
   int bi = offset*(2*tid + 2) - 1;
   float t = output[ai];
   output[ai] = output[bi];
   output[bi] += t;
  }
 }
__syncthreads();
}
__device__
G4bool NoStepReduction( G4bool * noStepArray, int length )
{
 int tid = (blockIdx.x * blockDim.x + threadIdx.x);
 int offset = 1;
 for(int d = length>>1; d > 0; d >>=1)
 {
     __syncthreads();
  if(tid<d)
  {
   int ai = offset*(2*tid + 1) - 1;
   int bi = offset*(2*tid + 2) - 1;
   noStepArray[bi] = (noStepArray[ai] || noStepArray[bi]);
  }
  offset *= 2;
 }
 G4bool result = noStepArray[ length - 1 ];
 __syncthreads();
 return result;
}
typedef struct
{
 G4double
  rxx, rxy, rxz,
  ryx, ryy, ryz,
  rzx, rzy, rzz;
 G4double align;
}
G4RotationMatrix;
__device__
G4RotationMatrix G4RotationMatrix_create_elements
   (G4double mxx, G4double mxy, G4double mxz,
    G4double myx, G4double myy, G4double myz,
    G4double mzx, G4double mzy, G4double mzz)
{
 G4RotationMatrix r =
  { mxx,mxy,mxz, myx,myy,myz, mzx,mzy,mzz
  , 0
  };
 return r;
}
__device__
G4ThreeVector G4RotationMatrix_apply (const G4RotationMatrix *This, G4ThreeVector p)
{
  return G4ThreeVector_create(
     This->rxx*p.x + This->rxy*p.y + This->rxz*p.z,
                    This->ryx*p.x + This->ryy*p.y + This->ryz*p.z,
                    This->rzx*p.x + This->rzy*p.y + This->rzz*p.z);
}
__device__
G4RotationMatrix G4RotationMatrix_mult (const G4RotationMatrix *This, const G4RotationMatrix *other)
{
 return G4RotationMatrix_create_elements(
  This->rxx*(*other).rxx + This->rxy*(*other).ryx + This->rxz*(*other).rzx,
  This->rxx*(*other).rxy + This->rxy*(*other).ryy + This->rxz*(*other).rzy,
  This->rxx*(*other).rxz + This->rxy*(*other).ryz + This->rxz*(*other).rzz,
  This->ryx*(*other).rxx + This->ryy*(*other).ryx + This->ryz*(*other).rzx,
  This->ryx*(*other).rxy + This->ryy*(*other).ryy + This->ryz*(*other).rzy,
  This->ryx*(*other).rxz + This->ryy*(*other).ryz + This->ryz*(*other).rzz,
  This->rzx*(*other).rxx + This->rzy*(*other).ryx + This->rzz*(*other).rzx,
  This->rzx*(*other).rxy + This->rzy*(*other).ryy + This->rzz*(*other).rzy,
  This->rzx*(*other).rxz + This->rzy*(*other).ryz + This->rzz*(*other).rzz );
}
__device__
G4RotationMatrix G4RotationMatrix_transform(G4RotationMatrix *This, const G4RotationMatrix *other)
{
 *This = G4RotationMatrix_mult(other,This);
 return *This;
}
__device__
G4RotationMatrix G4RotationMatrix_inverse(const G4RotationMatrix *This)
{
 return G4RotationMatrix_create_elements(
  This->rxx, This->ryx, This->rzx,
  This->rxy, This->ryy, This->rzy,
  This->rxz, This->ryz, This->rzz );
}
__device__
G4RotationMatrix G4RotationMatrix_invert(G4RotationMatrix *This)
{
 return *This = G4RotationMatrix_inverse(This);
}
typedef struct
{
  G4double rxx,rxy,rxz;
  G4double ryx,ryy,ryz;
  G4double rzx,rzy,rzz;
  G4double tx,ty,tz;
}
G4AffineTransform;
__device__
void G4AffineTransform_ctor_id( G4AffineTransform *This )
{
 This->rxx = 1;
 This->ryy = 1;
 This->rzz = 1;
 This->rxy = 0;
 This->rxz = 0;
 This->ryx = 0;
 This->ryz = 0;
 This->rzx = 0;
 This->rzy = 0;
 This->tx = 0;
 This->ty = 0;
 This->tz = 0;
}
__device__
void G4AffineTransform_ctor_vector( G4AffineTransform *This, G4ThreeVector tlate)
{
 G4AffineTransform_ctor_id( This );
 This->tx = tlate.x;
 This->ty = tlate.y;
 This->tz = tlate.z;
}
__device__
void G4AffineTransform_ctor_matrix( G4AffineTransform *This, G4RotationMatrix rot)
{
 G4AffineTransform_ctor_id( This );
 This->rxx = rot.rxx;
 This->ryy = rot.ryy;
 This->rzz = rot.rzz;
 This->rxy = rot.rxy;
 This->rxz = rot.rxz;
 This->ryx = rot.ryx;
 This->ryz = rot.ryz;
 This->rzx = rot.rzx;
 This->rzy = rot.rzy;
}
__device__
void G4AffineTransform_ctor_full(
 G4AffineTransform *This, G4RotationMatrix rot, G4ThreeVector tlate )
{
 This->rxx = rot.rxx;
 This->ryy = rot.ryy;
 This->rzz = rot.rzz;
 This->rxy = rot.rxy;
 This->rxz = rot.rxz;
 This->ryx = rot.ryx;
 This->ryz = rot.ryz;
 This->rzx = rot.rzx;
 This->rzy = rot.rzy;
 This->tx = tlate.x;
 This->ty = tlate.y;
 This->tz = tlate.z;
}
__device__
void G4AffineTransform_ctor_ptr(
 G4AffineTransform *This, const G4RotationMatrix *rot, G4ThreeVector tlate )
{
 if (rot) G4AffineTransform_ctor_full( This, *rot, tlate );
 else G4AffineTransform_ctor_vector( This, tlate );
}
__device__
void G4AffineTransform_ctor_elements(
  G4AffineTransform *This,
  const G4double prxx,const G4double prxy,const G4double prxz,
  const G4double pryx,const G4double pryy,const G4double pryz,
  const G4double przx,const G4double przy,const G4double przz,
  const G4double ptx,const G4double pty,const G4double ptz)
{
 This->rxx = prxx;
 This->ryy = pryy;
 This->rzz = przz;
 This->rxy = prxy;
 This->rxz = prxz;
 This->ryx = pryx;
 This->ryz = pryz;
 This->rzx = przx;
 This->rzy = przy;
 This->tx = ptx;
 This->ty = pty;
 This->tz = ptz;
}
__device__
G4AffineTransform G4AffineTransform_create_id(void)
{
 G4AffineTransform t;
 G4AffineTransform_ctor_id(&t);
 return t;
}
__device__
G4AffineTransform G4AffineTransform_create_vector(G4ThreeVector tlate)
{
 G4AffineTransform t;
 G4AffineTransform_ctor_vector(&t,tlate);
 return t;
}
__device__
G4AffineTransform G4AffineTransform_create_matrix( G4RotationMatrix rot )
{
 G4AffineTransform t;
 G4AffineTransform_ctor_matrix(&t,rot);
 return t;
}
__device__
G4AffineTransform G4AffineTransform_create_full(
 G4RotationMatrix rot, G4ThreeVector tlate )
{
 G4AffineTransform t;
 G4AffineTransform_ctor_full(&t,rot,tlate);
 return t;
}
__device__
G4AffineTransform G4AffineTransform_create_ptr(
 const G4RotationMatrix *rot, G4ThreeVector tlate )
{
 G4AffineTransform t;
 G4AffineTransform_ctor_ptr(&t,rot,tlate);
 return t;
}
__device__
G4AffineTransform G4AffineTransform_create_elements(
  const G4double prxx,const G4double prxy,const G4double prxz,
  const G4double pryx,const G4double pryy,const G4double pryz,
  const G4double przx,const G4double przy,const G4double przz,
  const G4double ptx,const G4double pty,const G4double ptz)
{
 G4AffineTransform t;
 G4AffineTransform_ctor_elements(&t,
  prxx,prxy,prxz,
  pryx,pryy,pryz,
  przx,przy,przz,
  ptx,pty,ptz);
 return t;
}
__device__ G4AffineTransform
G4AffineTransform_InverseProduct(
 G4AffineTransform *This,
 const G4AffineTransform* ptrtf1,
 const G4AffineTransform* ptrtf2)
{
        G4double itf2tx = - (*ptrtf2).tx*(*ptrtf2).rxx - (*ptrtf2).ty*(*ptrtf2).rxy - (*ptrtf2).tz*(*ptrtf2).rxz;
        G4double itf2ty = - (*ptrtf2).tx*(*ptrtf2).ryx - (*ptrtf2).ty*(*ptrtf2).ryy - (*ptrtf2).tz*(*ptrtf2).ryz;
        G4double itf2tz = - (*ptrtf2).tx*(*ptrtf2).rzx - (*ptrtf2).ty*(*ptrtf2).rzy - (*ptrtf2).tz*(*ptrtf2).rzz;
        This->rxx=(*ptrtf1).rxx*(*ptrtf2).rxx+(*ptrtf1).rxy*(*ptrtf2).rxy+(*ptrtf1).rxz*(*ptrtf2).rxz;
        This->rxy=(*ptrtf1).rxx*(*ptrtf2).ryx+(*ptrtf1).rxy*(*ptrtf2).ryy+(*ptrtf1).rxz*(*ptrtf2).ryz;
        This->rxz=(*ptrtf1).rxx*(*ptrtf2).rzx+(*ptrtf1).rxy*(*ptrtf2).rzy+(*ptrtf1).rxz*(*ptrtf2).rzz;
        This->ryx=(*ptrtf1).ryx*(*ptrtf2).rxx+(*ptrtf1).ryy*(*ptrtf2).rxy+(*ptrtf1).ryz*(*ptrtf2).rxz;
        This->ryy=(*ptrtf1).ryx*(*ptrtf2).ryx+(*ptrtf1).ryy*(*ptrtf2).ryy+(*ptrtf1).ryz*(*ptrtf2).ryz;
        This->ryz=(*ptrtf1).ryx*(*ptrtf2).rzx+(*ptrtf1).ryy*(*ptrtf2).rzy+(*ptrtf1).ryz*(*ptrtf2).rzz;
        This->rzx=(*ptrtf1).rzx*(*ptrtf2).rxx+(*ptrtf1).rzy*(*ptrtf2).rxy+(*ptrtf1).rzz*(*ptrtf2).rxz;
        This->rzy=(*ptrtf1).rzx*(*ptrtf2).ryx+(*ptrtf1).rzy*(*ptrtf2).ryy+(*ptrtf1).rzz*(*ptrtf2).ryz;
        This->rzz=(*ptrtf1).rzx*(*ptrtf2).rzx+(*ptrtf1).rzy*(*ptrtf2).rzy+(*ptrtf1).rzz*(*ptrtf2).rzz;
        This->tx=(*ptrtf1).tx*(*ptrtf2).rxx+(*ptrtf1).ty*(*ptrtf2).rxy+(*ptrtf1).tz*(*ptrtf2).rxz+itf2tx;
        This->ty=(*ptrtf1).tx*(*ptrtf2).ryx+(*ptrtf1).ty*(*ptrtf2).ryy+(*ptrtf1).tz*(*ptrtf2).ryz+itf2ty;
        This->tz=(*ptrtf1).tx*(*ptrtf2).rzx+(*ptrtf1).ty*(*ptrtf2).rzy+(*ptrtf1).tz*(*ptrtf2).rzz+itf2tz;
        return *This;
}
__device__
G4ThreeVector G4AffineTransform_TransformPoint(const G4AffineTransform *This, G4ThreeVector vec)
{
        return G4ThreeVector_create(
   vec.x*This->rxx + vec.y*This->ryx + vec.z*This->rzx + This->tx,
   vec.x*This->rxy + vec.y*This->ryy + vec.z*This->rzy + This->ty,
   vec.x*This->rxz + vec.y*This->ryz + vec.z*This->rzz + This->tz );
}
__device__
G4ThreeVector G4AffineTransform_TransformAxis(const G4AffineTransform *This, G4ThreeVector axis)
{
  return G4ThreeVector_create(
   axis.x*This->rxx + axis.y*This->ryx + axis.z*This->rzx,
   axis.x*This->rxy + axis.y*This->ryy + axis.z*This->rzy,
   axis.x*This->rxz + axis.y*This->ryz + axis.z*This->rzz );
}
__device__
G4AffineTransform G4AffineTransform_Inverse(const G4AffineTransform *This)
{
        return G4AffineTransform_create_elements(
    This->rxx, This->ryx, This->rzx,
    This->rxy, This->ryy, This->rzy,
    This->rxz, This->ryz, This->rzz,
    -This->tx*This->rxx - This->ty*This->rxy - This->tz*This->rxz,
    -This->tx*This->ryx - This->ty*This->ryy - This->tz*This->ryz,
    -This->tx*This->rzx - This->ty*This->rzy - This->tz*This->rzz );
}
__device__
G4AffineTransform G4AffineTransform_Invert(G4AffineTransform *This)
{
        G4double v1 = -This->tx*This->rxx - This->ty*This->rxy - This->tz*This->rxz;
        G4double v2 = -This->tx*This->ryx - This->ty*This->ryy - This->tz*This->ryz;
        G4double v3 = -This->tx*This->rzx - This->ty*This->rzy - This->tz*This->rzz;
        This->tx=v1; This->ty=v2; This->tz=v3;
        G4double tmp1=This->ryx; This->ryx=This->rxy; This->rxy=tmp1;
        G4double tmp2=This->rzx; This->rzx=This->rxz; This->rxz=tmp2;
        G4double tmp3=This->rzy; This->rzy=This->ryz; This->ryz=tmp3;
        return *This;
}
__device__
G4ThreeVector G4AffineTransform_NetTranslation(const G4AffineTransform *This)
{
        return G4ThreeVector_create(This->tx,This->ty,This->tz);
}
__device__
G4bool G4AffineTransform_IsRotated(const G4AffineTransform *This)
{
        return (This->rxx==1.0 && This->ryy==1.0 && This->rzz==1.0) ? false : true;
}
typedef struct
{
 G4double property;
}
StubMaterial;
struct G4SmartVoxelProxy;
typedef struct
{
 G4double fmaxExtent;
 G4double fminExtent;
 struct G4SmartVoxelProxy* * fslices;
 G4int fNumSlices;
 G4int fminEquivalent;
 G4int fmaxEquivalent;
 EAxis faxis;
 EAxis fparamAxis;
}
G4SmartVoxelHeader;
typedef struct
{
 G4int *fcontents;
 G4int fminEquivalent;
 G4int fmaxEquivalent;
 G4int fNumContents;
}
G4SmartVoxelNode;
typedef struct G4SmartVoxelProxy
{
 G4SmartVoxelHeader* fHeader;
    G4SmartVoxelNode* fNode;
}
G4SmartVoxelProxy;
__device__
void G4VoxelNode_ctor( G4SmartVoxelNode *This, G4int no )
{
 This->fmaxEquivalent = no;
 This->fminEquivalent = no;
 This->fcontents = 0;
 This->fNumContents = 0;
}
__device__ G4int
G4VoxelNode_GetNoContained( const G4SmartVoxelNode *This)
{
 return This->fNumContents;
}
__device__ G4int
G4VoxelNode_GetVolume(
 const G4SmartVoxelNode *This, G4int contentNo)
{
 (void)0;
 return This->fcontents[contentNo];
}
__device__ G4int
G4VoxelNode_GetMaxEquivalentSliceNo(
 const G4SmartVoxelNode *This )
{
 return This->fmaxEquivalent;
}
__device__ G4int
G4VoxelNode_GetMinEquivalentSliceNo(
 const G4SmartVoxelNode *This )
{
 return This->fminEquivalent;
}
__device__ G4int
G4VoxelHeader_GetMaxEquivalentSliceNo(
 const G4SmartVoxelHeader *This )
{
 return This->fmaxEquivalent;
}
__device__ G4int
G4VoxelHeader_GetMinEquivalentSliceNo(
 const G4SmartVoxelHeader *This )
{
 return This->fminEquivalent;
}
__device__ EAxis
G4VoxelHeader_GetAxis( const G4SmartVoxelHeader *This )
{
 return This->faxis;
}
__device__ G4int
G4VoxelHeader_GetNoSlices( const G4SmartVoxelHeader *This )
{
 return This->fNumSlices;
}
__device__ G4double
G4VoxelHeader_GetMinExtent( const G4SmartVoxelHeader *This )
{
 return This->fminExtent;
}
__device__ G4double
G4VoxelHeader_GetMaxExtent( const G4SmartVoxelHeader *This )
{
 return This->fmaxExtent;
}
__device__ G4SmartVoxelProxy*
G4VoxelHeader_GetSlice( const G4SmartVoxelHeader *This, G4int n )
{
 (void)0;
 return This->fslices[n];
}
__device__ G4bool
G4VoxelProxy_IsNode( const G4SmartVoxelProxy *This )
{
 return This->fNode != 0;
}
__device__ G4bool
G4VoxelProxy_IsHeader( const G4SmartVoxelProxy *This )
{
 return This->fHeader != 0;
}
__device__ G4SmartVoxelNode*
G4VoxelProxy_GetNode( const G4SmartVoxelProxy *This )
{
 return This->fNode;
}
__device__ G4SmartVoxelHeader*
G4VoxelProxy_GetHeader( const G4SmartVoxelProxy *This )
{
 return This->fHeader;
}
struct G4VPhysicalVolume;
struct G4VSolid;
typedef struct
{
 G4int fNoDaughters;
 struct G4VPhysicalVolume * *fDaughters;
   int check;
   StubMaterial* fMaterial;
 struct G4VSolid* fSolid;
 G4SmartVoxelHeader *fVoxel;
 int align;
}
G4LogicalVolume;
typedef struct G4VSolid
{
 ESolid type;
}
G4VSolid;
__device__
EInside G4VSolid_Inside( const G4VSolid *This, G4ThreeVector p);
__device__
G4ThreeVector G4VSolid_SurfaceNormal( const G4VSolid *This, G4ThreeVector p);
__device__
G4double G4VSolid_DistanceToIn_full(
    const G4VSolid *This,
    G4ThreeVector p,
    G4ThreeVector v);
__device__
G4double G4VSolid_DistanceToIn( const G4VSolid *This, G4ThreeVector p);
__device__
G4double G4VSolid_DistanceToOut_full(
      const G4VSolid *This,
      G4ThreeVector p,
      G4ThreeVector v,
      const G4bool calcNorm,
      G4bool *validNorm,
      G4ThreeVector *n);
__device__
G4double G4VSolid_DistanceToOut( const G4VSolid *This, G4ThreeVector p);
typedef struct
{
 G4VSolid solid;
    G4double fDx,fDy,fDz;
}
G4Box;
extern "C" {
__device__ EInside G4Box_Inside( const G4Box *This, G4ThreeVector p);
__device__ G4ThreeVector G4Box_SurfaceNormal( const G4Box *This, G4ThreeVector p);
__device__ G4double G4Box_DistanceToIn_full(
    const G4Box *This,
    G4ThreeVector p,
    G4ThreeVector v);
__device__ G4double G4Box_DistanceToIn( const G4Box *This, G4ThreeVector p);
__device__ G4double G4Box_DistanceToOut_full(
      const G4Box *This,
      G4ThreeVector p,
      G4ThreeVector v,
      const G4bool calcNorm,
      G4bool *validNorm,
      G4ThreeVector *n);
__device__ G4double G4Box_DistanceToOut( const G4Box *This, G4ThreeVector p);
__device__
G4ThreeVector G4Box_ApproxSurfaceNormal( const G4Box *This, G4ThreeVector p )
{
  G4double distx, disty, distz ;
  G4ThreeVector norm ;
  distx = fabs(fabs(p.x) - This->fDx) ;
  disty = fabs(fabs(p.y) - This->fDy) ;
  distz = fabs(fabs(p.z) - This->fDz) ;
  if ( distx <= disty )
  {
    if ( distx <= distz )
    {
      if ( p.x < 0 ) norm = G4ThreeVector_create(-1.0,0,0) ;
      else norm = G4ThreeVector_create( 1.0,0,0) ;
    }
    else
    {
      if ( p.z < 0 ) norm = G4ThreeVector_create(0,0,-1.0) ;
      else norm = G4ThreeVector_create(0,0, 1.0) ;
    }
  }
  else
  {
    if ( disty <= distz )
    {
      if ( p.y < 0 ) norm = G4ThreeVector_create(0,-1.0,0) ;
      else norm = G4ThreeVector_create(0, 1.0,0) ;
    }
    else
    {
      if ( p.z < 0 ) norm = G4ThreeVector_create(0,0,-1.0) ;
      else norm = G4ThreeVector_create(0,0, 1.0) ;
    }
  }
  return norm;
}
__device__
G4ThreeVector G4Box_SurfaceNormal( const G4Box *This, G4ThreeVector p)
{
  G4double distx, disty, distz ;
  G4ThreeVector norm ;
  const G4double kCarTolerance = 1E-3;
  distx = fabs(fabs(p.x) - This->fDx) ;
  disty = fabs(fabs(p.y) - This->fDy) ;
  distz = fabs(fabs(p.z) - This->fDz) ;
  const G4double delta = 0.5*kCarTolerance;
  const G4ThreeVector nX = G4ThreeVector_create( 1.0, 0,0 );
  const G4ThreeVector nmX = G4ThreeVector_create(-1.0, 0,0 );
  const G4ThreeVector nY = G4ThreeVector_create( 0, 1.0,0 );
  const G4ThreeVector nmY = G4ThreeVector_create( 0,-1.0,0 );
  const G4ThreeVector nZ = G4ThreeVector_create( 0, 0, 1.0);
  const G4ThreeVector nmZ = G4ThreeVector_create( 0, 0,- 1.0);
  G4ThreeVector
 normX = G4ThreeVector_create(0.,0.,0.),
 normY = G4ThreeVector_create(0.,0.,0.),
 normZ = G4ThreeVector_create(0.,0.,0.);
  G4ThreeVector sumnorm = G4ThreeVector_create(0., 0., 0.);
  G4int noSurfaces=0;
  if (distx <= delta)
  {
    noSurfaces ++;
    if ( p.x >= 0.){
      normX= nX ;
    }else{
      normX= nmX;
    }
    sumnorm= normX;
  }
  if (disty <= delta)
  {
    noSurfaces ++;
    if ( p.y >= 0.){
      normY= nY;
    }else{
      normY = nmY;
    }
    G4ThreeVector_sum_assign( &sumnorm, normY );
  }
  if (distz <= delta)
  {
    noSurfaces ++;
    if ( p.z >= 0.){
      normZ= nZ;
    }else{
      normZ = nmZ;
    }
    G4ThreeVector_sum_assign( &sumnorm, normZ );
  }
  const G4double invSqrt2 = 1.0 / sqrt( 2.0);
  const G4double invSqrt3 = 1.0 / sqrt( 3.0);
  norm= G4ThreeVector_create( 0., 0., 0.);
  if( noSurfaces > 0 )
  {
    if( noSurfaces == 1 ){
      norm= sumnorm;
    }else{
      if( noSurfaces == 2 ) {
        norm = G4ThreeVector_mult(sumnorm, invSqrt2);
      } else {
        norm = G4ThreeVector_mult(sumnorm, invSqrt3);
      }
    }
  }else{
     norm = G4Box_ApproxSurfaceNormal(This, p);
  }
  return norm;
}
__device__
G4double G4Box_DistanceToIn_full( const G4Box *This, G4ThreeVector p,G4ThreeVector v)
{
  G4double safx, safy, safz ;
  G4double smin=0.0, sminy, sminz ;
  G4double smax=kInfinity, smaxy, smaxz ;
  G4double stmp ;
  G4double sOut=kInfinity, sOuty=kInfinity, sOutz=kInfinity ;
  const G4double kCarTolerance = 1E-3;
  safx = fabs(p.x) - This->fDx ;
  safy = fabs(p.y) - This->fDy ;
  safz = fabs(p.z) - This->fDz ;
  if ( ((p.x*v.x >= 0.0) && safx > -kCarTolerance*0.5)
       || ((p.y*v.y >= 0.0) && safy > -kCarTolerance*0.5)
       || ((p.z*v.z >= 0.0) && safz > -kCarTolerance*0.5) )
  {
    return kInfinity ;
  }
  if ( v.x)
  {
    stmp = 1.0/fabs(v.x) ;
    if (safx >= 0.0)
    {
      smin = safx*stmp ;
      smax = (This->fDx+fabs(p.x))*stmp ;
    }
    else
    {
      if (v.x > 0) sOut = (This->fDx - p.x)*stmp ;
      if (v.x < 0) sOut = (This->fDx + p.x)*stmp ;
    }
  }
  if ( v.y)
  {
    stmp = 1.0/fabs(v.y) ;
    if (safy >= 0.0)
    {
      sminy = safy*stmp ;
      smaxy = (This->fDy+fabs(p.y))*stmp ;
      if (sminy > smin) smin=sminy ;
      if (smaxy < smax) smax=smaxy ;
      if (smin >= smax-kCarTolerance*0.5)
      {
        return kInfinity ;
      }
    }
    else
    {
      if (v.y > 0) sOuty = (This->fDy - p.y)*stmp ;
      if (v.y < 0) sOuty = (This->fDy + p.y)*stmp ;
      if( sOuty < sOut ) sOut = sOuty ;
    }
  }
  if ( v.z )
  {
    stmp = 1.0/fabs(v.z) ;
    if ( safz >= 0.0)
    {
      sminz = safz*stmp ;
      smaxz = (This->fDz+fabs(p.z))*stmp ;
      if (sminz > smin) smin = sminz ;
      if (smaxz < smax) smax = smaxz ;
      if (smin >= smax-kCarTolerance*0.5)
      {
        return kInfinity ;
      }
    }
    else
    {
      if (v.z > 0) sOutz = (This->fDz - p.z)*stmp ;
      if (v.z < 0) sOutz = (This->fDz + p.z)*stmp ;
      if( sOutz < sOut ) sOut = sOutz ;
    }
  }
  if ( sOut <= smin + 0.5*kCarTolerance)
  {
    return kInfinity ;
  }
  if (smin < 0.5*kCarTolerance) smin = 0.0 ;
  return smin ;
}
__device__
G4double G4Box_DistanceToIn( const G4Box *This, G4ThreeVector p)
{
  G4double safex, safey, safez, safe = 0.0 ;
  safex = fabs(p.x) - This->fDx ;
  safey = fabs(p.y) - This->fDy ;
  safez = fabs(p.z) - This->fDz ;
  if (safex > safe) safe = safex ;
  if (safey > safe) safe = safey ;
  if (safez > safe) safe = safez ;
  return safe ;
}
__device__
G4double G4Box_DistanceToOut_full( const G4Box *This, G4ThreeVector p,G4ThreeVector v,
                               const G4bool calcNorm,
                                G4bool *validNorm,G4ThreeVector *n)
{
  const G4double kCarTolerance = 1E-3;
  enum {kBoxUndefined,kPX,kMX,kPY,kMY,kPZ,kMZ} side = kBoxUndefined ;
  G4double pdist,stmp,snxt;
  if (calcNorm) *validNorm = true ;
  if (v.x > 0)
  {
    pdist = This->fDx - p.x ;
    if (pdist > kCarTolerance*0.5)
    {
      snxt = pdist/v.x ;
      side = kPX ;
    }
    else
    {
      if (calcNorm) *n = G4ThreeVector_create(1,0,0) ;
      return snxt = 0 ;
    }
  }
  else if (v.x < 0)
  {
    pdist = This->fDx + p.x ;
    if (pdist > kCarTolerance*0.5)
    {
      snxt = -pdist/v.x ;
      side = kMX ;
    }
    else
    {
      if (calcNorm) *n = G4ThreeVector_create(-1,0,0) ;
      return snxt = 0 ;
    }
  }
  else snxt = kInfinity ;
  if ( v.y > 0 )
  {
    pdist=This->fDy-p.y;
    if (pdist>kCarTolerance*0.5)
    {
      stmp=pdist/v.y;
      if (stmp<snxt)
      {
        snxt=stmp;
        side=kPY;
      }
    }
    else
    {
      if (calcNorm) *n = G4ThreeVector_create(0,1,0) ;
      return snxt = 0 ;
    }
  }
  else if ( v.y < 0 )
  {
    pdist = This->fDy + p.y ;
    if (pdist > kCarTolerance*0.5)
    {
      stmp=-pdist/v.y;
      if (stmp<snxt)
      {
        snxt=stmp;
        side=kMY;
      }
    }
    else
    {
      if (calcNorm) *n = G4ThreeVector_create(0,-1,0) ;
      return snxt = 0 ;
    }
  }
  if (v.z>0)
  {
    pdist=This->fDz-p.z;
    if (pdist > kCarTolerance*0.5)
    {
      stmp=pdist/v.z;
      if (stmp < snxt)
      {
        snxt=stmp;
        side=kPZ;
      }
    }
    else
    {
      if (calcNorm) *n = G4ThreeVector_create(0,0,1) ;
      return snxt = 0 ;
    }
  }
  else if (v.z<0)
  {
    pdist = This->fDz + p.z ;
    if (pdist > kCarTolerance*0.5)
    {
      stmp=-pdist/v.z;
      if (stmp < snxt)
      {
        snxt=stmp;
        side=kMZ;
      }
    }
    else
    {
      if (calcNorm) *n = G4ThreeVector_create(0,0,-1) ;
      return snxt = 0 ;
    }
  }
  if (calcNorm)
  {
    switch (side)
    {
      case kPX:
        *n=G4ThreeVector_create(1,0,0);
        break;
      case kMX:
        *n=G4ThreeVector_create(-1,0,0);
        break;
      case kPY:
        *n=G4ThreeVector_create(0,1,0);
        break;
      case kMY:
        *n=G4ThreeVector_create(0,-1,0);
        break;
      case kPZ:
        *n=G4ThreeVector_create(0,0,1);
        break;
      case kMZ:
        *n=G4ThreeVector_create(0,0,-1);
        break;
      default:
        break;
    }
  }
  return snxt;
}
__device__
G4double G4Box_DistanceToOut( const G4Box *This, G4ThreeVector p )
{
  G4double safx1,safx2,safy1,safy2,safz1,safz2,safe=0.0;
  safx1 = This->fDx - p.x ;
  safx2 = This->fDx + p.x ;
  safy1 = This->fDy - p.y ;
  safy2 = This->fDy + p.y ;
  safz1 = This->fDz - p.z ;
  safz2 = This->fDz + p.z ;
  if (safx2 < safx1) safe = safx2 ;
  else safe = safx1 ;
  if (safy1 < safe) safe = safy1 ;
  if (safy2 < safe) safe = safy2 ;
  if (safz1 < safe) safe = safz1 ;
  if (safz2 < safe) safe = safz2 ;
  if (safe < 0) safe = 0 ;
  return safe ;
}
__device__
EInside G4Box_Inside( const G4Box *This, G4ThreeVector p)
{
  const G4double kCarTolerance = 1E-3;
  EInside in = kOutside ;
  if ( fabs(p.x) <= This->fDx - kCarTolerance*0.5 )
  {
    if (fabs(p.y) <= This->fDy - kCarTolerance*0.5 )
    {
      if (fabs(p.z) <= This->fDz - kCarTolerance*0.5 ) in = kInside ;
      else if (fabs(p.z) <= This->fDz + kCarTolerance*0.5 ) in = kSurface ;
    }
    else if (fabs(p.y) <= This->fDy + kCarTolerance*0.5 )
    {
      if (fabs(p.z) <= This->fDz + kCarTolerance*0.5 ) in = kSurface ;
    }
  }
  else if (fabs(p.x) <= This->fDx + kCarTolerance*0.5 )
  {
    if (fabs(p.y) <= This->fDy + kCarTolerance*0.5 )
    {
      if (fabs(p.z) <= This->fDz + kCarTolerance*0.5) in = kSurface ;
    }
  }
  return in ;
}
}
typedef struct
{
 G4VSolid solid;
    G4double fRmax;
    G4double fRmaxTolerance;
 G4double align;
}
G4Orb;
extern "C" {
__device__ EInside G4Orb_Inside( const G4Orb *This, G4ThreeVector p);
__device__ G4ThreeVector G4Orb_SurfaceNormal( const G4Orb *This, G4ThreeVector p);
__device__ G4double G4Orb_DistanceToIn_full(
    const G4Orb *This,
    G4ThreeVector p,
    G4ThreeVector v);
__device__ G4double G4Orb_DistanceToIn( const G4Orb *This, G4ThreeVector p);
__device__ G4double G4Orb_DistanceToOut_full(
      const G4Orb *This,
      G4ThreeVector p,
      G4ThreeVector v,
      const G4bool calcNorm,
      G4bool *validNorm,
      G4ThreeVector *n);
__device__ G4double G4Orb_DistanceToOut( const G4Orb *This, G4ThreeVector p);
__device__
EInside G4Orb_Inside( const G4Orb *This, G4ThreeVector p)
{
  G4double rad2,tolRMax;
  EInside in;
  rad2 = G4ThreeVector_mag2(p);
  G4double rad = sqrt(rad2);
  tolRMax = This->fRmax - This->fRmaxTolerance*0.5 ;
  if ( rad <= tolRMax ) { in = kInside ; }
  else
  {
    tolRMax = This->fRmax + This->fRmaxTolerance*0.5 ;
    if ( rad <= tolRMax ) { in = kSurface ; }
    else { in = kOutside ; }
  }
  return in;
}
__device__
G4ThreeVector G4Orb_SurfaceNormal( const G4Orb *This, G4ThreeVector p)
{
  (void)This;
  return G4ThreeVector_unit(p);
}
__device__
G4double G4Orb_DistanceToIn_full( const G4Orb *This, G4ThreeVector p,G4ThreeVector v)
{
  G4double snxt = kInfinity ;
  G4double rad2, pDotV3d;
  G4double c, d2, s = kInfinity ;
  rad2 = G4ThreeVector_mag2(p);
  pDotV3d = G4ThreeVector_dot(p,v);
  G4double rad = sqrt(rad2);
  c = (rad - This->fRmax)*(rad + This->fRmax);
  if ( c > This->fRmaxTolerance*This->fRmax )
  {
    d2 = pDotV3d*pDotV3d - c ;
    if ( d2 >= 0 )
    {
      s = -pDotV3d - sqrt(d2) ;
      if ( s >= 0 )
      {
        return snxt = s;
      }
    }
    else
    {
      return snxt = kInfinity;
    }
  }
  else
  {
    if ( c > -This->fRmaxTolerance*This->fRmax )
    {
      d2 = pDotV3d*pDotV3d - c ;
      if ( (d2 < This->fRmaxTolerance*This->fRmax) || (pDotV3d >= 0) )
      {
        return snxt = kInfinity;
      }
      else
      {
        return snxt = 0.;
      }
    }
  }
  return snxt;
}
__device__
G4double G4Orb_DistanceToIn( const G4Orb *This, G4ThreeVector p)
{
  G4double safe = 0.0,
           rad = G4ThreeVector_mag(p);
  safe = rad - This->fRmax;
  if( safe < 0 ) { safe = 0.; }
  return safe;
}
__device__
G4double G4Orb_DistanceToOut_full( const G4Orb *This, G4ThreeVector p,G4ThreeVector v,
                               const G4bool calcNorm,
                                G4bool *validNorm,G4ThreeVector *n)
{
 G4double snxt = kInfinity;
  enum {kNull,kRMax} side = kNull;
  G4double rad2,pDotV3d;
  G4ThreeVector ipoint;
  G4double c,d2;
  rad2 = G4ThreeVector_mag2(p);
  pDotV3d = G4ThreeVector_dot(p,v);
  const G4double Rmax_plus = This->fRmax + This->fRmaxTolerance*0.5;
  G4double rad = sqrt(rad2);
  if ( rad <= Rmax_plus )
  {
    c = (rad - This->fRmax)*(rad + This->fRmax);
    if ( c < This->fRmaxTolerance*This->fRmax )
    {
      d2 = pDotV3d*pDotV3d - c;
      if( ( c > -This->fRmaxTolerance*This->fRmax) &&
          ( ( pDotV3d >= 0 ) || ( d2 < 0 )) )
      {
        if(calcNorm)
        {
          *validNorm = true ;
          *n = G4ThreeVector_create(p.x/This->fRmax,p.y/This->fRmax,p.z/This->fRmax) ;
        }
        return snxt = 0;
      }
      else
      {
        snxt = -pDotV3d + sqrt(d2);
        side = kRMax ;
      }
    }
  }
  else
  {
  }
  if (calcNorm)
  {
    switch( side )
    {
      case kRMax:
  ipoint = G4ThreeVector_saxpy(snxt,v,p);
  *n=G4ThreeVector_mult(ipoint,1.0/This->fRmax);
        *validNorm=true;
        break;
      default:
        break;
    }
  }
  return snxt;
}
__device__
G4double G4Orb_DistanceToOut( const G4Orb *This, G4ThreeVector p )
{
   G4double safe=0.0,rad = G4ThreeVector_mag(p);
  safe = This->fRmax - rad;
  if ( safe < 0. ) safe = 0.;
  return safe;
}
}
__device__
EInside G4VSolid_Inside( const G4VSolid *This, G4ThreeVector p)
{
 switch(This->type)
 {
  case kBox:
   return G4Box_Inside(( const G4Box*)This,p);
  case kOrb:
   return G4Orb_Inside(( const G4Orb*)This,p);
  default:
   (void)0;
   return kOutside;
 }
}
__device__
G4ThreeVector G4VSolid_SurfaceNormal( const G4VSolid *This, G4ThreeVector p)
{
 switch(This->type)
 {
  case kBox:
   return G4Box_SurfaceNormal(( const G4Box*)This,p);
  case kOrb:
   return G4Orb_SurfaceNormal(( const G4Orb*)This,p);
  default:
   (void)0;
   return G4ThreeVector_create(0,0,0);
 }
}
__device__
G4double G4VSolid_DistanceToIn_full(
    const G4VSolid *This,
    G4ThreeVector p,
    G4ThreeVector v)
{
 switch(This->type)
 {
  case kBox:
   return G4Box_DistanceToIn_full(( const G4Box*)This,p,v);
  case kOrb:
   return G4Orb_DistanceToIn_full(( const G4Orb*)This,p,v);
  default:
   (void)0;
   return 0;
 }
}
__device__
G4double G4VSolid_DistanceToIn( const G4VSolid *This, G4ThreeVector p)
{
 switch(This->type)
 {
  case kBox:
   return G4Box_DistanceToIn(( const G4Box*)This,p);
  case kOrb:
   return G4Orb_DistanceToIn(( const G4Orb*)This,p);
  default:
   (void)0;
   return 0;
 }
}
__device__
G4double G4VSolid_DistanceToOut_full(
      const G4VSolid *This,
      G4ThreeVector p,
      G4ThreeVector v,
      const G4bool calcNorm,
      G4bool *validNorm,
      G4ThreeVector *n)
{
 switch(This->type)
 {
  case kBox:
   return G4Box_DistanceToOut_full(( const G4Box*)This,p,v,calcNorm,validNorm,n);
  case kOrb:
   return G4Orb_DistanceToOut_full(( const G4Orb*)This,p,v,calcNorm,validNorm,n);
  default:
   (void)0;
   return 0;
 }
}
__device__
G4double G4VSolid_DistanceToOut( const G4VSolid *This, G4ThreeVector p)
{
 switch(This->type)
 {
  case kBox:
   return G4Box_DistanceToOut(( const G4Box*)This,p);
  case kOrb:
   return G4Orb_DistanceToOut(( const G4Orb*)This,p);
  default:
   (void)0;
   return 0;
 }
}
__device__
 G4SmartVoxelHeader * G4LogicalVolume_GetVoxelHeader( const G4LogicalVolume* This)
{
 return This->fVoxel;
}
__device__
G4int G4LogicalVolume_GetNoDaughters( const G4LogicalVolume* This)
{
  return This->fNoDaughters;
}
__device__
 struct G4VPhysicalVolume* G4LogicalVolume_GetDaughter( const G4LogicalVolume* This, const G4int i)
{
  return This->fDaughters[i];
}
__device__
 struct G4VSolid* G4LogicalVolume_GetSolid( const G4LogicalVolume* This)
{
  return This->fSolid;
}
__device__
 StubMaterial* G4LogicalVolume_GetMaterial( const G4LogicalVolume* This)
{
  return This->fMaterial;
}
typedef struct G4VPhysicalVolume
{
    G4RotationMatrix frot;
    G4ThreeVector ftrans;
 int guard1;
    G4LogicalVolume *flogical;
 int guard2;
 G4LogicalVolume *flmother;
 int guard3;
 int count;
 int counter_shadow;
}
G4VPhysicalVolume;
__device__
G4ThreeVector G4VPhysicalVolume_GetTranslation( const G4VPhysicalVolume *This)
{
  return This->ftrans;
}
__device__
 G4LogicalVolume* G4VPhysicalVolume_GetLogicalVolume( const G4VPhysicalVolume *This)
{
  return This->flogical;
}
__device__
 G4LogicalVolume* G4VPhysicalVolume_GetMotherLogical( const G4VPhysicalVolume *This)
{
  return This->flmother;
}
__device__
G4RotationMatrix G4VPhysicalVolume_GetObjectRotationValue( const G4VPhysicalVolume *This)
{
  return This->frot;
}
__device__
G4ThreeVector G4VPhysicalVolume_GetObjectTranslation( const G4VPhysicalVolume *This)
{
 return This->ftrans;
}
typedef struct
{
   G4AffineTransform fTransform;
   G4VPhysicalVolume* fPhysicalVolumePtr;
   EVolume fVolumeType;
}
G4NavigationLevel;
typedef struct
{
 G4NavigationLevel fNavHistory[16];
 G4int fStackDepth;
 int align;
}
G4NavigationHistory;
__device__
void G4NavigationLevel_ctor(
   G4NavigationLevel *This,
   G4VPhysicalVolume* pPhysVol,
   G4AffineTransform afTransform,
   EVolume volTp )
{
 This->fTransform = afTransform;
 This->fPhysicalVolumePtr = pPhysVol;
 This->fVolumeType = volTp;
}
__device__
void G4NavigationLevel_ctor_relative(
   G4NavigationLevel *This,
   G4VPhysicalVolume* pPhysVol,
   G4AffineTransform levelAbove,
   G4AffineTransform relativeCurrent,
   EVolume volTp )
{
 This->fPhysicalVolumePtr = pPhysVol;
 This->fVolumeType = volTp;
 G4AffineTransform_InverseProduct(&(This->fTransform), &levelAbove, &relativeCurrent );
}
__device__
G4NavigationLevel G4NavigationLevel_create(
   G4VPhysicalVolume* pPhysVol,
   G4AffineTransform afTransform,
   EVolume volTp )
{
 G4NavigationLevel lev;
 G4NavigationLevel_ctor( &lev, pPhysVol, afTransform, volTp );
 return lev;
}
__device__
G4NavigationLevel G4NavigationLevel_create_relative(
 G4VPhysicalVolume* pPhysVol,
 G4AffineTransform levelAbove,
 G4AffineTransform relativeCurrent,
 EVolume volTp)
{
 G4NavigationLevel lev;
 G4NavigationLevel_ctor_relative( &lev, pPhysVol, levelAbove, relativeCurrent, volTp );
 return lev;
}
__device__
 G4VPhysicalVolume* G4NavigationLevel_GetPhysicalVolume(
 const G4NavigationLevel *This )
{
  return This->fPhysicalVolumePtr;
}
__device__
G4AffineTransform G4NavigationLevel_GetTransform(
 const G4NavigationLevel *This )
{
  return This->fTransform;
}
__device__
const G4AffineTransform* G4NavigationLevel_GetPtrTransform(
 const G4NavigationLevel *This )
{
  return &(This->fTransform);
}
__device__
EVolume G4NavigationLevel_GetVolumeType(
 const G4NavigationLevel *This )
{
  return This->fVolumeType;
}
__device__
void G4NavigationHistory_Reset( G4NavigationHistory *This )
{
 This->fStackDepth = 0;
}
__device__
void G4NavigationHistory_Clear( G4NavigationHistory *This )
{
  G4AffineTransform origin = G4AffineTransform_create_vector(G4ThreeVector_create(0.,0.,0.));
  G4NavigationLevel tmpNavLevel = G4NavigationLevel_create(0, origin, kNormal) ;
  G4NavigationHistory_Reset( This );
  for (G4int ilev=16 -1; ilev>=0; ilev--)
  {
     This->fNavHistory[ilev] = tmpNavLevel;
  }
}
__device__
void G4NavigationHistory_ctor( G4NavigationHistory *This )
{
 This->fStackDepth = 0;
 G4NavigationHistory_Clear( This );
}
__device__
void G4NavigationHistory_dtor( G4NavigationHistory *This )
{
 (void)This;
}
__device__
void G4NavigationHistory_SetFirstEntry(
 G4NavigationHistory *This, G4VPhysicalVolume* pVol)
{
  G4ThreeVector translation = G4ThreeVector_create(0.,0.,0.);
  if( pVol!=0 )
  {
    translation = G4VPhysicalVolume_GetTranslation( pVol );
  }
  This->fNavHistory[0] =
    G4NavigationLevel_create( pVol, G4AffineTransform_create_vector(translation), kNormal );
}
__device__
const G4AffineTransform* G4NavigationHistory_GetPtrTopTransform(
 const G4NavigationHistory *This )
{
  return G4NavigationLevel_GetPtrTransform( &(This->fNavHistory[This->fStackDepth]) );
}
__device__
G4AffineTransform G4NavigationHistory_GetTopTransform(
 const G4NavigationHistory *This )
{
  return G4NavigationLevel_GetTransform( &(This->fNavHistory[This->fStackDepth]) );
}
__device__
EVolume G4NavigationHistory_GetTopVolumeType(
 const G4NavigationHistory *This )
{
  return G4NavigationLevel_GetVolumeType( &(This->fNavHistory[This->fStackDepth]) );
}
__device__
 G4VPhysicalVolume* G4NavigationHistory_GetTopVolume(
 const G4NavigationHistory *This )
{
  return G4NavigationLevel_GetPhysicalVolume( &(This->fNavHistory[This->fStackDepth]) );
}
__device__
G4int G4NavigationHistory_GetDepth(
 const G4NavigationHistory *This )
{
  return This->fStackDepth;
}
__device__
G4AffineTransform
G4NavigationHistory_GetTransform(
 const G4NavigationHistory *This, G4int n )
{
  return G4NavigationLevel_GetTransform( &(This->fNavHistory[n]) );
}
__device__
EVolume G4NavigationHistory_GetVolumeType(
 const G4NavigationHistory *This, G4int n )
{
  return G4NavigationLevel_GetVolumeType( &(This->fNavHistory[n]) );
}
__device__
 G4VPhysicalVolume* G4NavigationHistory_GetVolume(
 const G4NavigationHistory *This, G4int n )
{
  return G4NavigationLevel_GetPhysicalVolume( &(This->fNavHistory[n]) );
}
__device__
G4int G4NavigationHistory_GetMaxDepth(
 const G4NavigationHistory *This )
{
 (void)This;
 return 16;
}
__device__
void G4NavigationHistory_BackLevel( G4NavigationHistory *This )
{
  (void)0;
  This->fStackDepth--;
}
__device__
void G4NavigationHistory_NewLevel(
  G4NavigationHistory *This,
  G4VPhysicalVolume *pNewMother,
  EVolume vType )
{
  This->fStackDepth++;
  (void)0;
  This->fNavHistory[This->fStackDepth] =
    G4NavigationLevel_create_relative(
   pNewMother,
   G4NavigationLevel_GetTransform( &(This->fNavHistory[This->fStackDepth-1]) ),
   G4AffineTransform_create_full(
    G4VPhysicalVolume_GetObjectRotationValue( pNewMother ),
    G4VPhysicalVolume_GetTranslation( pNewMother )),
   vType );
}
typedef struct{
  G4VPhysicalVolume * PVolume;
  G4int trackId;
  }SolidInfo;
   typedef struct{
   float safety;
   float step;
   int trackId;
   G4VPhysicalVolume * PVolume;
   }ResultInfo;
   typedef struct{
   float safety;
   float step;
   G4VPhysicalVolume * PVolume;
   }
   FinalResult;
   typedef struct{
   G4ThreeVector Point;
   G4ThreeVector Direction;
   }PointInformation;
   __device__ void Find_minimum ( ResultInfo * Result_For_Current_Solid, FinalResult * Compacter_Result, int PrevSum, int size)
{
 int locationId = (blockIdx.x * blockDim.x + threadIdx.x);
 int i, loc ;
 float result_step, result_safety, Current_result_step, Current_result_safety;
 float Initial_result_step = (Compacter_Result [ locationId ]).step;
 float Initial_result_safety = (Compacter_Result [ locationId ]).safety;
 Current_result_step = Initial_result_step;
 Current_result_safety = Initial_result_safety;
 for( i = 0; i < size ; i++)
 {
  result_step = Result_For_Current_Solid[ PrevSum + i].step;
  result_safety = Result_For_Current_Solid[ PrevSum + i].safety;
  if ( result_step < Current_result_step)
  {
   loc = PrevSum + i;
   Current_result_step = result_step;
  }
  if ( result_safety < Current_result_safety)
  {
   Current_result_safety = result_safety;
  }
 }
 if( Current_result_step != Initial_result_step)
 {
  FinalResult final = { Current_result_safety, Current_result_step, (Result_For_Current_Solid[ loc ].PVolume)};
  Compacter_Result[ locationId ] = final;
 }
__syncthreads();
}
typedef struct
{
 G4double fVoxelSliceWidthStack[4];
 G4SmartVoxelHeader* fVoxelHeaderStack[4];
 G4int fVoxelNodeNoStack[4];
 G4int fVoxelNoSlicesStack[4];
 EAxis fVoxelAxisStack[4];
 G4int fVoxelDepth;
 G4SmartVoxelNode *fVoxelNode;
}
G4VoxelNavigation;
__device__ void G4VoxelNavigation_ctor( G4VoxelNavigation *This );
__device__ G4bool G4VoxelNavigation_LevelLocate(
 G4VoxelNavigation *This,
 G4NavigationHistory *history,
 const G4VPhysicalVolume *blockedVol,
 G4ThreeVector globalPoint,
 const G4ThreeVector* globalDirection,
 const G4bool pLocatedOnEdge,
 G4ThreeVector *localPoint );
__device__ G4SmartVoxelNode* G4VoxelNavigation_VoxelLocate(
 G4VoxelNavigation *This,
 G4SmartVoxelHeader *voxelHeader,
 G4ThreeVector point);
__device__
G4double
G4VoxelNavigation_ComputeStep(
   G4VoxelNavigation *This,
   G4ThreeVector localPoint,
   G4ThreeVector localDirection,
   const G4double currentProposedStepLength,
   G4double *newSafety,
   G4NavigationHistory *history,
   G4bool *validExitNormal,
   G4ThreeVector *exitNormal,
   G4bool *exiting,
   G4bool *entering,
   G4VPhysicalVolume *(*pBlockedPhysical)
   , G4double * Result
   );
__device__ G4double G4VoxelNavigation_ComputeSafety(
 G4VoxelNavigation *This,
 G4ThreeVector localPoint,
 const G4NavigationHistory *history);
typedef struct
{
 G4NavigationHistory fHistory;
 G4VoxelNavigation fVoxelNav;
 G4ThreeVector fStepEndPoint;
 G4ThreeVector fLastLocatedPointLocal;
 G4ThreeVector fExitNormal;
 G4ThreeVector fGrandMotherExitNormal;
 G4bool fEnteredDaughter;
 G4bool fExitedMother;
 G4bool fWasLimitedByGeometry;
 G4bool fEntering;
 G4bool fExiting;
 G4bool fLastStepWasZero;
 G4bool fLocatedOnEdge;
 G4bool fLocatedOutsideWorld;
 G4bool fValidExitNormal;
 G4bool fPushed;
 G4int fNumberZeroSteps;
 int align1;
 G4double fPreviousSafety;
 G4VPhysicalVolume *fBlockedPhysicalVolume;
 G4VPhysicalVolume *fTopPhysical;
}
G4Navigator;
__device__ void G4Navigator_ctor( G4Navigator *This );
__device__ void G4Navigator_SetWorldVolume(
 G4Navigator *This,
 G4VPhysicalVolume* pWorld );
__device__ G4VPhysicalVolume* G4Navigator_LocateGlobalPointAndSetup(
  G4Navigator *This,
  G4ThreeVector globalPoint,
  const G4ThreeVector* pGlobalDirection,
  G4bool relativeSearch,
  G4bool ignoreDirection,
  float * Result);
__device__
G4double G4Navigator_ComputeStep(
  G4Navigator *This,
  G4ThreeVector pGlobalpoint,
  G4ThreeVector pDirection,
  const G4double pCurrentProposedStepLength,
  G4double *pNewSafety
   , G4bool cur_vol_local
   , G4double * Result
  );
__device__ void G4Navigator_SetGeometricallyLimitedStep( G4Navigator *This );
__device__ G4double G4NormalNavigation_ComputeStep(
 G4ThreeVector localPoint,
 G4ThreeVector localDirection,
 const G4double currentProposedStepLength,
 G4double *newSafety,
 G4NavigationHistory *history,
 G4bool *validExitNormal,
 G4ThreeVector *exitNormal,
 G4bool *exiting,
 G4bool *entering,
 G4VPhysicalVolume *(*pBlockedPhysical));
__device__ G4double G4NormalNavigation_ComputeSafety(
 G4ThreeVector localPoint,
 const G4NavigationHistory *history );
__device__ G4bool G4NormalNavigation_LevelLocate(
 G4NavigationHistory *history,
 const G4VPhysicalVolume *blockedVol,
 G4ThreeVector* globalPoint,
 const G4ThreeVector* globalDirection,
 G4bool pLocatedOnEdge,
 G4ThreeVector* localPoint );
  __device__ void G4VoxelNavigation_ctor( G4VoxelNavigation *This );
__device__ void G4Navigator_ResetState( G4Navigator *This )
{
  This->fWasLimitedByGeometry = false;
  This->fEntering = false;
  This->fExiting = false;
  This->fLocatedOnEdge = false;
  This->fLastStepWasZero = false;
  This->fEnteredDaughter = false;
  This->fExitedMother = false;
  This->fPushed = false;
  This->fValidExitNormal = false;
  This->fExitNormal = G4ThreeVector_create(0,0,0);
  This->fPreviousSafety = 0.0;
  This->fNumberZeroSteps = 0;
  This->fBlockedPhysicalVolume = 0;
  This->fLastLocatedPointLocal = G4ThreeVector_create( 1e37, -1e37, 0.0 );
  This->fLocatedOutsideWorld = false;
}
__device__
G4ThreeVector G4Navigator_ComputeLocalAxis( const G4Navigator *This, G4ThreeVector pVec)
{
 G4AffineTransform t =
  G4NavigationHistory_GetTopTransform( &(This->fHistory) );
 return G4AffineTransform_TransformAxis(&t, pVec);
}
__device__ G4ThreeVector
G4Navigator_ComputeLocalPoint( const G4Navigator *This, G4ThreeVector pGlobalPoint)
{
 G4AffineTransform t =
  G4NavigationHistory_GetTopTransform( &(This->fHistory) );
 return G4AffineTransform_TransformPoint(&t, pGlobalPoint);
}
__device__ void G4Navigator_SetWorldVolume( G4Navigator *This, G4VPhysicalVolume* pWorld )
{
 This->fTopPhysical = pWorld;
 G4NavigationHistory_SetFirstEntry( &(This->fHistory), pWorld );
}
__device__ void G4Navigator_SetGeometricallyLimitedStep( G4Navigator *This )
{
 This->fWasLimitedByGeometry = true;
}
__device__
void G4Navigator_ResetStackAndState( G4Navigator *This )
{
 G4NavigationHistory_Reset( &(This->fHistory) );
 G4Navigator_ResetState( This );
}
__device__
EVolume G4Navigator_VolumeType( const G4Navigator *This, const G4VPhysicalVolume *pVol )
{
 (void)This;
 (void)pVol;
 return kNormal;
}
__device__ void G4Navigator_ctor( G4Navigator *This )
{
 G4NavigationHistory_ctor( &(This->fHistory) );
 G4VoxelNavigation_ctor( &(This->fVoxelNav ) );
 G4Navigator_ResetStackAndState( This );
 This->fWasLimitedByGeometry = false;
 This->fTopPhysical = 0;
 This->fPushed = false;
 This->fStepEndPoint = G4ThreeVector_create( kInfinity, kInfinity, kInfinity );
}
__device__
 G4VPhysicalVolume*
G4Navigator_LocateGlobalPointAndSetup(
  G4Navigator *This,
  G4ThreeVector globalPoint,
  const G4ThreeVector* pGlobalDirection,
  G4bool relativeSearch,
  G4bool ignoreDirection,
  float * Result
  )
{
  G4bool notKnownContained=true, noResult;
  G4VPhysicalVolume *targetPhysical;
  G4VSolid *targetSolid = 0;
  G4ThreeVector localPoint = G4ThreeVector_create(0,0,0);
  G4ThreeVector globalDirection = G4ThreeVector_create(0,0,0);
  EInside insideCode;
  G4bool considerDirection = (!ignoreDirection) || This->fLocatedOnEdge;
  if( considerDirection && pGlobalDirection != 0 )
  {
    globalDirection=*pGlobalDirection;
  }
  if ( 1 )
  {
     G4Navigator_ResetStackAndState( This );
  }
  else
  {
    if ( This->fWasLimitedByGeometry )
    {
      This->fWasLimitedByGeometry = false;
      This->fEnteredDaughter = This->fEntering;
      This->fExitedMother = This->fExiting;
      if ( This->fExiting )
      {
        if ( G4NavigationHistory_GetDepth( &(This->fHistory) ) )
        {
          This->fBlockedPhysicalVolume = G4NavigationHistory_GetTopVolume( &(This->fHistory) );
          G4NavigationHistory_BackLevel( &(This->fHistory) );
        }
        else
        {
          This->fLastLocatedPointLocal = localPoint;
          This->fLocatedOutsideWorld = true;
          return 0;
        }
        if ( This->fLocatedOnEdge )
        {
          This->fExiting= false;
        }
      }
      else
        if ( This->fEntering )
        {
    G4NavigationHistory_NewLevel( &(This->fHistory), This->fBlockedPhysicalVolume, kNormal);
          This->fEntering = false;
          This->fBlockedPhysicalVolume = 0;
          G4AffineTransform t = G4NavigationHistory_GetTopTransform( &(This->fHistory) );
          localPoint = G4AffineTransform_TransformPoint(&t,globalPoint);
          notKnownContained = false;
        }
    }
    else
    {
      This->fBlockedPhysicalVolume = 0;
      This->fEntering = false;
      This->fEnteredDaughter = false;
      This->fExiting = false;
      This->fExitedMother = false;
    }
  }
  while (notKnownContained)
  {
 targetSolid =
   G4LogicalVolume_GetSolid(
    G4VPhysicalVolume_GetLogicalVolume(
    G4NavigationHistory_GetTopVolume(&(This->fHistory))));
 G4AffineTransform t = G4NavigationHistory_GetTopTransform( &(This->fHistory) );
 localPoint = G4AffineTransform_TransformPoint(&t,globalPoint);
 insideCode = G4VSolid_Inside(targetSolid,localPoint);
    if ( insideCode==kOutside )
    {
      if ( G4NavigationHistory_GetDepth( &(This->fHistory) ) )
      {
        This->fBlockedPhysicalVolume = G4NavigationHistory_GetTopVolume( &(This->fHistory) );
        G4NavigationHistory_BackLevel( &(This->fHistory) );
        This->fExiting = false;
      }
      else
      {
        This->fLastLocatedPointLocal = localPoint;
        This->fLocatedOutsideWorld = true;
        return 0;
      }
    }
    else
      if ( insideCode==kSurface )
      {
        G4bool isExiting = This->fExiting;
        if( (!This->fExiting)&&considerDirection )
        {
   G4bool directionExiting = false;
   G4AffineTransform t = G4NavigationHistory_GetTopTransform( &(This->fHistory) );
   G4ThreeVector localDirection =G4AffineTransform_TransformAxis(&t,globalDirection);
   G4ThreeVector normal = G4VSolid_SurfaceNormal(targetSolid, localPoint);
   directionExiting = G4ThreeVector_dot(normal,localDirection) > 0.0;
   isExiting = isExiting || directionExiting;
        }
        if( isExiting )
        {
          if ( G4NavigationHistory_GetDepth( &(This->fHistory) ) )
          {
            This->fBlockedPhysicalVolume = G4NavigationHistory_GetTopVolume( &(This->fHistory) );
            G4NavigationHistory_BackLevel( &(This->fHistory) );
            This->fValidExitNormal = false;
          }
          else
          {
            This->fLastLocatedPointLocal = localPoint;
            This->fLocatedOutsideWorld = true;
            return 0;
          }
        }
        else
        {
          notKnownContained=false;
        }
      }
      else
      {
        notKnownContained=false;
      }
  }
  noResult = true;
  do
  {
    targetPhysical = G4NavigationHistory_GetTopVolume( &(This->fHistory) );
  G4LogicalVolume *targetLogical = G4VPhysicalVolume_GetLogicalVolume(targetPhysical);
    if ( G4LogicalVolume_GetVoxelHeader( targetLogical ) != 0 )
    {
  noResult =
   G4VoxelNavigation_LevelLocate(
    &(This->fVoxelNav),
    &(This->fHistory),
    This->fBlockedPhysicalVolume,
    globalPoint,
    pGlobalDirection,
    considerDirection,
    &localPoint);
 }
 else
 {
  noResult = G4NormalNavigation_LevelLocate(
    &(This->fHistory),
    This->fBlockedPhysicalVolume,
    &globalPoint,
    pGlobalDirection,
    considerDirection,
    &localPoint);
 }
    if ( noResult )
    {
      This->fBlockedPhysicalVolume = 0;
      This->fEntering = false;
      This->fEnteredDaughter = true;
    }
  } while (noResult);
  This->fLastLocatedPointLocal = localPoint;
  This->fLocatedOutsideWorld= false;
  return targetPhysical;
}
__device__ void
G4Navigator_LocateGlobalPointWithinVolume( G4Navigator *This, G4ThreeVector pGlobalpoint)
{
 This->fLastLocatedPointLocal = G4Navigator_ComputeLocalPoint( This, pGlobalpoint );
 G4VPhysicalVolume* motherPhysical = G4NavigationHistory_GetTopVolume( &(This->fHistory) );
 G4LogicalVolume* motherLogical = G4VPhysicalVolume_GetLogicalVolume( motherPhysical );
 G4SmartVoxelHeader* pVoxelHeader = G4LogicalVolume_GetVoxelHeader( motherLogical );
 if ( pVoxelHeader )
 {
  G4VoxelNavigation_VoxelLocate( &(This->fVoxelNav), pVoxelHeader, This->fLastLocatedPointLocal );
 }
 This->fBlockedPhysicalVolume = 0;
 This->fEntering = false;
 This->fEnteredDaughter = false;
 This->fExiting = false;
 This->fExitedMother = false;
}
__device__
G4double G4Navigator_ComputeStep(
  G4Navigator *This,
  G4ThreeVector pGlobalpoint,
  G4ThreeVector pDirection,
  const G4double pCurrentProposedStepLength,
  G4double *pNewSafety
   , G4bool cur_vol_local
   , G4double * Result
  )
{
  G4ThreeVector localDirection = G4Navigator_ComputeLocalAxis(This,pDirection);
  G4double Step = 1e37;
  G4VPhysicalVolume *motherPhysical = G4NavigationHistory_GetTopVolume( &(This->fHistory) );
  const G4double kCarTolerance = 1E-3;
  G4LogicalVolume *motherLogical = G4VPhysicalVolume_GetLogicalVolume(motherPhysical);
  G4ThreeVector newLocalPoint = G4Navigator_ComputeLocalPoint( This, pGlobalpoint);
  if( !G4ThreeVector_equal(newLocalPoint, This->fLastLocatedPointLocal) )
  {
    G4ThreeVector oldLocalPoint = This->fLastLocatedPointLocal;
    G4double moveLenSq = G4ThreeVector_diff2(newLocalPoint,oldLocalPoint);
    if ( moveLenSq >= kCarTolerance*kCarTolerance )
    {
      G4Navigator_LocateGlobalPointWithinVolume( This, pGlobalpoint );
    }
  }
  if ( G4LogicalVolume_GetVoxelHeader(motherLogical) != 0 )
  {
 if( cur_vol_local )
 Step = G4VoxelNavigation_ComputeStep(
   &(This->fVoxelNav),
   This->fLastLocatedPointLocal,
   localDirection,
   pCurrentProposedStepLength,
   pNewSafety,
   &(This->fHistory),
   &(This->fValidExitNormal),
   &(This->fExitNormal),
   &(This->fExiting),
   &(This->fEntering),
   &(This->fBlockedPhysicalVolume)
   , Result
   );
 else
     return 0;
  }
  else
  {
 Step = G4NormalNavigation_ComputeStep(
   This->fLastLocatedPointLocal,
   localDirection,
   pCurrentProposedStepLength,
   pNewSafety,
   &(This->fHistory),
   &(This->fValidExitNormal),
   &(This->fExitNormal),
   &(This->fExiting),
   &(This->fEntering),
   &(This->fBlockedPhysicalVolume));
  }
  This->fPreviousSafety = *pNewSafety;
  This->fLocatedOnEdge = This->fLastStepWasZero && (Step==0.0);
  This->fLastStepWasZero = (Step==0.0);
  if (This->fPushed) This->fPushed = This->fLastStepWasZero;
  if ( This->fLastStepWasZero )
  {
    This->fNumberZeroSteps++;
    if( This->fNumberZeroSteps > 10 -1 )
    {
       Step += 0.9*kCarTolerance;
       This->fPushed = true;
    }
    if( This->fNumberZeroSteps > 25 -1 )
    {
   (void)0;
    }
  }
  else
  {
    if (!This->fPushed) This->fNumberZeroSteps = 0;
  }
  This->fEnteredDaughter = This->fEntering;
  This->fExitedMother = This->fExiting;
  if( This->fExiting )
  {
    if(This->fValidExitNormal)
    {
      This->fGrandMotherExitNormal= This->fExitNormal;
    }
    else
    {
      G4ThreeVector finalLocalPoint =
  G4ThreeVector_saxpy( Step, localDirection, This->fLastLocatedPointLocal );
      This->fGrandMotherExitNormal =
  G4VSolid_SurfaceNormal(
   G4LogicalVolume_GetSolid(motherLogical),finalLocalPoint);
      G4RotationMatrix mRot = G4VPhysicalVolume_GetObjectRotationValue(motherPhysical);
      G4RotationMatrix inv = G4RotationMatrix_inverse(&mRot);
      This->fGrandMotherExitNormal
       = G4RotationMatrix_apply(&inv,This->fGrandMotherExitNormal);
    }
  }
  This->fStepEndPoint =
 G4ThreeVector_saxpy(Step, pDirection, pGlobalpoint );
  if( (Step == pCurrentProposedStepLength) && (!This->fExiting) && (!This->fEntering) )
  {
    Step = kInfinity;
  }
  return Step;
}
__device__ G4bool
G4AuxiliaryNavServices_CheckPointOnSurface(
         const G4VSolid* sampleSolid,
                     G4ThreeVector localPoint,
                     const G4ThreeVector* globalDirection,
                     G4AffineTransform sampleTransform,
                     const G4bool locatedOnEdge)
{
  G4ThreeVector localDirection, sampleNormal;
  G4bool enter = false;
  EInside insideSolid =
 G4VSolid_Inside(sampleSolid, localPoint);
  if ( insideSolid!=kOutside )
  {
    G4bool checkDirection= locatedOnEdge && (globalDirection!=0);
    if( (insideSolid==kSurface) && checkDirection)
    {
      localDirection= G4AffineTransform_TransformAxis(&sampleTransform,*globalDirection);
      sampleNormal = G4VSolid_SurfaceNormal(sampleSolid,localPoint);
      if ( G4ThreeVector_dot(sampleNormal,localDirection) <= 0 )
      {
        if( G4ThreeVector_dot(sampleNormal,localDirection) == 0 )
        {
          G4double distanceToIn =
   G4VSolid_DistanceToIn_full( sampleSolid, localPoint, localDirection );
          if( distanceToIn != kInfinity )
          {
            enter = true;
          }
        }
        else
        {
          enter = true;
        }
      }
    }
    else
    {
      enter = true;
    }
  }
  return enter;
}
__device__ G4bool
G4NormalNavigation_LevelLocate(
 G4NavigationHistory *history,
 const G4VPhysicalVolume *blockedVol,
 G4ThreeVector* globalPoint,
 const G4ThreeVector* globalDirection,
 G4bool pLocatedOnEdge,
 G4ThreeVector* localPoint )
{
  G4VPhysicalVolume *targetPhysical, *samplePhysical;
  G4LogicalVolume *targetLogical;
  G4VSolid *sampleSolid;
  G4ThreeVector samplePoint;
  G4int targetNoDaughters;
  targetPhysical = G4NavigationHistory_GetTopVolume(history);
  targetLogical = G4VPhysicalVolume_GetLogicalVolume(targetPhysical);
  targetNoDaughters = G4LogicalVolume_GetNoDaughters(targetLogical);
  if (targetNoDaughters == 0) return false;
  for ( int sampleNo=targetNoDaughters-1; sampleNo>=0; sampleNo-- )
  {
   samplePhysical =
  G4LogicalVolume_GetDaughter(targetLogical,sampleNo);
   if ( samplePhysical!=blockedVol )
   {
  G4NavigationHistory_NewLevel(history, samplePhysical, kNormal );
  sampleSolid =
   G4LogicalVolume_GetSolid(
    G4VPhysicalVolume_GetLogicalVolume(samplePhysical));
  G4AffineTransform tf =
   G4NavigationHistory_GetTopTransform(history);
  samplePoint =
   G4AffineTransform_TransformPoint( &tf, *globalPoint );
  if( G4AuxiliaryNavServices_CheckPointOnSurface(
   sampleSolid, samplePoint, globalDirection,
   tf, pLocatedOnEdge) )
  {
    *localPoint = samplePoint;
    return true;
  }
  else
  {
   G4NavigationHistory_BackLevel(history);
  }
   }
  }
  return false;
}
__device__
G4double
G4NormalNavigation_ComputeStep(
 G4ThreeVector localPoint,
 G4ThreeVector localDirection,
 const G4double currentProposedStepLength,
 G4double *newSafety,
 G4NavigationHistory *history,
 G4bool *validExitNormal,
 G4ThreeVector *exitNormal,
 G4bool *exiting,
 G4bool *entering,
 G4VPhysicalVolume *(*pBlockedPhysical))
{
  G4VPhysicalVolume *motherPhysical, *samplePhysical, *blockedExitedVol=0;
  G4LogicalVolume *motherLogical;
  G4VSolid *motherSolid;
  G4ThreeVector sampleDirection;
  G4double ourStep=currentProposedStepLength, motherSafety, ourSafety;
  G4int localNoDaughters, sampleNo;
  motherPhysical = G4NavigationHistory_GetTopVolume(history);
  motherLogical = G4VPhysicalVolume_GetLogicalVolume(motherPhysical);
  motherSolid = G4LogicalVolume_GetSolid(motherLogical);
  motherSafety = G4VSolid_DistanceToOut(motherSolid,localPoint);
  ourSafety = motherSafety;
  if ( *exiting && *validExitNormal )
  {
    if ( G4ThreeVector_dot(localDirection,*exitNormal)>=kMinExitingNormalCosine )
    {
      blockedExitedVol =* pBlockedPhysical;
      ourSafety = 0;
    }
  }
  *exiting = false;
  *entering = false;
  localNoDaughters = G4LogicalVolume_GetNoDaughters(motherLogical);
  for ( sampleNo=localNoDaughters-1; sampleNo>=0; sampleNo--)
  {
    samplePhysical = G4LogicalVolume_GetDaughter(motherLogical,sampleNo);
    if ( samplePhysical!=blockedExitedVol )
    {
      G4AffineTransform sampleTf =
       G4AffineTransform_create_full(
  G4VPhysicalVolume_GetObjectRotationValue(samplePhysical),
  G4VPhysicalVolume_GetTranslation(samplePhysical));
   G4AffineTransform_Invert(&sampleTf);
      const G4ThreeVector samplePoint =
   G4AffineTransform_TransformPoint(&sampleTf, localPoint);
      const G4VSolid *sampleSolid =
  G4LogicalVolume_GetSolid(
   G4VPhysicalVolume_GetLogicalVolume( samplePhysical ));
      const G4double sampleSafety =
  G4VSolid_DistanceToIn(sampleSolid,samplePoint);
      if ( sampleSafety<ourSafety )
      {
        ourSafety=sampleSafety;
      }
      if ( sampleSafety<=ourStep )
      {
        sampleDirection = G4AffineTransform_TransformAxis(&sampleTf, localDirection);
        const G4double sampleStep =
   G4VSolid_DistanceToIn_full(sampleSolid,samplePoint,sampleDirection);
        if ( sampleStep<=ourStep )
        {
          ourStep = sampleStep;
          *entering = true;
          *exiting = false;
          *pBlockedPhysical = samplePhysical;
        }
      }
    }
  }
  if ( currentProposedStepLength<ourSafety )
  {
    *entering = false;
    *exiting = false;
    *pBlockedPhysical = 0;
    ourStep = kInfinity;
  }
  else
  {
    if ( motherSafety<=ourStep )
    {
      G4double motherStep =
  G4VSolid_DistanceToOut_full(
   motherSolid,
   localPoint,
   localDirection,
   true,
   validExitNormal,
   exitNormal);
      if ( motherStep<=ourStep )
      {
        ourStep = motherStep;
        *exiting = true;
        *entering = false;
        if ( *validExitNormal )
        {
          G4RotationMatrix rot = G4VPhysicalVolume_GetObjectRotationValue(motherPhysical);
    G4RotationMatrix inv = G4RotationMatrix_inverse(&rot);
          *exitNormal = G4RotationMatrix_apply(&inv, *exitNormal);
        }
      }
      else
      {
        *validExitNormal = false;
      }
    }
  }
  *newSafety = ourSafety;
  return ourStep;
}
__device__ G4bool
G4AuxiliaryNavServices_CheckPointOnSurface(
         const G4VSolid* sampleSolid,
                     G4ThreeVector localPoint,
                     const G4ThreeVector* globalDirection,
                     G4AffineTransform sampleTransform,
                     const G4bool locatedOnEdge);
__device__ G4bool
G4AuxiliaryNavServices_CheckPointExiting(
       const G4VSolid* sampleSolid,
                   G4ThreeVector localPoint,
                   const G4ThreeVector* globalDirection,
                   G4AffineTransform sampleTransform );
__device__
 G4SmartVoxelNode*
G4VoxelNavigation_VoxelLocate(
   G4VoxelNavigation *This,
   G4SmartVoxelHeader* pHead,
   G4ThreeVector localPoint )
{
  G4SmartVoxelHeader *targetVoxelHeader=pHead;
  G4SmartVoxelNode *targetVoxelNode = 0;
  const G4SmartVoxelProxy *sampleProxy;
  EAxis targetHeaderAxis;
  G4double targetHeaderMin, targetHeaderNodeWidth;
  G4int targetHeaderNoSlices, targetNodeNo;
  This->fVoxelDepth = 0;
  while ( targetVoxelNode == 0 )
  {
    targetHeaderAxis = G4VoxelHeader_GetAxis(targetVoxelHeader);
    targetHeaderNoSlices = G4VoxelHeader_GetNoSlices(targetVoxelHeader);
    targetHeaderMin = G4VoxelHeader_GetMinExtent(targetVoxelHeader);
    targetHeaderNodeWidth =
  (G4VoxelHeader_GetMaxExtent(targetVoxelHeader)-targetHeaderMin)
                          / targetHeaderNoSlices;
    targetNodeNo = (G4int)(
  (G4ThreeVector_coord(localPoint,targetHeaderAxis)-targetHeaderMin)
                          / targetHeaderNodeWidth);
    if ( targetNodeNo<0 )
    {
  targetNodeNo = 0;
    }
    else if ( targetNodeNo>=targetHeaderNoSlices )
 {
  targetNodeNo = targetHeaderNoSlices-1;
 }
    This->fVoxelAxisStack[This->fVoxelDepth] = targetHeaderAxis;
    This->fVoxelNoSlicesStack[This->fVoxelDepth] = targetHeaderNoSlices;
    This->fVoxelSliceWidthStack[This->fVoxelDepth] = targetHeaderNodeWidth;
    This->fVoxelNodeNoStack[This->fVoxelDepth] = targetNodeNo;
    This->fVoxelHeaderStack[This->fVoxelDepth] = targetVoxelHeader;
    sampleProxy = G4VoxelHeader_GetSlice(targetVoxelHeader, targetNodeNo);
    if ( G4VoxelProxy_IsNode(sampleProxy) )
    {
      targetVoxelNode = G4VoxelProxy_GetNode(sampleProxy);
    }
    else
    {
      targetVoxelHeader = G4VoxelProxy_GetHeader(sampleProxy);
      This->fVoxelDepth++;
      (void)0;
    }
  }
  This->fVoxelNode = targetVoxelNode;
  return targetVoxelNode;
}
__device__
G4bool
G4VoxelNavigation_LocateNextVoxel(
   G4VoxelNavigation *This,
   G4ThreeVector localPoint,
   G4ThreeVector localDirection,
   const G4double currentStep )
{
  G4SmartVoxelHeader *workHeader=0, *newHeader=0;
  G4SmartVoxelProxy *newProxy=0;
  G4SmartVoxelNode *newVoxelNode= 0;
  G4ThreeVector targetPoint, voxelPoint;
  G4double workNodeWidth, workMinExtent, workCoord;
  G4double minVal, maxVal, newDistance=0.;
  G4double newHeaderMin, newHeaderNodeWidth;
  G4int depth=0, newDepth=0, workNodeNo=0, newNodeNo=0, newHeaderNoSlices=0;
  EAxis workHeaderAxis, newHeaderAxis;
  G4bool isNewVoxel=false;
  G4double currentDistance = currentStep;
  for (depth=0; depth<This->fVoxelDepth; depth++)
  {
    targetPoint =
  G4ThreeVector_saxpy(currentDistance,localDirection,localPoint);
    newDistance = currentDistance;
    workHeader = This->fVoxelHeaderStack[depth];
    workHeaderAxis = This->fVoxelAxisStack[depth];
    workNodeNo = This->fVoxelNodeNoStack[depth];
    workNodeWidth = This->fVoxelSliceWidthStack[depth];
    workMinExtent = G4VoxelHeader_GetMinExtent(workHeader);
    workCoord = G4ThreeVector_coord(targetPoint,workHeaderAxis);
    minVal = workMinExtent+workNodeNo*workNodeWidth;
    if ( minVal<=workCoord+1E-3*0.5 )
    {
      maxVal = minVal+workNodeWidth;
      if ( maxVal<=workCoord-1E-3*0.5 )
      {
        newNodeNo = workNodeNo+1;
        newHeader = workHeader;
        newDistance = (maxVal-G4ThreeVector_coord(localPoint,workHeaderAxis))
                    / G4ThreeVector_coord(localDirection,workHeaderAxis);
        isNewVoxel = true;
        newDepth = depth;
      }
    }
    else
    {
      newNodeNo = workNodeNo-1;
      newHeader = workHeader;
      newDistance = (minVal-G4ThreeVector_coord(localPoint,workHeaderAxis))
                  / G4ThreeVector_coord(localDirection,workHeaderAxis);
      isNewVoxel = true;
      newDepth = depth;
    }
    currentDistance = newDistance;
  }
  targetPoint =
 G4ThreeVector_saxpy(currentDistance,localDirection,localPoint);
  depth = This->fVoxelDepth;
  {
    workHeader = This->fVoxelHeaderStack[depth];
    workHeaderAxis = This->fVoxelAxisStack[depth];
    workNodeNo = This->fVoxelNodeNoStack[depth];
    workNodeWidth = This->fVoxelSliceWidthStack[depth];
    workMinExtent = G4VoxelHeader_GetMinExtent(workHeader);
    workCoord = G4ThreeVector_coord(targetPoint,workHeaderAxis);
    minVal = workMinExtent+G4VoxelNode_GetMinEquivalentSliceNo(This->fVoxelNode)*workNodeWidth;
    if ( minVal<=workCoord+1E-3*0.5 )
    {
      maxVal = workMinExtent+(G4VoxelNode_GetMaxEquivalentSliceNo(This->fVoxelNode)+1)
                            *workNodeWidth;
      if ( maxVal<=workCoord-1E-3*0.5 )
      {
        newNodeNo = G4VoxelNode_GetMaxEquivalentSliceNo(This->fVoxelNode)+1;
        newHeader = workHeader;
        newDistance = (maxVal-G4ThreeVector_coord(localPoint,workHeaderAxis))
                    / G4ThreeVector_coord(localDirection,workHeaderAxis);
        isNewVoxel = true;
        newDepth = depth;
      }
    }
    else
    {
      newNodeNo = G4VoxelNode_GetMinEquivalentSliceNo(This->fVoxelNode)-1;
      newHeader = workHeader;
      newDistance = (minVal-G4ThreeVector_coord(localPoint,workHeaderAxis))
                  / G4ThreeVector_coord(localDirection,workHeaderAxis);
      isNewVoxel = true;
      newDepth = depth;
    }
    currentDistance = newDistance;
  }
  if (isNewVoxel)
  {
    if ( (newNodeNo<0) || (newNodeNo>=G4VoxelHeader_GetNoSlices(newHeader)))
    {
      isNewVoxel = false;
    }
    else
    {
      voxelPoint = G4ThreeVector_saxpy(newDistance,localDirection,localPoint);
      (void)0;
      This->fVoxelNodeNoStack[newDepth] = newNodeNo;
      This->fVoxelDepth = newDepth;
      newVoxelNode = 0;
      while ( newVoxelNode == 0 )
      {
        newProxy = G4VoxelHeader_GetSlice(newHeader,newNodeNo);
        if ( G4VoxelProxy_IsNode(newProxy) )
        {
          newVoxelNode = G4VoxelProxy_GetNode(newProxy);
        }
        else
        {
          This->fVoxelDepth++;
          (void)0;
          newHeader = G4VoxelProxy_GetHeader(newProxy);
          newHeaderAxis = G4VoxelHeader_GetAxis(newHeader);
          newHeaderNoSlices = G4VoxelHeader_GetNoSlices(newHeader);
          newHeaderMin = G4VoxelHeader_GetMinExtent(newHeader);
          newHeaderNodeWidth =
   (G4VoxelHeader_GetMaxExtent(newHeader)-newHeaderMin)
                             / newHeaderNoSlices;
          newNodeNo = (G4int)(
   (G4ThreeVector_coord(voxelPoint,newHeaderAxis)-newHeaderMin)
                             / newHeaderNodeWidth );
          if ( newNodeNo<0 )
          {
            newNodeNo=0;
          }
          else if ( newNodeNo>=newHeaderNoSlices )
               {
                 newNodeNo = newHeaderNoSlices-1;
               }
          This->fVoxelAxisStack[This->fVoxelDepth] = newHeaderAxis;
          This->fVoxelNoSlicesStack[This->fVoxelDepth] = newHeaderNoSlices;
          This->fVoxelSliceWidthStack[This->fVoxelDepth] = newHeaderNodeWidth;
          This->fVoxelNodeNoStack[This->fVoxelDepth] = newNodeNo;
          This->fVoxelHeaderStack[This->fVoxelDepth] = newHeader;
        }
      }
      This->fVoxelNode = newVoxelNode;
    }
  }
  return isNewVoxel;
}
__device__
G4double
G4VoxelNavigation_ComputeVoxelSafety(
   const G4VoxelNavigation *This,
   G4ThreeVector localPoint)
{
  G4SmartVoxelHeader *curHeader;
  G4double voxelSafety, curNodeWidth;
  G4double curNodeOffset, minCurCommonDelta, maxCurCommonDelta;
  G4int minCurNodeNoDelta, maxCurNodeNoDelta;
  G4int localVoxelDepth, curNodeNo;
  EAxis curHeaderAxis;
  localVoxelDepth = This->fVoxelDepth;
  curHeader = This->fVoxelHeaderStack[localVoxelDepth];
  curHeaderAxis = This->fVoxelAxisStack[localVoxelDepth];
  curNodeNo = This->fVoxelNodeNoStack[localVoxelDepth];
  curNodeWidth = This->fVoxelSliceWidthStack[localVoxelDepth];
  curNodeOffset = curNodeNo*curNodeWidth;
  maxCurNodeNoDelta = G4VoxelNode_GetMaxEquivalentSliceNo(This->fVoxelNode)-curNodeNo;
  minCurNodeNoDelta = curNodeNo-G4VoxelNode_GetMaxEquivalentSliceNo(This->fVoxelNode);
  minCurCommonDelta = G4ThreeVector_coord(localPoint,curHeaderAxis)
 - G4VoxelHeader_GetMinExtent(curHeader) - curNodeOffset;
  maxCurCommonDelta = curNodeWidth-minCurCommonDelta;
  if ( minCurNodeNoDelta<maxCurNodeNoDelta )
  {
    voxelSafety = minCurNodeNoDelta*curNodeWidth;
    voxelSafety += minCurCommonDelta;
  }
  else if (maxCurNodeNoDelta < minCurNodeNoDelta)
       {
         voxelSafety = maxCurNodeNoDelta*curNodeWidth;
         voxelSafety += maxCurCommonDelta;
        }
        else
        {
          voxelSafety = minCurNodeNoDelta*curNodeWidth;
          voxelSafety += (((minCurCommonDelta)<(maxCurCommonDelta))?(minCurCommonDelta):(maxCurCommonDelta));
        }
  while ( (localVoxelDepth>0) && (voxelSafety>0) )
  {
    localVoxelDepth--;
    curHeader = This->fVoxelHeaderStack[localVoxelDepth];
    curHeaderAxis = This->fVoxelAxisStack[localVoxelDepth];
    curNodeNo = This->fVoxelNodeNoStack[localVoxelDepth];
    curNodeWidth = This->fVoxelSliceWidthStack[localVoxelDepth];
    curNodeOffset = curNodeNo*curNodeWidth;
    minCurCommonDelta = G4ThreeVector_coord(localPoint,curHeaderAxis)
                        - G4VoxelHeader_GetMinExtent(curHeader) - curNodeOffset;
    maxCurCommonDelta = curNodeWidth-minCurCommonDelta;
    if ( minCurCommonDelta<voxelSafety )
    {
      voxelSafety = minCurCommonDelta;
    }
    if ( maxCurCommonDelta<voxelSafety )
    {
      voxelSafety = maxCurCommonDelta;
    }
  }
  if ( voxelSafety<0 )
  {
    voxelSafety = 0;
  }
  return voxelSafety;
}
__device__
void G4VoxelNavigation_ctor( G4VoxelNavigation *This )
{
 This->fVoxelDepth = -1;
 This->fVoxelNode = 0;
}
__device__
G4bool
G4VoxelNavigation_LevelLocate(
   G4VoxelNavigation *This,
   G4NavigationHistory* history,
   const G4VPhysicalVolume* blockedVol,
   G4ThreeVector globalPoint,
   const G4ThreeVector* globalDirection,
   const G4bool pLocatedOnEdge,
   G4ThreeVector *localPoint )
{
  G4SmartVoxelHeader *targetVoxelHeader;
  G4SmartVoxelNode *targetVoxelNode;
  G4VPhysicalVolume *targetPhysical, *samplePhysical;
  G4LogicalVolume *targetLogical;
  G4VSolid *sampleSolid;
  G4ThreeVector samplePoint;
  G4int targetNoDaughters;
  targetPhysical = G4NavigationHistory_GetTopVolume(history);
  targetLogical = G4VPhysicalVolume_GetLogicalVolume(targetPhysical);
  targetVoxelHeader = G4LogicalVolume_GetVoxelHeader(targetLogical);
  targetVoxelNode =
 G4VoxelNavigation_VoxelLocate(This,targetVoxelHeader,*localPoint);
  targetNoDaughters=G4VoxelNode_GetNoContained(targetVoxelNode);
  if ( targetNoDaughters==0 ) return false;
  for ( int sampleNo=targetNoDaughters-1; sampleNo>=0; sampleNo-- )
  {
    samplePhysical =
  G4LogicalVolume_GetDaughter( targetLogical,
   G4VoxelNode_GetVolume(targetVoxelNode,sampleNo));
    if ( samplePhysical!=blockedVol )
    {
      G4NavigationHistory_NewLevel(history, samplePhysical, kNormal);
      sampleSolid =
  G4LogicalVolume_GetSolid(
   G4VPhysicalVolume_GetLogicalVolume( samplePhysical ));
   G4AffineTransform tf = G4NavigationHistory_GetTopTransform( history );
      samplePoint =
  G4AffineTransform_TransformPoint( &tf, globalPoint );
      if( G4AuxiliaryNavServices_CheckPointOnSurface(
   sampleSolid, samplePoint, globalDirection,
   tf, pLocatedOnEdge) )
      {
        *localPoint = samplePoint;
        return true;
      }
      else
      {
    G4NavigationHistory_BackLevel( history );
      }
    }
  }
  return false;
}
__device__
G4double
G4VoxelNavigation_ComputeStep(
   G4VoxelNavigation *This,
   G4ThreeVector localPoint,
   G4ThreeVector localDirection,
   const G4double currentProposedStepLength,
   G4double *newSafety,
   G4NavigationHistory *history,
   G4bool *validExitNormal,
   G4ThreeVector *exitNormal,
   G4bool *exiting,
   G4bool *entering,
   G4VPhysicalVolume *(*pBlockedPhysical)
    , G4double * Result
   )
{
  G4VPhysicalVolume *motherPhysical, *samplePhysical,
 *blockedExitedVol = 0;
  G4LogicalVolume *motherLogical;
  G4VSolid *motherSolid;
  G4ThreeVector sampleDirection;
  G4double ourStep=currentProposedStepLength, motherSafety, ourSafety;
  G4int sampleNo;
  G4bool initialNode, noStep;
  const G4SmartVoxelNode *curVoxelNode;
  G4int curNoVolumes, contentNo;
  G4double voxelSafety;
  motherPhysical = G4NavigationHistory_GetTopVolume( history );
  motherLogical = G4VPhysicalVolume_GetLogicalVolume(motherPhysical);
  motherSolid = G4LogicalVolume_GetSolid(motherLogical);
  motherSafety = G4VSolid_DistanceToOut(motherSolid, localPoint);
  ourSafety = motherSafety;
  if ( *exiting && *validExitNormal )
  {
    if ( G4ThreeVector_dot(localDirection,*exitNormal)>=kMinExitingNormalCosine )
    {
      blockedExitedVol = *pBlockedPhysical;
      ourSafety = 0;
    }
  }
  *exiting = false;
  *entering = false;
  initialNode = true;
  noStep = true;
  while ( noStep )
  {
    curVoxelNode = This->fVoxelNode;
    curNoVolumes = G4VoxelNode_GetNoContained(curVoxelNode);
    for (contentNo=curNoVolumes-1; contentNo>=0; contentNo--)
    {
      sampleNo = G4VoxelNode_GetVolume( curVoxelNode, contentNo);
        samplePhysical = G4LogicalVolume_GetDaughter(motherLogical,sampleNo);
        if ( samplePhysical!=blockedExitedVol )
        {
    G4AffineTransform sampleTf =
   G4AffineTransform_create_full(
    G4VPhysicalVolume_GetObjectRotationValue(samplePhysical),
    G4VPhysicalVolume_GetTranslation(samplePhysical));
          G4AffineTransform_Invert(&sampleTf);
          const G4ThreeVector samplePoint =
    G4AffineTransform_TransformPoint(&sampleTf,localPoint);
          const G4VSolid *sampleSolid =
    G4LogicalVolume_GetSolid(
     G4VPhysicalVolume_GetLogicalVolume(
      samplePhysical ));
          const G4double sampleSafety =
   G4VSolid_DistanceToIn(sampleSolid,samplePoint);
          if ( sampleSafety<ourSafety )
          {
            ourSafety = sampleSafety;
          }
          if ( sampleSafety<=ourStep )
          {
            sampleDirection =
    G4AffineTransform_TransformAxis( &sampleTf, localDirection );
            G4double sampleStep =
    G4VSolid_DistanceToIn_full(sampleSolid, samplePoint, sampleDirection);
            if ( sampleStep<=ourStep )
            {
              ourStep = sampleStep;
              *entering = true;
              *exiting = false;
              *pBlockedPhysical = samplePhysical;
            }
        }
      }
    }
    if (initialNode)
    {
      initialNode = false;
      voxelSafety = G4VoxelNavigation_ComputeVoxelSafety(This,localPoint);
      if ( voxelSafety<ourSafety )
      {
        ourSafety = voxelSafety;
      }
      if ( currentProposedStepLength<ourSafety )
      {
        noStep = false;
        *entering = false;
        *exiting = false;
        *pBlockedPhysical = 0;
        ourStep = kInfinity;
      }
      else
      {
        if ( motherSafety<=ourStep )
        {
          G4double motherStep =
   G4VSolid_DistanceToOut_full( motherSolid, localPoint, localDirection,
                                         true, validExitNormal, exitNormal);
          if ( motherStep<=ourStep )
          {
            ourStep = motherStep;
            *exiting = true;
            *entering = false;
            if ( *validExitNormal )
            {
    G4RotationMatrix rot = G4VPhysicalVolume_GetObjectRotationValue(motherPhysical);
    G4RotationMatrix inv = G4RotationMatrix_inverse(&rot);
    *exitNormal = G4RotationMatrix_apply( &inv, *exitNormal );
            }
          }
          else
          {
            *validExitNormal = false;
          }
        }
      }
      *newSafety = ourSafety;
    }
    if (noStep)
    {
      noStep = G4VoxelNavigation_LocateNextVoxel(This, localPoint, localDirection, ourStep);
    }
  }
  int locationId = (blockIdx.x * blockDim.x + threadIdx.x);
  return ourStep;
}
__global__ void trace(
 Particle *input,
 G4double *output,
 G4VPhysicalVolume *worldVolumeAndGeomBuffer,
 G4double phys_step,
 int totalSize
 , G4double * Result
 , SolidInfo * Solids
 , ResultInfo * Result_For_Current_Solid
 , FinalResult * Compacter_Result,
 G4SmartVoxelNode * nullVNode
 )
{
 const unsigned globalIdx = (blockIdx.x * blockDim.x + threadIdx.x);
 const unsigned localIdx = threadIdx.x;
 const unsigned locationId = globalIdx;
 if (globalIdx >= totalSize ) return;
 __shared__ int Numbers_Of_Solid[ BlockSize ];
 __shared__ int Sum_Of_Solids[ BlockSize ];
 __shared__ bool noStepArray [ BlockSize ];
 __shared__ PointInformation LocationArray[ BlockSize ];
 __shared__ G4VPhysicalVolume * info[ BlockSize ];
 G4VoxelNode_ctor( nullVNode ,1 );
 G4bool Cur_Vol_Store [ BlockSize ];
 G4Navigator navi;
 G4Navigator *nav = &navi;
 G4Navigator_ctor(nav);
 G4Navigator_SetWorldVolume( nav, worldVolumeAndGeomBuffer );
 Particle p = input[globalIdx];
 if( globalIdx == 0)
 {
 }
    const G4VPhysicalVolume * cur_vol =
  G4Navigator_LocateGlobalPointAndSetup(
   nav, p.pos, 0, false, true, Result );
 G4bool cur_vol_local = true, cur_vol_all = true;
 G4double step, safety = 0.1;
 G4double integratedDensity = 0;
 int temp = 0;
 while ( cur_vol_all )
 {
  {
  const G4double curDensity =
    G4LogicalVolume_GetMaterial( G4VPhysicalVolume_GetLogicalVolume( cur_vol ))->property;
  PointInformation NewPoint = { p.pos, p.dir };
  LocationArray[ locationId ] = NewPoint;
  if( temp == 1)
  {
   Result[ locationId ] = step;
  }
  step = G4Navigator_ComputeStep( nav, p.pos, p.dir, phys_step, &safety
       , cur_vol_local
       , Result
       );
  if ( step == kInfinity ) step = phys_step;
  const G4double nextStepIntegratedD = curDensity * step;
  int locationId = (blockIdx.x * blockDim.x + threadIdx.x);
  integratedDensity += nextStepIntegratedD;
  G4ThreeVector_sum_assign( &(p.pos), G4ThreeVector_mult( p.dir, step ) );
  G4Navigator_SetGeometricallyLimitedStep( nav );
  if( globalIdx == 0 ){
  }
  cur_vol =
   G4Navigator_LocateGlobalPointAndSetup(
    nav, p.pos, &(p.dir), true, false, Result );
  if ( !cur_vol )
   cur_vol_local = false;
  }
  Cur_Vol_Store[ locationId ] = cur_vol_local;
  __syncthreads();
  cur_vol_all = NoStepReduction( Cur_Vol_Store, BlockSize );
  __syncthreads();
  temp++;
}
 output[globalIdx] = integratedDensity;
}
__global__ void relocate ( int * ptr, void * buf, int size )
{
  typedef unsigned char byte;
  const unsigned globalidx = (blockIdx.x * blockDim.x + threadIdx.x);
  if(globalidx>=size) return;
  int destoffs, targoffs;
  destoffs = *(ptr + 2*globalidx);
  targoffs = *(ptr + 2*globalidx+ 1);
  *((byte*)buf+destoffs) = (byte)((byte*)buf + targoffs);
}
__global__ void check( G4VPhysicalVolume *worldVolumeAndGeomBuffer, unsigned long * result)
{
 unsigned int hope = ( unsigned int )worldVolumeAndGeomBuffer;
 *result = hope;
}
__global__ void test ( bool * output
 , bool * input
 )
{
 int tid = (blockIdx.x * blockDim.x + threadIdx.x);
 int offset = 1;
 G4bool result;
 if( tid == 0)
 {
 input[ 0] = true;
 input[ 1] = true;
 input[ 2] = true;
 input[ 3] = true;
 input[ 4] = true;
 input[ 5] = true;
    input[ 6] = false;
    input[ 7] = true;
 }
 __syncthreads();
}
__global__ void checkgeom( G4VPhysicalVolume *worldVolumeAndGeomBuffer, int * result, int number_of_increments)
{
 const unsigned globalid = (blockIdx.x * blockDim.x + threadIdx.x);
 if(globalid>=1) return;
 int i=0;
 G4Navigator navi;
 G4Navigator *nav = &navi;
 G4Navigator_ctor(nav);
 G4Navigator_SetWorldVolume( nav, worldVolumeAndGeomBuffer );
 G4ThreeVector pos = G4ThreeVector_create( 0.0, 0.0, 0.0);
 const G4VPhysicalVolume * cur_vol;
 unsigned int geom_start = ( unsigned int )worldVolumeAndGeomBuffer;
 pos = G4ThreeVector_create( 0.7, 1.0, 0.7);
 float x_increment = 0.2, y_increment = 0.2, z_increment = 0.2;
 for( i=0; i < number_of_increments*3 ; i+=3)
 {
  result[i] = ( int ) cur_vol->count;
  result[i + 1] = (( unsigned int )(cur_vol->flogical) - geom_start);
  result[i + 2] = ( int ) G4LogicalVolume_GetMaterial( G4VPhysicalVolume_GetLogicalVolume( cur_vol ))->property;
  pos.x+=x_increment;
  pos.y+=y_increment;
  pos.z+=z_increment;
 }
}
struct CameraParameters
{
 double
  heading,
  pitch,
  roll,
  dist,
  yfov,
  target_x,
  target_y,
  target_z;
 CameraParameters()
 :
  heading(0), pitch(0), roll(0), dist(1),
  yfov(90), target_x(0), target_y(0), target_z(0)
 {}
};
struct EventOrigin
{
 double x,y,z;
};
class Geometry
{
public:
 typedef unsigned char byte;
 virtual ~Geometry() {}
 virtual void create() = 0;
 virtual void relocate( void *newbegin ) = 0;
 virtual int size() const = 0;
 virtual int ptrs_size() const=0;
 virtual void *getBuffer() = 0;
 virtual double getScale() const = 0;
 virtual CameraParameters getCamera() const = 0;
 virtual EventOrigin getEvent() const
 {
  EventOrigin e = { 0,0,0 };
  return e;
 }
 virtual int getNumVoxelNodes() const { return 0; }
};
typedef struct { const char *err, *fn; int line, errcode; } my_cuda_err;
typedef struct { int secs; int usecs; } mytimet;
extern "C"
{
 void myprint( const char *chr );
 void myprint1( const char *chr, int n );
 mytimet mytimer();
 void myprinttdiff(mytimet a, mytimet b);
 void mysleep(int n);
}
static inline int ceilDiv( int a, int d )
{
 return a/d + ((a%d)?1:0);
}
Particle *gpuInput;
G4double *gpuOutput;
Geometry::byte *gpuGeom;
int numInput, numOutput, numInputPerRound;
const int WARP_SIZE = 32;
void createGrid( int numInput, dim3* grid, dim3* block )
{
 const int MAXSIZE = 10000000;
 const int NUMCORES = 448;
 const int NUMMULTIPROC = 14;
 const int BLOCKS_PER_MULTIPROC = 8;
 const int MAX_WARPS_PER_MULTIPROC = 48;
 const int MAX_DATA_PER_MULTIPROC = MAX_WARPS_PER_MULTIPROC*WARP_SIZE;
 int size = numInput;
 if (size > MAXSIZE) size = MAXSIZE;
 int dataPerMultiproc = ceilDiv(size,NUMMULTIPROC);
 if ( dataPerMultiproc > MAX_DATA_PER_MULTIPROC )
  dataPerMultiproc = MAX_DATA_PER_MULTIPROC;
 int blockSize = ceilDiv(dataPerMultiproc,BLOCKS_PER_MULTIPROC);
 const int MAX_BLOCK_SIZE = 1024;
 if (blockSize > MAX_BLOCK_SIZE) blockSize = MAX_BLOCK_SIZE;
 int numBlocks = ceilDiv(size,blockSize);
 int numWarps = ceilDiv(blockSize,WARP_SIZE) * numBlocks;
 if (numWarps > NUMCORES)
 {
  blockSize = ceilDiv(blockSize,WARP_SIZE)*WARP_SIZE;
  dataPerMultiproc = blockSize * BLOCKS_PER_MULTIPROC;
  if ( dataPerMultiproc > MAX_DATA_PER_MULTIPROC )
   blockSize -= WARP_SIZE;
 }
 size = blockSize*ceilDiv(size,blockSize);
 if (size > MAXSIZE) size = MAXSIZE;
 block->x = blockSize;
 block->y = block->z = 1;
 grid->x = size/blockSize;
 grid->y = 1;
 grid->z = 1;
}
my_cuda_err cudainit( Geometry *geom, int N )
{
 const mytimet t0 = mytimer();
 numOutput = numInput = numInputPerRound = N;
 do { hipError_t errc = hipSetDeviceFlags(0); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 157, errc }; return r; } } while(0);
 do { hipError_t errc = hipMalloc( (void**)&gpuInput, sizeof(Particle)*numInput ); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 159, errc }; return r; } } while(0);
 do { hipError_t errc = hipMalloc( (void**)&gpuOutput, sizeof(G4double)*numOutput ); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 160, errc }; return r; } } while(0);
 do { hipError_t errc = hipMalloc( (void**)&gpuGeom, geom->size() ); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 161, errc }; return r; } } while(0);
 geom->relocate( gpuGeom );
 hipFuncSetCacheConfig(reinterpret_cast<const void*>(trace), hipFuncCachePreferL1);
 do { hipError_t errc = hipMemcpy( gpuGeom, geom->getBuffer(), geom->size(), hipMemcpyHostToDevice ); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 197, errc }; return r; } } while(0);
 const mytimet t1 = mytimer();
 myprint("Initialization: ");
 myprinttdiff(t0, t1);
 my_cuda_err ok = { 0, 0, 0, hipSuccess }; return ok;
}
my_cuda_err cudaexec( G4double phys_step, int totalInput, Particle *input, G4double *output )
{
   for ( int i = 0; i < totalInput; i += numInput )
   {
 if ( i + numInput > totalInput ) numInput = totalInput-i;
 do { hipError_t errc = hipMemcpy( gpuInput, input+i, sizeof(Particle)*numInput, hipMemcpyHostToDevice ); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 333, errc }; return r; } } while(0);
 dim3 grid, block;
 createGrid( numInput, &grid, &block );
 trace <<< grid, block >>>( gpuInput, gpuOutput, (G4VPhysicalVolume*)gpuGeom, phys_step, numInput, 0, 0, 0, 0, 0 );
 do { hipError_t errc = hipGetLastError(); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 340, errc }; return r; } } while(0);
 do { hipError_t errc = hipMemcpy( output+i, gpuOutput, sizeof(G4double)*numOutput, hipMemcpyDeviceToHost ); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 343, errc }; return r; } } while(0);
   }
   my_cuda_err ok = { 0, 0, 0, hipSuccess }; return ok;
}
my_cuda_err cudafinish()
{
 const mytimet t0 = mytimer();
 do { hipError_t errc = hipFree( gpuInput ); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 356, errc }; return r; } } while(0);
 do { hipError_t errc = hipFree( gpuOutput ); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 357, errc }; return r; } } while(0);
 do { hipError_t errc = hipFree( gpuGeom ); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 358, errc }; return r; } } while(0);
 do { hipError_t errc = hipDeviceReset(); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 375, errc }; return r; } } while(0);
 const mytimet t1 = mytimer();
 myprint("Finalization: ");
 myprinttdiff(t0, t1);
 my_cuda_err ok = { 0, 0, 0, hipSuccess }; return ok;
}
