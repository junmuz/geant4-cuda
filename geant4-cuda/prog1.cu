#include "hip/hip_runtime.h"
extern "C" {
extern void __assert_fail (__const char *__assertion, __const char *__file,
      unsigned int __line, __const char *__function)
     throw () __attribute__ ((__noreturn__));
extern void __assert_perror_fail (int __errnum, __const char *__file,
      unsigned int __line,
      __const char *__function)
     throw () __attribute__ ((__noreturn__));
extern void __assert (const char *__assertion, const char *__file, int __line)
     throw () __attribute__ ((__noreturn__));
}
extern "C" {
typedef long double float_t;
typedef long double double_t;

extern double acos (double __x) throw (); extern double __acos (double __x) throw ();
extern double asin (double __x) throw (); extern double __asin (double __x) throw ();
extern double atan (double __x) throw (); extern double __atan (double __x) throw ();
extern double atan2 (double __y, double __x) throw (); extern double __atan2 (double __y, double __x) throw ();
extern double cos (double __x) throw (); extern double __cos (double __x) throw ();
extern double sin (double __x) throw (); extern double __sin (double __x) throw ();
extern double tan (double __x) throw (); extern double __tan (double __x) throw ();
extern double cosh (double __x) throw (); extern double __cosh (double __x) throw ();
extern double sinh (double __x) throw (); extern double __sinh (double __x) throw ();
extern double tanh (double __x) throw (); extern double __tanh (double __x) throw ();

extern void sincos (double __x, double *__sinx, double *__cosx) throw (); extern void __sincos (double __x, double *__sinx, double *__cosx) throw ();

extern double acosh (double __x) throw (); extern double __acosh (double __x) throw ();
extern double asinh (double __x) throw (); extern double __asinh (double __x) throw ();
extern double atanh (double __x) throw (); extern double __atanh (double __x) throw ();


extern double exp (double __x) throw (); extern double __exp (double __x) throw ();
extern double frexp (double __x, int *__exponent) throw (); extern double __frexp (double __x, int *__exponent) throw ();
extern double ldexp (double __x, int __exponent) throw (); extern double __ldexp (double __x, int __exponent) throw ();
extern double log (double __x) throw (); extern double __log (double __x) throw ();
extern double log10 (double __x) throw (); extern double __log10 (double __x) throw ();
extern double modf (double __x, double *__iptr) throw (); extern double __modf (double __x, double *__iptr) throw ();

extern double exp10 (double __x) throw (); extern double __exp10 (double __x) throw ();
extern double pow10 (double __x) throw (); extern double __pow10 (double __x) throw ();

extern double expm1 (double __x) throw (); extern double __expm1 (double __x) throw ();
extern double log1p (double __x) throw (); extern double __log1p (double __x) throw ();
extern double logb (double __x) throw (); extern double __logb (double __x) throw ();


extern double exp2 (double __x) throw (); extern double __exp2 (double __x) throw ();
extern double log2 (double __x) throw (); extern double __log2 (double __x) throw ();


extern double pow (double __x, double __y) throw (); extern double __pow (double __x, double __y) throw ();
extern double sqrt (double __x) throw (); extern double __sqrt (double __x) throw ();


extern double hypot (double __x, double __y) throw (); extern double __hypot (double __x, double __y) throw ();


extern double cbrt (double __x) throw (); extern double __cbrt (double __x) throw ();


extern double ceil (double __x) throw () __attribute__ ((__const__)); extern double __ceil (double __x) throw () __attribute__ ((__const__));
extern double fabs (double __x) throw () __attribute__ ((__const__)); extern double __fabs (double __x) throw () __attribute__ ((__const__));
extern double floor (double __x) throw () __attribute__ ((__const__)); extern double __floor (double __x) throw () __attribute__ ((__const__));
extern double fmod (double __x, double __y) throw (); extern double __fmod (double __x, double __y) throw ();
extern int __isinf (double __value) throw () __attribute__ ((__const__));
extern int __finite (double __value) throw () __attribute__ ((__const__));

extern int isinf (double __value) throw () __attribute__ ((__const__));
extern int finite (double __value) throw () __attribute__ ((__const__));
extern double drem (double __x, double __y) throw (); extern double __drem (double __x, double __y) throw ();
extern double significand (double __x) throw (); extern double __significand (double __x) throw ();

extern double copysign (double __x, double __y) throw () __attribute__ ((__const__)); extern double __copysign (double __x, double __y) throw () __attribute__ ((__const__));


extern double nan (__const char *__tagb) throw () __attribute__ ((__const__)); extern double __nan (__const char *__tagb) throw () __attribute__ ((__const__));

extern int __isnan (double __value) throw () __attribute__ ((__const__));
extern int isnan (double __value) throw () __attribute__ ((__const__));
extern double j0 (double) throw (); extern double __j0 (double) throw ();
extern double j1 (double) throw (); extern double __j1 (double) throw ();
extern double jn (int, double) throw (); extern double __jn (int, double) throw ();
extern double y0 (double) throw (); extern double __y0 (double) throw ();
extern double y1 (double) throw (); extern double __y1 (double) throw ();
extern double yn (int, double) throw (); extern double __yn (int, double) throw ();

extern double erf (double) throw (); extern double __erf (double) throw ();
extern double erfc (double) throw (); extern double __erfc (double) throw ();
extern double lgamma (double) throw (); extern double __lgamma (double) throw ();


extern double tgamma (double) throw (); extern double __tgamma (double) throw ();

extern double gamma (double) throw (); extern double __gamma (double) throw ();
extern double lgamma_r (double, int *__signgamp) throw (); extern double __lgamma_r (double, int *__signgamp) throw ();

extern double rint (double __x) throw (); extern double __rint (double __x) throw ();
extern double nextafter (double __x, double __y) throw () __attribute__ ((__const__)); extern double __nextafter (double __x, double __y) throw () __attribute__ ((__const__));
extern double nexttoward (double __x, long double __y) throw () __attribute__ ((__const__)); extern double __nexttoward (double __x, long double __y) throw () __attribute__ ((__const__));
extern double remainder (double __x, double __y) throw (); extern double __remainder (double __x, double __y) throw ();
extern double scalbn (double __x, int __n) throw (); extern double __scalbn (double __x, int __n) throw ();
extern int ilogb (double __x) throw (); extern int __ilogb (double __x) throw ();
extern double scalbln (double __x, long int __n) throw (); extern double __scalbln (double __x, long int __n) throw ();
extern double nearbyint (double __x) throw (); extern double __nearbyint (double __x) throw ();
extern double round (double __x) throw () __attribute__ ((__const__)); extern double __round (double __x) throw () __attribute__ ((__const__));
extern double trunc (double __x) throw () __attribute__ ((__const__)); extern double __trunc (double __x) throw () __attribute__ ((__const__));
extern double remquo (double __x, double __y, int *__quo) throw (); extern double __remquo (double __x, double __y, int *__quo) throw ();
extern long int lrint (double __x) throw (); extern long int __lrint (double __x) throw ();
extern long long int llrint (double __x) throw (); extern long long int __llrint (double __x) throw ();
extern long int lround (double __x) throw (); extern long int __lround (double __x) throw ();
extern long long int llround (double __x) throw (); extern long long int __llround (double __x) throw ();
extern double fdim (double __x, double __y) throw (); extern double __fdim (double __x, double __y) throw ();
extern double fmax (double __x, double __y) throw (); extern double __fmax (double __x, double __y) throw ();
extern double fmin (double __x, double __y) throw (); extern double __fmin (double __x, double __y) throw ();
extern int __fpclassify (double __value) throw ()
     __attribute__ ((__const__));
extern int __signbit (double __value) throw ()
     __attribute__ ((__const__));
extern double fma (double __x, double __y, double __z) throw (); extern double __fma (double __x, double __y, double __z) throw ();

extern double scalb (double __x, double __n) throw (); extern double __scalb (double __x, double __n) throw ();

extern float acosf (float __x) throw (); extern float __acosf (float __x) throw ();
extern float asinf (float __x) throw (); extern float __asinf (float __x) throw ();
extern float atanf (float __x) throw (); extern float __atanf (float __x) throw ();
extern float atan2f (float __y, float __x) throw (); extern float __atan2f (float __y, float __x) throw ();
extern float cosf (float __x) throw (); extern float __cosf (float __x) throw ();
extern float sinf (float __x) throw (); extern float __sinf (float __x) throw ();
extern float tanf (float __x) throw (); extern float __tanf (float __x) throw ();
extern float coshf (float __x) throw (); extern float __coshf (float __x) throw ();
extern float sinhf (float __x) throw (); extern float __sinhf (float __x) throw ();
extern float tanhf (float __x) throw (); extern float __tanhf (float __x) throw ();

extern void sincosf (float __x, float *__sinx, float *__cosx) throw (); extern void __sincosf (float __x, float *__sinx, float *__cosx) throw ();

extern float acoshf (float __x) throw (); extern float __acoshf (float __x) throw ();
extern float asinhf (float __x) throw (); extern float __asinhf (float __x) throw ();
extern float atanhf (float __x) throw (); extern float __atanhf (float __x) throw ();


extern float expf (float __x) throw (); extern float __expf (float __x) throw ();
extern float frexpf (float __x, int *__exponent) throw (); extern float __frexpf (float __x, int *__exponent) throw ();
extern float ldexpf (float __x, int __exponent) throw (); extern float __ldexpf (float __x, int __exponent) throw ();
extern float logf (float __x) throw (); extern float __logf (float __x) throw ();
extern float log10f (float __x) throw (); extern float __log10f (float __x) throw ();
extern float modff (float __x, float *__iptr) throw (); extern float __modff (float __x, float *__iptr) throw ();

extern float exp10f (float __x) throw (); extern float __exp10f (float __x) throw ();
extern float pow10f (float __x) throw (); extern float __pow10f (float __x) throw ();

extern float expm1f (float __x) throw (); extern float __expm1f (float __x) throw ();
extern float log1pf (float __x) throw (); extern float __log1pf (float __x) throw ();
extern float logbf (float __x) throw (); extern float __logbf (float __x) throw ();


extern float exp2f (float __x) throw (); extern float __exp2f (float __x) throw ();
extern float log2f (float __x) throw (); extern float __log2f (float __x) throw ();


extern float powf (float __x, float __y) throw (); extern float __powf (float __x, float __y) throw ();
extern float sqrtf (float __x) throw (); extern float __sqrtf (float __x) throw ();


extern float hypotf (float __x, float __y) throw (); extern float __hypotf (float __x, float __y) throw ();



extern float cbrtf (float __x) throw (); extern float __cbrtf (float __x) throw ();








extern float ceilf (float __x) throw () __attribute__ ((__const__)); extern float __ceilf (float __x) throw () __attribute__ ((__const__));


extern float fabsf (float __x) throw () __attribute__ ((__const__)); extern float __fabsf (float __x) throw () __attribute__ ((__const__));


extern float floorf (float __x) throw () __attribute__ ((__const__)); extern float __floorf (float __x) throw () __attribute__ ((__const__));


extern float fmodf (float __x, float __y) throw (); extern float __fmodf (float __x, float __y) throw ();




extern int __isinff (float __value) throw () __attribute__ ((__const__));


extern int __finitef (float __value) throw () __attribute__ ((__const__));





extern int isinff (float __value) throw () __attribute__ ((__const__));


extern int finitef (float __value) throw () __attribute__ ((__const__));


extern float dremf (float __x, float __y) throw (); extern float __dremf (float __x, float __y) throw ();



extern float significandf (float __x) throw (); extern float __significandf (float __x) throw ();





extern float copysignf (float __x, float __y) throw () __attribute__ ((__const__)); extern float __copysignf (float __x, float __y) throw () __attribute__ ((__const__));






extern float nanf (__const char *__tagb) throw () __attribute__ ((__const__)); extern float __nanf (__const char *__tagb) throw () __attribute__ ((__const__));





extern int __isnanf (float __value) throw () __attribute__ ((__const__));



extern int isnanf (float __value) throw () __attribute__ ((__const__));


extern float j0f (float) throw (); extern float __j0f (float) throw ();
extern float j1f (float) throw (); extern float __j1f (float) throw ();
extern float jnf (int, float) throw (); extern float __jnf (int, float) throw ();
extern float y0f (float) throw (); extern float __y0f (float) throw ();
extern float y1f (float) throw (); extern float __y1f (float) throw ();
extern float ynf (int, float) throw (); extern float __ynf (int, float) throw ();






extern float erff (float) throw (); extern float __erff (float) throw ();
extern float erfcf (float) throw (); extern float __erfcf (float) throw ();
extern float lgammaf (float) throw (); extern float __lgammaf (float) throw ();






extern float tgammaf (float) throw (); extern float __tgammaf (float) throw ();





extern float gammaf (float) throw (); extern float __gammaf (float) throw ();






extern float lgammaf_r (float, int *__signgamp) throw (); extern float __lgammaf_r (float, int *__signgamp) throw ();







extern float rintf (float __x) throw (); extern float __rintf (float __x) throw ();


extern float nextafterf (float __x, float __y) throw () __attribute__ ((__const__)); extern float __nextafterf (float __x, float __y) throw () __attribute__ ((__const__));

extern float nexttowardf (float __x, long double __y) throw () __attribute__ ((__const__)); extern float __nexttowardf (float __x, long double __y) throw () __attribute__ ((__const__));



extern float remainderf (float __x, float __y) throw (); extern float __remainderf (float __x, float __y) throw ();



extern float scalbnf (float __x, int __n) throw (); extern float __scalbnf (float __x, int __n) throw ();



extern int ilogbf (float __x) throw (); extern int __ilogbf (float __x) throw ();




extern float scalblnf (float __x, long int __n) throw (); extern float __scalblnf (float __x, long int __n) throw ();



extern float nearbyintf (float __x) throw (); extern float __nearbyintf (float __x) throw ();



extern float roundf (float __x) throw () __attribute__ ((__const__)); extern float __roundf (float __x) throw () __attribute__ ((__const__));



extern float truncf (float __x) throw () __attribute__ ((__const__)); extern float __truncf (float __x) throw () __attribute__ ((__const__));




extern float remquof (float __x, float __y, int *__quo) throw (); extern float __remquof (float __x, float __y, int *__quo) throw ();






extern long int lrintf (float __x) throw (); extern long int __lrintf (float __x) throw ();
extern long long int llrintf (float __x) throw (); extern long long int __llrintf (float __x) throw ();



extern long int lroundf (float __x) throw (); extern long int __lroundf (float __x) throw ();
extern long long int llroundf (float __x) throw (); extern long long int __llroundf (float __x) throw ();



extern float fdimf (float __x, float __y) throw (); extern float __fdimf (float __x, float __y) throw ();


extern float fmaxf (float __x, float __y) throw (); extern float __fmaxf (float __x, float __y) throw ();


extern float fminf (float __x, float __y) throw (); extern float __fminf (float __x, float __y) throw ();



extern int __fpclassifyf (float __value) throw ()
     __attribute__ ((__const__));


extern int __signbitf (float __value) throw ()
     __attribute__ ((__const__));



extern float fmaf (float __x, float __y, float __z) throw (); extern float __fmaf (float __x, float __y, float __z) throw ();








extern float scalbf (float __x, float __n) throw (); extern float __scalbf (float __x, float __n) throw ();

extern long double acosl (long double __x) throw (); extern long double __acosl (long double __x) throw ();
extern long double asinl (long double __x) throw (); extern long double __asinl (long double __x) throw ();
extern long double atanl (long double __x) throw (); extern long double __atanl (long double __x) throw ();
extern long double atan2l (long double __y, long double __x) throw (); extern long double __atan2l (long double __y, long double __x) throw ();
extern long double cosl (long double __x) throw (); extern long double __cosl (long double __x) throw ();
extern long double sinl (long double __x) throw (); extern long double __sinl (long double __x) throw ();
extern long double tanl (long double __x) throw (); extern long double __tanl (long double __x) throw ();
extern long double coshl (long double __x) throw (); extern long double __coshl (long double __x) throw ();
extern long double sinhl (long double __x) throw (); extern long double __sinhl (long double __x) throw ();
extern long double tanhl (long double __x) throw (); extern long double __tanhl (long double __x) throw ();

extern void sincosl (long double __x, long double *__sinx, long double *__cosx) throw (); extern void __sincosl (long double __x, long double *__sinx, long double *__cosx) throw ();

extern long double acoshl (long double __x) throw (); extern long double __acoshl (long double __x) throw ();
extern long double asinhl (long double __x) throw (); extern long double __asinhl (long double __x) throw ();
extern long double atanhl (long double __x) throw (); extern long double __atanhl (long double __x) throw ();


extern long double expl (long double __x) throw (); extern long double __expl (long double __x) throw ();
extern long double frexpl (long double __x, int *__exponent) throw (); extern long double __frexpl (long double __x, int *__exponent) throw ();
extern long double ldexpl (long double __x, int __exponent) throw (); extern long double __ldexpl (long double __x, int __exponent) throw ();
extern long double logl (long double __x) throw (); extern long double __logl (long double __x) throw ();
extern long double log10l (long double __x) throw (); extern long double __log10l (long double __x) throw ();
extern long double modfl (long double __x, long double *__iptr) throw (); extern long double __modfl (long double __x, long double *__iptr) throw ();

extern long double exp10l (long double __x) throw (); extern long double __exp10l (long double __x) throw ();
extern long double pow10l (long double __x) throw (); extern long double __pow10l (long double __x) throw ();

extern long double expm1l (long double __x) throw (); extern long double __expm1l (long double __x) throw ();
extern long double log1pl (long double __x) throw (); extern long double __log1pl (long double __x) throw ();
extern long double logbl (long double __x) throw (); extern long double __logbl (long double __x) throw ();


extern long double exp2l (long double __x) throw (); extern long double __exp2l (long double __x) throw ();
extern long double log2l (long double __x) throw (); extern long double __log2l (long double __x) throw ();


extern long double powl (long double __x, long double __y) throw (); extern long double __powl (long double __x, long double __y) throw ();
extern long double sqrtl (long double __x) throw (); extern long double __sqrtl (long double __x) throw ();


extern long double hypotl (long double __x, long double __y) throw (); extern long double __hypotl (long double __x, long double __y) throw ();


extern long double cbrtl (long double __x) throw (); extern long double __cbrtl (long double __x) throw ();


extern long double ceill (long double __x) throw () __attribute__ ((__const__)); extern long double __ceill (long double __x) throw () __attribute__ ((__const__));
extern long double fabsl (long double __x) throw () __attribute__ ((__const__)); extern long double __fabsl (long double __x) throw () __attribute__ ((__const__));
extern long double floorl (long double __x) throw () __attribute__ ((__const__)); extern long double __floorl (long double __x) throw () __attribute__ ((__const__));
extern long double fmodl (long double __x, long double __y) throw (); extern long double __fmodl (long double __x, long double __y) throw ();
extern int __isinfl (long double __value) throw () __attribute__ ((__const__));
extern int __finitel (long double __value) throw () __attribute__ ((__const__));

extern int isinfl (long double __value) throw () __attribute__ ((__const__));
extern int finitel (long double __value) throw () __attribute__ ((__const__));
extern long double dreml (long double __x, long double __y) throw (); extern long double __dreml (long double __x, long double __y) throw ();
extern long double significandl (long double __x) throw (); extern long double __significandl (long double __x) throw ();

extern long double copysignl (long double __x, long double __y) throw () __attribute__ ((__const__)); extern long double __copysignl (long double __x, long double __y) throw () __attribute__ ((__const__));


extern long double nanl (__const char *__tagb) throw () __attribute__ ((__const__)); extern long double __nanl (__const char *__tagb) throw () __attribute__ ((__const__));

extern int __isnanl (long double __value) throw () __attribute__ ((__const__));
extern int isnanl (long double __value) throw () __attribute__ ((__const__));
extern long double j0l (long double) throw (); extern long double __j0l (long double) throw ();
extern long double j1l (long double) throw (); extern long double __j1l (long double) throw ();
extern long double jnl (int, long double) throw (); extern long double __jnl (int, long double) throw ();
extern long double y0l (long double) throw (); extern long double __y0l (long double) throw ();
extern long double y1l (long double) throw (); extern long double __y1l (long double) throw ();
extern long double ynl (int, long double) throw (); extern long double __ynl (int, long double) throw ();

extern long double erfl (long double) throw (); extern long double __erfl (long double) throw ();
extern long double erfcl (long double) throw (); extern long double __erfcl (long double) throw ();
extern long double lgammal (long double) throw (); extern long double __lgammal (long double) throw ();


extern long double tgammal (long double) throw (); extern long double __tgammal (long double) throw ();

extern long double gammal (long double) throw (); extern long double __gammal (long double) throw ();
extern long double lgammal_r (long double, int *__signgamp) throw (); extern long double __lgammal_r (long double, int *__signgamp) throw ();

extern long double rintl (long double __x) throw (); extern long double __rintl (long double __x) throw ();
extern long double nextafterl (long double __x, long double __y) throw () __attribute__ ((__const__)); extern long double __nextafterl (long double __x, long double __y) throw () __attribute__ ((__const__));
extern long double nexttowardl (long double __x, long double __y) throw () __attribute__ ((__const__)); extern long double __nexttowardl (long double __x, long double __y) throw () __attribute__ ((__const__));
extern long double remainderl (long double __x, long double __y) throw (); extern long double __remainderl (long double __x, long double __y) throw ();
extern long double scalbnl (long double __x, int __n) throw (); extern long double __scalbnl (long double __x, int __n) throw ();
extern int ilogbl (long double __x) throw (); extern int __ilogbl (long double __x) throw ();
extern long double scalblnl (long double __x, long int __n) throw (); extern long double __scalblnl (long double __x, long int __n) throw ();
extern long double nearbyintl (long double __x) throw (); extern long double __nearbyintl (long double __x) throw ();
extern long double roundl (long double __x) throw () __attribute__ ((__const__)); extern long double __roundl (long double __x) throw () __attribute__ ((__const__));
extern long double truncl (long double __x) throw () __attribute__ ((__const__)); extern long double __truncl (long double __x) throw () __attribute__ ((__const__));
extern long double remquol (long double __x, long double __y, int *__quo) throw (); extern long double __remquol (long double __x, long double __y, int *__quo) throw ();
extern long int lrintl (long double __x) throw (); extern long int __lrintl (long double __x) throw ();
extern long long int llrintl (long double __x) throw (); extern long long int __llrintl (long double __x) throw ();
extern long int lroundl (long double __x) throw (); extern long int __lroundl (long double __x) throw ();
extern long long int llroundl (long double __x) throw (); extern long long int __llroundl (long double __x) throw ();
extern long double fdiml (long double __x, long double __y) throw (); extern long double __fdiml (long double __x, long double __y) throw ();
extern long double fmaxl (long double __x, long double __y) throw (); extern long double __fmaxl (long double __x, long double __y) throw ();
extern long double fminl (long double __x, long double __y) throw (); extern long double __fminl (long double __x, long double __y) throw ();
extern int __fpclassifyl (long double __value) throw ()
     __attribute__ ((__const__));
extern int __signbitl (long double __value) throw ()
     __attribute__ ((__const__));
extern long double fmal (long double __x, long double __y, long double __z) throw (); extern long double __fmal (long double __x, long double __y, long double __z) throw ();

extern long double scalbl (long double __x, long double __n) throw (); extern long double __scalbl (long double __x, long double __n) throw ();
extern int signgam;
enum
  {
    FP_NAN,
    FP_INFINITE,
    FP_ZERO,
    FP_SUBNORMAL,
    FP_NORMAL
  };
typedef enum
{
  _IEEE_ = -1,
  _SVID_,
  _XOPEN_,
  _POSIX_,
  _ISOC_
} _LIB_VERSION_TYPE;
extern _LIB_VERSION_TYPE _LIB_VERSION;
struct __exception
  {
    int type;
    char *name;
    double arg1;
    double arg2;
    double retval;
  };
extern int matherr (struct __exception *__exc) throw ();
}
typedef unsigned int size_t;
extern "C" {
union wait
  {
    int w_status;
    struct
      {
 unsigned int __w_termsig:7;
 unsigned int __w_coredump:1;
 unsigned int __w_retcode:8;
 unsigned int:16;
      } __wait_terminated;
    struct
      {
 unsigned int __w_stopval:8;
 unsigned int __w_stopsig:8;
 unsigned int:16;
      } __wait_stopped;
  };

typedef struct
  {
    int quot;
    int rem;
  } div_t;
typedef struct
  {
    long int quot;
    long int rem;
  } ldiv_t;


__extension__ typedef struct
  {
    long long int quot;
    long long int rem;
  } lldiv_t;

extern size_t __ctype_get_mb_cur_max (void) throw () ;

extern double atof (__const char *__nptr)
     throw () __attribute__ ((__pure__)) __attribute__ ((__nonnull__ (1))) ;
extern int atoi (__const char *__nptr)
     throw () __attribute__ ((__pure__)) __attribute__ ((__nonnull__ (1))) ;
extern long int atol (__const char *__nptr)
     throw () __attribute__ ((__pure__)) __attribute__ ((__nonnull__ (1))) ;


__extension__ extern long long int atoll (__const char *__nptr)
     throw () __attribute__ ((__pure__)) __attribute__ ((__nonnull__ (1))) ;


extern double strtod (__const char *__restrict __nptr,
        char **__restrict __endptr)
     throw () __attribute__ ((__nonnull__ (1))) ;


extern float strtof (__const char *__restrict __nptr,
       char **__restrict __endptr) throw () __attribute__ ((__nonnull__ (1))) ;
extern long double strtold (__const char *__restrict __nptr,
       char **__restrict __endptr)
     throw () __attribute__ ((__nonnull__ (1))) ;


extern long int strtol (__const char *__restrict __nptr,
   char **__restrict __endptr, int __base)
     throw () __attribute__ ((__nonnull__ (1))) ;
extern unsigned long int strtoul (__const char *__restrict __nptr,
      char **__restrict __endptr, int __base)
     throw () __attribute__ ((__nonnull__ (1))) ;

__extension__
extern long long int strtoq (__const char *__restrict __nptr,
        char **__restrict __endptr, int __base)
     throw () __attribute__ ((__nonnull__ (1))) ;
__extension__
extern unsigned long long int strtouq (__const char *__restrict __nptr,
           char **__restrict __endptr, int __base)
     throw () __attribute__ ((__nonnull__ (1))) ;

__extension__
extern long long int strtoll (__const char *__restrict __nptr,
         char **__restrict __endptr, int __base)
     throw () __attribute__ ((__nonnull__ (1))) ;
__extension__
extern unsigned long long int strtoull (__const char *__restrict __nptr,
     char **__restrict __endptr, int __base)
     throw () __attribute__ ((__nonnull__ (1))) ;

typedef struct __locale_struct
{
  struct __locale_data *__locales[13];
  const unsigned short int *__ctype_b;
  const int *__ctype_tolower;
  const int *__ctype_toupper;
  const char *__names[13];
} *__locale_t;
typedef __locale_t locale_t;
extern long int strtol_l (__const char *__restrict __nptr,
     char **__restrict __endptr, int __base,
     __locale_t __loc) throw () __attribute__ ((__nonnull__ (1, 4))) ;
extern unsigned long int strtoul_l (__const char *__restrict __nptr,
        char **__restrict __endptr,
        int __base, __locale_t __loc)
     throw () __attribute__ ((__nonnull__ (1, 4))) ;
__extension__
extern long long int strtoll_l (__const char *__restrict __nptr,
    char **__restrict __endptr, int __base,
    __locale_t __loc)
     throw () __attribute__ ((__nonnull__ (1, 4))) ;
__extension__
extern unsigned long long int strtoull_l (__const char *__restrict __nptr,
       char **__restrict __endptr,
       int __base, __locale_t __loc)
     throw () __attribute__ ((__nonnull__ (1, 4))) ;
extern double strtod_l (__const char *__restrict __nptr,
   char **__restrict __endptr, __locale_t __loc)
     throw () __attribute__ ((__nonnull__ (1, 3))) ;
extern float strtof_l (__const char *__restrict __nptr,
         char **__restrict __endptr, __locale_t __loc)
     throw () __attribute__ ((__nonnull__ (1, 3))) ;
extern long double strtold_l (__const char *__restrict __nptr,
         char **__restrict __endptr,
         __locale_t __loc)
     throw () __attribute__ ((__nonnull__ (1, 3))) ;
extern char *l64a (long int __n) throw () ;
extern long int a64l (__const char *__s)
     throw () __attribute__ ((__pure__)) __attribute__ ((__nonnull__ (1))) ;
extern "C" {
typedef unsigned char __u_char;
typedef unsigned short int __u_short;
typedef unsigned int __u_int;
typedef unsigned long int __u_long;
typedef signed char __int8_t;
typedef unsigned char __uint8_t;
typedef signed short int __int16_t;
typedef unsigned short int __uint16_t;
typedef signed int __int32_t;
typedef unsigned int __uint32_t;
__extension__ typedef signed long long int __int64_t;
__extension__ typedef unsigned long long int __uint64_t;
__extension__ typedef long long int __quad_t;
__extension__ typedef unsigned long long int __u_quad_t;
__extension__ typedef __u_quad_t __dev_t;
__extension__ typedef unsigned int __uid_t;
__extension__ typedef unsigned int __gid_t;
__extension__ typedef unsigned long int __ino_t;
__extension__ typedef __u_quad_t __ino64_t;
__extension__ typedef unsigned int __mode_t;
__extension__ typedef unsigned int __nlink_t;
__extension__ typedef long int __off_t;
__extension__ typedef __quad_t __off64_t;
__extension__ typedef int __pid_t;
__extension__ typedef struct { int __val[2]; } __fsid_t;
__extension__ typedef long int __clock_t;
__extension__ typedef unsigned long int __rlim_t;
__extension__ typedef __u_quad_t __rlim64_t;
__extension__ typedef unsigned int __id_t;
__extension__ typedef long int __time_t;
__extension__ typedef unsigned int __useconds_t;
__extension__ typedef long int __suseconds_t;
__extension__ typedef int __daddr_t;
__extension__ typedef long int __swblk_t;
__extension__ typedef int __key_t;
__extension__ typedef int __clockid_t;
__extension__ typedef void * __timer_t;
__extension__ typedef long int __blksize_t;
__extension__ typedef long int __blkcnt_t;
__extension__ typedef __quad_t __blkcnt64_t;
__extension__ typedef unsigned long int __fsblkcnt_t;
__extension__ typedef __u_quad_t __fsblkcnt64_t;
__extension__ typedef unsigned long int __fsfilcnt_t;
__extension__ typedef __u_quad_t __fsfilcnt64_t;
__extension__ typedef int __ssize_t;
typedef __off64_t __loff_t;
typedef __quad_t *__qaddr_t;
typedef char *__caddr_t;
__extension__ typedef int __intptr_t;
__extension__ typedef unsigned int __socklen_t;
typedef __u_char u_char;
typedef __u_short u_short;
typedef __u_int u_int;
typedef __u_long u_long;
typedef __quad_t quad_t;
typedef __u_quad_t u_quad_t;
typedef __fsid_t fsid_t;
typedef __loff_t loff_t;
typedef __ino_t ino_t;
typedef __ino64_t ino64_t;
typedef __dev_t dev_t;
typedef __gid_t gid_t;
typedef __mode_t mode_t;
typedef __nlink_t nlink_t;
typedef __uid_t uid_t;
typedef __off_t off_t;
typedef __off64_t off64_t;
typedef __pid_t pid_t;
typedef __id_t id_t;
typedef __ssize_t ssize_t;
typedef __daddr_t daddr_t;
typedef __caddr_t caddr_t;
typedef __key_t key_t;

typedef __clock_t clock_t;



typedef __time_t time_t;


typedef __clockid_t clockid_t;
typedef __timer_t timer_t;
typedef __useconds_t useconds_t;
typedef __suseconds_t suseconds_t;
typedef unsigned long int ulong;
typedef unsigned short int ushort;
typedef unsigned int uint;
typedef int int8_t __attribute__ ((__mode__ (__QI__)));
typedef int int16_t __attribute__ ((__mode__ (__HI__)));
typedef int int32_t __attribute__ ((__mode__ (__SI__)));
typedef int int64_t __attribute__ ((__mode__ (__DI__)));
typedef unsigned int u_int8_t __attribute__ ((__mode__ (__QI__)));
typedef unsigned int u_int16_t __attribute__ ((__mode__ (__HI__)));
typedef unsigned int u_int32_t __attribute__ ((__mode__ (__SI__)));
typedef unsigned int u_int64_t __attribute__ ((__mode__ (__DI__)));
typedef int register_t __attribute__ ((__mode__ (__word__)));
typedef int __sig_atomic_t;
typedef struct
  {
    unsigned long int __val[(1024 / (8 * sizeof (unsigned long int)))];
  } __sigset_t;
typedef __sigset_t sigset_t;
struct timespec
  {
    __time_t tv_sec;
    long int tv_nsec;
  };
struct timeval
  {
    __time_t tv_sec;
    __suseconds_t tv_usec;
  };
typedef long int __fd_mask;
typedef struct
  {
    __fd_mask fds_bits[1024 / (8 * (int) sizeof (__fd_mask))];
  } fd_set;
typedef __fd_mask fd_mask;
extern "C" {
extern int select (int __nfds, fd_set *__restrict __readfds,
     fd_set *__restrict __writefds,
     fd_set *__restrict __exceptfds,
     struct timeval *__restrict __timeout);
extern int pselect (int __nfds, fd_set *__restrict __readfds,
      fd_set *__restrict __writefds,
      fd_set *__restrict __exceptfds,
      const struct timespec *__restrict __timeout,
      const __sigset_t *__restrict __sigmask);
}
__extension__
extern unsigned int gnu_dev_major (unsigned long long int __dev)
     throw ();
__extension__
extern unsigned int gnu_dev_minor (unsigned long long int __dev)
     throw ();
__extension__
extern unsigned long long int gnu_dev_makedev (unsigned int __major,
            unsigned int __minor)
     throw ();
typedef __blksize_t blksize_t;
typedef __blkcnt_t blkcnt_t;
typedef __fsblkcnt_t fsblkcnt_t;
typedef __fsfilcnt_t fsfilcnt_t;
typedef __blkcnt64_t blkcnt64_t;
typedef __fsblkcnt64_t fsblkcnt64_t;
typedef __fsfilcnt64_t fsfilcnt64_t;
typedef unsigned long int pthread_t;
typedef union
{
  char __size[36];
  long int __align;
} pthread_attr_t;
typedef struct __pthread_internal_slist
{
  struct __pthread_internal_slist *__next;
} __pthread_slist_t;
typedef union
{
  struct __pthread_mutex_s
  {
    int __lock;
    unsigned int __count;
    int __owner;
    int __kind;
    unsigned int __nusers;
    __extension__ union
    {
      int __spins;
      __pthread_slist_t __list;
    };
  } __data;
  char __size[24];
  long int __align;
} pthread_mutex_t;
typedef union
{
  char __size[4];
  int __align;
} pthread_mutexattr_t;
typedef union
{
  struct
  {
    int __lock;
    unsigned int __futex;
    __extension__ unsigned long long int __total_seq;
    __extension__ unsigned long long int __wakeup_seq;
    __extension__ unsigned long long int __woken_seq;
    void *__mutex;
    unsigned int __nwaiters;
    unsigned int __broadcast_seq;
  } __data;
  char __size[48];
  __extension__ long long int __align;
} pthread_cond_t;
typedef union
{
  char __size[4];
  int __align;
} pthread_condattr_t;
typedef unsigned int pthread_key_t;
typedef int pthread_once_t;
typedef union
{
  struct
  {
    int __lock;
    unsigned int __nr_readers;
    unsigned int __readers_wakeup;
    unsigned int __writer_wakeup;
    unsigned int __nr_readers_queued;
    unsigned int __nr_writers_queued;
    unsigned char __flags;
    unsigned char __shared;
    unsigned char __pad1;
    unsigned char __pad2;
    int __writer;
  } __data;
  char __size[32];
  long int __align;
} pthread_rwlock_t;
typedef union
{
  char __size[8];
  long int __align;
} pthread_rwlockattr_t;
typedef volatile int pthread_spinlock_t;
typedef union
{
  char __size[20];
  long int __align;
} pthread_barrier_t;
typedef union
{
  char __size[4];
  int __align;
} pthread_barrierattr_t;
}
extern long int random (void) throw ();
extern void srandom (unsigned int __seed) throw ();
extern char *initstate (unsigned int __seed, char *__statebuf,
   size_t __statelen) throw () __attribute__ ((__nonnull__ (2)));
extern char *setstate (char *__statebuf) throw () __attribute__ ((__nonnull__ (1)));
struct random_data
  {
    int32_t *fptr;
    int32_t *rptr;
    int32_t *state;
    int rand_type;
    int rand_deg;
    int rand_sep;
    int32_t *end_ptr;
  };
extern int random_r (struct random_data *__restrict __buf,
       int32_t *__restrict __result) throw () __attribute__ ((__nonnull__ (1, 2)));
extern int srandom_r (unsigned int __seed, struct random_data *__buf)
     throw () __attribute__ ((__nonnull__ (2)));
extern int initstate_r (unsigned int __seed, char *__restrict __statebuf,
   size_t __statelen,
   struct random_data *__restrict __buf)
     throw () __attribute__ ((__nonnull__ (2, 4)));
extern int setstate_r (char *__restrict __statebuf,
         struct random_data *__restrict __buf)
     throw () __attribute__ ((__nonnull__ (1, 2)));

extern int rand (void) throw ();
extern void srand (unsigned int __seed) throw ();

extern int rand_r (unsigned int *__seed) throw ();
extern double drand48 (void) throw ();
extern double erand48 (unsigned short int __xsubi[3]) throw () __attribute__ ((__nonnull__ (1)));
extern long int lrand48 (void) throw ();
extern long int nrand48 (unsigned short int __xsubi[3])
     throw () __attribute__ ((__nonnull__ (1)));
extern long int mrand48 (void) throw ();
extern long int jrand48 (unsigned short int __xsubi[3])
     throw () __attribute__ ((__nonnull__ (1)));
extern void srand48 (long int __seedval) throw ();
extern unsigned short int *seed48 (unsigned short int __seed16v[3])
     throw () __attribute__ ((__nonnull__ (1)));
extern void lcong48 (unsigned short int __param[7]) throw () __attribute__ ((__nonnull__ (1)));
struct drand48_data
  {
    unsigned short int __x[3];
    unsigned short int __old_x[3];
    unsigned short int __c;
    unsigned short int __init;
    unsigned long long int __a;
  };
extern int drand48_r (struct drand48_data *__restrict __buffer,
        double *__restrict __result) throw () __attribute__ ((__nonnull__ (1, 2)));
extern int erand48_r (unsigned short int __xsubi[3],
        struct drand48_data *__restrict __buffer,
        double *__restrict __result) throw () __attribute__ ((__nonnull__ (1, 2)));
extern int lrand48_r (struct drand48_data *__restrict __buffer,
        long int *__restrict __result)
     throw () __attribute__ ((__nonnull__ (1, 2)));
extern int nrand48_r (unsigned short int __xsubi[3],
        struct drand48_data *__restrict __buffer,
        long int *__restrict __result)
     throw () __attribute__ ((__nonnull__ (1, 2)));
extern int mrand48_r (struct drand48_data *__restrict __buffer,
        long int *__restrict __result)
     throw () __attribute__ ((__nonnull__ (1, 2)));
extern int jrand48_r (unsigned short int __xsubi[3],
        struct drand48_data *__restrict __buffer,
        long int *__restrict __result)
     throw () __attribute__ ((__nonnull__ (1, 2)));
extern int srand48_r (long int __seedval, struct drand48_data *__buffer)
     throw () __attribute__ ((__nonnull__ (2)));
extern int seed48_r (unsigned short int __seed16v[3],
       struct drand48_data *__buffer) throw () __attribute__ ((__nonnull__ (1, 2)));
extern int lcong48_r (unsigned short int __param[7],
        struct drand48_data *__buffer)
     throw () __attribute__ ((__nonnull__ (1, 2)));

extern void *malloc (size_t __size) throw () __attribute__ ((__malloc__)) ;
extern void *calloc (size_t __nmemb, size_t __size)
     throw () __attribute__ ((__malloc__)) ;


extern void *realloc (void *__ptr, size_t __size)
     throw () __attribute__ ((__warn_unused_result__));
extern void free (void *__ptr) throw ();

extern void cfree (void *__ptr) throw ();
extern "C" {
extern void *alloca (size_t __size) throw ();
}
extern void *valloc (size_t __size) throw () __attribute__ ((__malloc__)) ;
extern int posix_memalign (void **__memptr, size_t __alignment, size_t __size)
     throw () __attribute__ ((__nonnull__ (1))) ;

extern void abort (void) throw () __attribute__ ((__noreturn__));
extern int atexit (void (*__func) (void)) throw () __attribute__ ((__nonnull__ (1)));
extern "C++" int at_quick_exit (void (*__func) (void))
     throw () __asm ("at_quick_exit") __attribute__ ((__nonnull__ (1)));

extern int on_exit (void (*__func) (int __status, void *__arg), void *__arg)
     throw () __attribute__ ((__nonnull__ (1)));

extern void exit (int __status) throw () __attribute__ ((__noreturn__));
extern void quick_exit (int __status) throw () __attribute__ ((__noreturn__));


extern void _Exit (int __status) throw () __attribute__ ((__noreturn__));


extern char *getenv (__const char *__name) throw () __attribute__ ((__nonnull__ (1))) ;

extern char *__secure_getenv (__const char *__name)
     throw () __attribute__ ((__nonnull__ (1))) ;
extern int putenv (char *__string) throw () __attribute__ ((__nonnull__ (1)));
extern int setenv (__const char *__name, __const char *__value, int __replace)
     throw () __attribute__ ((__nonnull__ (2)));
extern int unsetenv (__const char *__name) throw () __attribute__ ((__nonnull__ (1)));
extern int clearenv (void) throw ();
extern char *mktemp (char *__template) throw () __attribute__ ((__nonnull__ (1))) ;
extern int mkstemp (char *__template) __attribute__ ((__nonnull__ (1))) ;
extern int mkstemp64 (char *__template) __attribute__ ((__nonnull__ (1))) ;
extern int mkstemps (char *__template, int __suffixlen) __attribute__ ((__nonnull__ (1))) ;
extern int mkstemps64 (char *__template, int __suffixlen)
     __attribute__ ((__nonnull__ (1))) ;
extern char *mkdtemp (char *__template) throw () __attribute__ ((__nonnull__ (1))) ;
extern int mkostemp (char *__template, int __flags) __attribute__ ((__nonnull__ (1))) ;
extern int mkostemp64 (char *__template, int __flags) __attribute__ ((__nonnull__ (1))) ;
extern int mkostemps (char *__template, int __suffixlen, int __flags)
     __attribute__ ((__nonnull__ (1))) ;
extern int mkostemps64 (char *__template, int __suffixlen, int __flags)
     __attribute__ ((__nonnull__ (1))) ;

extern int system (__const char *__command) ;

extern char *canonicalize_file_name (__const char *__name)
     throw () __attribute__ ((__nonnull__ (1))) ;
extern char *realpath (__const char *__restrict __name,
         char *__restrict __resolved) throw () ;
typedef int (*__compar_fn_t) (__const void *, __const void *);
typedef __compar_fn_t comparison_fn_t;
typedef int (*__compar_d_fn_t) (__const void *, __const void *, void *);

extern void *bsearch (__const void *__key, __const void *__base,
        size_t __nmemb, size_t __size, __compar_fn_t __compar)
     __attribute__ ((__nonnull__ (1, 2, 5))) ;
extern void qsort (void *__base, size_t __nmemb, size_t __size,
     __compar_fn_t __compar) __attribute__ ((__nonnull__ (1, 4)));
extern void qsort_r (void *__base, size_t __nmemb, size_t __size,
       __compar_d_fn_t __compar, void *__arg)
  __attribute__ ((__nonnull__ (1, 4)));
extern int abs (int __x) throw () __attribute__ ((__const__)) ;
extern long int labs (long int __x) throw () __attribute__ ((__const__)) ;

__extension__ extern long long int llabs (long long int __x)
     throw () __attribute__ ((__const__)) ;

extern div_t div (int __numer, int __denom)
     throw () __attribute__ ((__const__)) ;
extern ldiv_t ldiv (long int __numer, long int __denom)
     throw () __attribute__ ((__const__)) ;


__extension__ extern lldiv_t lldiv (long long int __numer,
        long long int __denom)
     throw () __attribute__ ((__const__)) ;

extern char *ecvt (double __value, int __ndigit, int *__restrict __decpt,
     int *__restrict __sign) throw () __attribute__ ((__nonnull__ (3, 4))) ;
extern char *fcvt (double __value, int __ndigit, int *__restrict __decpt,
     int *__restrict __sign) throw () __attribute__ ((__nonnull__ (3, 4))) ;
extern char *gcvt (double __value, int __ndigit, char *__buf)
     throw () __attribute__ ((__nonnull__ (3))) ;
extern char *qecvt (long double __value, int __ndigit,
      int *__restrict __decpt, int *__restrict __sign)
     throw () __attribute__ ((__nonnull__ (3, 4))) ;
extern char *qfcvt (long double __value, int __ndigit,
      int *__restrict __decpt, int *__restrict __sign)
     throw () __attribute__ ((__nonnull__ (3, 4))) ;
extern char *qgcvt (long double __value, int __ndigit, char *__buf)
     throw () __attribute__ ((__nonnull__ (3))) ;
extern int ecvt_r (double __value, int __ndigit, int *__restrict __decpt,
     int *__restrict __sign, char *__restrict __buf,
     size_t __len) throw () __attribute__ ((__nonnull__ (3, 4, 5)));
extern int fcvt_r (double __value, int __ndigit, int *__restrict __decpt,
     int *__restrict __sign, char *__restrict __buf,
     size_t __len) throw () __attribute__ ((__nonnull__ (3, 4, 5)));
extern int qecvt_r (long double __value, int __ndigit,
      int *__restrict __decpt, int *__restrict __sign,
      char *__restrict __buf, size_t __len)
     throw () __attribute__ ((__nonnull__ (3, 4, 5)));
extern int qfcvt_r (long double __value, int __ndigit,
      int *__restrict __decpt, int *__restrict __sign,
      char *__restrict __buf, size_t __len)
     throw () __attribute__ ((__nonnull__ (3, 4, 5)));

extern int mblen (__const char *__s, size_t __n) throw () ;
extern int mbtowc (wchar_t *__restrict __pwc,
     __const char *__restrict __s, size_t __n) throw () ;
extern int wctomb (char *__s, wchar_t __wchar) throw () ;
extern size_t mbstowcs (wchar_t *__restrict __pwcs,
   __const char *__restrict __s, size_t __n) throw ();
extern size_t wcstombs (char *__restrict __s,
   __const wchar_t *__restrict __pwcs, size_t __n)
     throw ();

extern int rpmatch (__const char *__response) throw () __attribute__ ((__nonnull__ (1))) ;
extern int getsubopt (char **__restrict __optionp,
        char *__const *__restrict __tokens,
        char **__restrict __valuep)
     throw () __attribute__ ((__nonnull__ (1, 2, 3))) ;
extern void setkey (__const char *__key) throw () __attribute__ ((__nonnull__ (1)));
extern int posix_openpt (int __oflag) ;
extern int grantpt (int __fd) throw ();
extern int unlockpt (int __fd) throw ();
extern char *ptsname (int __fd) throw () ;
extern int ptsname_r (int __fd, char *__buf, size_t __buflen)
     throw () __attribute__ ((__nonnull__ (2)));
extern int getpt (void);
extern int getloadavg (double __loadavg[], int __nelem)
     throw () __attribute__ ((__nonnull__ (1)));
}
 typedef float G4double;
typedef float G4float;
typedef int G4int;
typedef int G4bool;
typedef long G4long;
 G4double kInfinity = 1.0E37;
 int BlockSize = 32;
 int Multiplier = 4;
 G4double twopi = 2.0*3.14159265358979323846;
 G4double kMinExitingNormalCosine = 1E-3;
typedef enum {kOutside,kSurface,kInside} EInside;
typedef enum {kNormal,kReplica,kParameterised} EVolume;
typedef enum {kXAxis,kYAxis,kZAxis,kRho,kRadial3D,kPhi,kUndefined} EAxis;
typedef enum { kBox = 0 , kOrb, kTubs, kCons, kPolyCone, Solidcount } ESolid;
typedef struct
{
 G4double x,y,z;
}
G4ThreeVector;
inline
G4ThreeVector G4ThreeVector_create( G4double x, G4double y, G4double z )
{
 G4ThreeVector v =
   {x,y,z};
 return v;
}
inline
G4ThreeVector G4ThreeVector_saxpy( G4double a, G4ThreeVector x, G4ThreeVector y )
{
 return G4ThreeVector_create(
  a*x.x + y.x,
  a*x.y + y.y,
  a*x.z + y.z );
}
inline
G4ThreeVector G4ThreeVector_sum( G4ThreeVector a, G4ThreeVector b )
{
 return G4ThreeVector_create( a.x+b.x, a.y+b.y, a.z+b.z );
}
inline
G4ThreeVector G4ThreeVector_subtract( G4ThreeVector a, G4ThreeVector b )
{
 return G4ThreeVector_create( a.x-b.x, a.y-b.y, a.z-b.z );
}
inline
G4ThreeVector G4ThreeVector_sum_assign( G4ThreeVector *This, G4ThreeVector b )
{
 (*This).x += b.x;
 (*This).y += b.y;
 (*This).z += b.z;
 return *This;
}
inline
G4ThreeVector G4ThreeVector_subtract_assign( G4ThreeVector *This, G4ThreeVector b )
{
 (*This).x -= b.x;
 (*This).y -= b.y;
 (*This).z -= b.z;
 return *This;
}
inline
G4ThreeVector G4ThreeVector_mult_assign( G4ThreeVector *This, G4double m )
{
 (*This).x *= m;
 (*This).y *= m;
 (*This).z *= m;
 return *This;
}
inline
G4ThreeVector G4ThreeVector_negation( G4ThreeVector a )
{
 return G4ThreeVector_create( -a.x, -a.y, -a.z );
}
inline
G4double G4ThreeVector_mag2( G4ThreeVector v )
{
 return v.x*v.x + v.y*v.y + v.z*v.z;
}
inline
G4double G4ThreeVector_mag( G4ThreeVector v )
{
 return sqrt(G4ThreeVector_mag2(v));
}
inline
G4double G4ThreeVector_dot( G4ThreeVector a, G4ThreeVector b )
{
 return a.x*b.x + a.y*b.y + a.z*b.z;
}
inline
G4ThreeVector G4ThreeVector_cross( G4ThreeVector a, G4ThreeVector p )
{
 return G4ThreeVector_create(
  a.y*p.z-p.y*a.z,
  a.z*p.x-p.z*a.x,
  a.x*p.y-p.x*a.y );
}
inline
G4ThreeVector G4ThreeVector_mult( G4ThreeVector a, G4double m )
{
 return G4ThreeVector_create( a.x*m, a.y*m, a.z*m );
}
inline
G4ThreeVector G4ThreeVector_unit( G4ThreeVector v )
{
 G4double l = G4ThreeVector_mag(v);
 if ( l > 0 )
  return G4ThreeVector_mult( v, 1.0/l );
 return v;
}
inline
G4bool G4ThreeVector_equal( G4ThreeVector a, G4ThreeVector b )
{
 return a.x == b.x && a.y == b.y && a.z == b.z;
}
inline
G4double G4ThreeVector_diff2( G4ThreeVector a, G4ThreeVector b )
{
 return G4ThreeVector_mag2( G4ThreeVector_subtract(a,b) );
}
inline
G4double G4ThreeVector_coord( G4ThreeVector v, EAxis axis )
{
 switch( axis )
 {
 case kXAxis: return v.x;
 case kYAxis: return v.y;
 case kZAxis: return v.z;
 default:
  ((false) ? static_cast<void> (0) : __assert_fail ("false", "G4ThreeVector_inline.c", 175, __PRETTY_FUNCTION__));
  return 0;
 }
}
inline
void G4ThreeVector_set_coord( G4ThreeVector *v, EAxis axis, G4double val )
{
 switch( axis )
 {
 case kXAxis: v->x = val; break;
 case kYAxis: v->y = val; break;
 case kZAxis: v->z = val; break;
 default:
  ((false) ? static_cast<void> (0) : __assert_fail ("false", "G4ThreeVector_inline.c", 189, __PRETTY_FUNCTION__));
  break;
 }
}
typedef struct
{
 G4ThreeVector pos, dir;
}
StubParticle;
typedef struct
{
 G4ThreeVector pos, dir;
 G4double t;
}
ParticleWithLifetime;
typedef StubParticle Particle;
 void Prefix_Sum ( SHAREDMEM int * input, SHAREDMEM int * output, int length)
{
 int tid = get_global_id(0);
 int offset = 1;
 if ( tid< length)
  output[tid] = input[ tid ];
 for(int d = length>>1; d > 0; d >>=1)
 {
BARRIER_FLEXIBLE;
  if(tid<d)
  {
   int ai = offset*(2*tid + 1) - 1;
   int bi = offset*(2*tid + 2) - 1;
   output[bi] += output[ai];
  }
  offset *= 2;
 }
 if(tid == 0)
 {
  output[length - 1] = 0;
 }
 for(int d = 1; d < length ; d *= 2)
 {
  offset >>=1;
  BARRIER_FLEXIBLE;
  if(tid < d)
  {
   int ai = offset*(2*tid + 1) - 1;
   int bi = offset*(2*tid + 2) - 1;
   float t = output[ai];
   output[ai] = output[bi];
   output[bi] += t;
  }
 }
BARRIER_FLEXIBLE;
}

G4bool NoStepReduction( SHAREDMEM G4bool * noStepArray, int length )
{
 int tid = get_global_id(0);
 int offset = 1;
 for(int d = length>>1; d > 0; d >>=1)
 {
     BARRIER_FLEXIBLE;
  if(tid<d)
  {
   int ai = offset*(2*tid + 1) - 1;
   int bi = offset*(2*tid + 2) - 1;
   noStepArray[bi] = (noStepArray[ai] || noStepArray[bi]);
  }
  offset *= 2;
 }
 G4bool result = noStepArray[ length - 1 ];
 BARRIER_FLEXIBLE;
 return result;
}
typedef struct
{
 G4double
  rxx, rxy, rxz,
  ryx, ryy, ryz,
  rzx, rzy, rzz;
 G4double align;
}
G4RotationMatrix;
inline
G4RotationMatrix G4RotationMatrix_create_elements
   (G4double mxx, G4double mxy, G4double mxz,
    G4double myx, G4double myy, G4double myz,
    G4double mzx, G4double mzy, G4double mzz)
{
 G4RotationMatrix r =
  { mxx,mxy,mxz, myx,myy,myz, mzx,mzy,mzz
  , 0
  };
 return r;
}
inline
G4ThreeVector G4RotationMatrix_apply (const G4RotationMatrix *This, G4ThreeVector p)
{
  return G4ThreeVector_create(
     This->rxx*p.x + This->rxy*p.y + This->rxz*p.z,
                    This->ryx*p.x + This->ryy*p.y + This->ryz*p.z,
                    This->rzx*p.x + This->rzy*p.y + This->rzz*p.z);
}
inline
G4RotationMatrix G4RotationMatrix_mult (const G4RotationMatrix *This, const G4RotationMatrix *other)
{
 return G4RotationMatrix_create_elements(
  This->rxx*(*other).rxx + This->rxy*(*other).ryx + This->rxz*(*other).rzx,
  This->rxx*(*other).rxy + This->rxy*(*other).ryy + This->rxz*(*other).rzy,
  This->rxx*(*other).rxz + This->rxy*(*other).ryz + This->rxz*(*other).rzz,
  This->ryx*(*other).rxx + This->ryy*(*other).ryx + This->ryz*(*other).rzx,
  This->ryx*(*other).rxy + This->ryy*(*other).ryy + This->ryz*(*other).rzy,
  This->ryx*(*other).rxz + This->ryy*(*other).ryz + This->ryz*(*other).rzz,
  This->rzx*(*other).rxx + This->rzy*(*other).ryx + This->rzz*(*other).rzx,
  This->rzx*(*other).rxy + This->rzy*(*other).ryy + This->rzz*(*other).rzy,
  This->rzx*(*other).rxz + This->rzy*(*other).ryz + This->rzz*(*other).rzz );
}
inline
G4RotationMatrix G4RotationMatrix_transform(G4RotationMatrix *This, const G4RotationMatrix *other)
{
 *This = G4RotationMatrix_mult(other,This);
 return *This;
}
inline
G4RotationMatrix G4RotationMatrix_inverse(const G4RotationMatrix *This)
{
 return G4RotationMatrix_create_elements(
  This->rxx, This->ryx, This->rzx,
  This->rxy, This->ryy, This->rzy,
  This->rxz, This->ryz, This->rzz );
}
inline
G4RotationMatrix G4RotationMatrix_invert(G4RotationMatrix *This)
{
 return *This = G4RotationMatrix_inverse(This);
}
typedef struct
{
  G4double rxx,rxy,rxz;
  G4double ryx,ryy,ryz;
  G4double rzx,rzy,rzz;
  G4double tx,ty,tz;
}
G4AffineTransform;
inline
void G4AffineTransform_ctor_id( G4AffineTransform *This )
{
 This->rxx = 1;
 This->ryy = 1;
 This->rzz = 1;
 This->rxy = 0;
 This->rxz = 0;
 This->ryx = 0;
 This->ryz = 0;
 This->rzx = 0;
 This->rzy = 0;
 This->tx = 0;
 This->ty = 0;
 This->tz = 0;
}
inline
void G4AffineTransform_ctor_vector( G4AffineTransform *This, G4ThreeVector tlate)
{
 G4AffineTransform_ctor_id( This );
 This->tx = tlate.x;
 This->ty = tlate.y;
 This->tz = tlate.z;
}
inline
void G4AffineTransform_ctor_matrix( G4AffineTransform *This, G4RotationMatrix rot)
{
 G4AffineTransform_ctor_id( This );
 This->rxx = rot.rxx;
 This->ryy = rot.ryy;
 This->rzz = rot.rzz;
 This->rxy = rot.rxy;
 This->rxz = rot.rxz;
 This->ryx = rot.ryx;
 This->ryz = rot.ryz;
 This->rzx = rot.rzx;
 This->rzy = rot.rzy;
}
inline
void G4AffineTransform_ctor_full(
 G4AffineTransform *This, G4RotationMatrix rot, G4ThreeVector tlate )
{
 This->rxx = rot.rxx;
 This->ryy = rot.ryy;
 This->rzz = rot.rzz;
 This->rxy = rot.rxy;
 This->rxz = rot.rxz;
 This->ryx = rot.ryx;
 This->ryz = rot.ryz;
 This->rzx = rot.rzx;
 This->rzy = rot.rzy;
 This->tx = tlate.x;
 This->ty = tlate.y;
 This->tz = tlate.z;
}
inline
void G4AffineTransform_ctor_ptr(
 G4AffineTransform *This, const G4RotationMatrix *rot, G4ThreeVector tlate )
{
 if (rot) G4AffineTransform_ctor_full( This, *rot, tlate );
 else G4AffineTransform_ctor_vector( This, tlate );
}
inline
void G4AffineTransform_ctor_elements(
  G4AffineTransform *This,
  const G4double prxx,const G4double prxy,const G4double prxz,
  const G4double pryx,const G4double pryy,const G4double pryz,
  const G4double przx,const G4double przy,const G4double przz,
  const G4double ptx,const G4double pty,const G4double ptz)
{
 This->rxx = prxx;
 This->ryy = pryy;
 This->rzz = przz;
 This->rxy = prxy;
 This->rxz = prxz;
 This->ryx = pryx;
 This->ryz = pryz;
 This->rzx = przx;
 This->rzy = przy;
 This->tx = ptx;
 This->ty = pty;
 This->tz = ptz;
}
inline
G4AffineTransform G4AffineTransform_create_id(void)
{
 G4AffineTransform t;
 G4AffineTransform_ctor_id(&t);
 return t;
}
inline
G4AffineTransform G4AffineTransform_create_vector(G4ThreeVector tlate)
{
 G4AffineTransform t;
 G4AffineTransform_ctor_vector(&t,tlate);
 return t;
}
inline
G4AffineTransform G4AffineTransform_create_matrix( G4RotationMatrix rot )
{
 G4AffineTransform t;
 G4AffineTransform_ctor_matrix(&t,rot);
 return t;
}
inline
G4AffineTransform G4AffineTransform_create_full(
 G4RotationMatrix rot, G4ThreeVector tlate )
{
 G4AffineTransform t;
 G4AffineTransform_ctor_full(&t,rot,tlate);
 return t;
}
inline
G4AffineTransform G4AffineTransform_create_ptr(
 const G4RotationMatrix *rot, G4ThreeVector tlate )
{
 G4AffineTransform t;
 G4AffineTransform_ctor_ptr(&t,rot,tlate);
 return t;
}
inline
G4AffineTransform G4AffineTransform_create_elements(
  const G4double prxx,const G4double prxy,const G4double prxz,
  const G4double pryx,const G4double pryy,const G4double pryz,
  const G4double przx,const G4double przy,const G4double przz,
  const G4double ptx,const G4double pty,const G4double ptz)
{
 G4AffineTransform t;
 G4AffineTransform_ctor_elements(&t,
  prxx,prxy,prxz,
  pryx,pryy,pryz,
  przx,przy,przz,
  ptx,pty,ptz);
 return t;
}
inline G4AffineTransform
G4AffineTransform_InverseProduct(
 G4AffineTransform *This,
 const G4AffineTransform* ptrtf1,
 const G4AffineTransform* ptrtf2)
{
        G4double itf2tx = - (*ptrtf2).tx*(*ptrtf2).rxx - (*ptrtf2).ty*(*ptrtf2).rxy - (*ptrtf2).tz*(*ptrtf2).rxz;
        G4double itf2ty = - (*ptrtf2).tx*(*ptrtf2).ryx - (*ptrtf2).ty*(*ptrtf2).ryy - (*ptrtf2).tz*(*ptrtf2).ryz;
        G4double itf2tz = - (*ptrtf2).tx*(*ptrtf2).rzx - (*ptrtf2).ty*(*ptrtf2).rzy - (*ptrtf2).tz*(*ptrtf2).rzz;
        This->rxx=(*ptrtf1).rxx*(*ptrtf2).rxx+(*ptrtf1).rxy*(*ptrtf2).rxy+(*ptrtf1).rxz*(*ptrtf2).rxz;
        This->rxy=(*ptrtf1).rxx*(*ptrtf2).ryx+(*ptrtf1).rxy*(*ptrtf2).ryy+(*ptrtf1).rxz*(*ptrtf2).ryz;
        This->rxz=(*ptrtf1).rxx*(*ptrtf2).rzx+(*ptrtf1).rxy*(*ptrtf2).rzy+(*ptrtf1).rxz*(*ptrtf2).rzz;
        This->ryx=(*ptrtf1).ryx*(*ptrtf2).rxx+(*ptrtf1).ryy*(*ptrtf2).rxy+(*ptrtf1).ryz*(*ptrtf2).rxz;
        This->ryy=(*ptrtf1).ryx*(*ptrtf2).ryx+(*ptrtf1).ryy*(*ptrtf2).ryy+(*ptrtf1).ryz*(*ptrtf2).ryz;
        This->ryz=(*ptrtf1).ryx*(*ptrtf2).rzx+(*ptrtf1).ryy*(*ptrtf2).rzy+(*ptrtf1).ryz*(*ptrtf2).rzz;
        This->rzx=(*ptrtf1).rzx*(*ptrtf2).rxx+(*ptrtf1).rzy*(*ptrtf2).rxy+(*ptrtf1).rzz*(*ptrtf2).rxz;
        This->rzy=(*ptrtf1).rzx*(*ptrtf2).ryx+(*ptrtf1).rzy*(*ptrtf2).ryy+(*ptrtf1).rzz*(*ptrtf2).ryz;
        This->rzz=(*ptrtf1).rzx*(*ptrtf2).rzx+(*ptrtf1).rzy*(*ptrtf2).rzy+(*ptrtf1).rzz*(*ptrtf2).rzz;
        This->tx=(*ptrtf1).tx*(*ptrtf2).rxx+(*ptrtf1).ty*(*ptrtf2).rxy+(*ptrtf1).tz*(*ptrtf2).rxz+itf2tx;
        This->ty=(*ptrtf1).tx*(*ptrtf2).ryx+(*ptrtf1).ty*(*ptrtf2).ryy+(*ptrtf1).tz*(*ptrtf2).ryz+itf2ty;
        This->tz=(*ptrtf1).tx*(*ptrtf2).rzx+(*ptrtf1).ty*(*ptrtf2).rzy+(*ptrtf1).tz*(*ptrtf2).rzz+itf2tz;
        return *This;
}
inline
G4ThreeVector G4AffineTransform_TransformPoint(const G4AffineTransform *This, G4ThreeVector vec)
{
        return G4ThreeVector_create(
   vec.x*This->rxx + vec.y*This->ryx + vec.z*This->rzx + This->tx,
   vec.x*This->rxy + vec.y*This->ryy + vec.z*This->rzy + This->ty,
   vec.x*This->rxz + vec.y*This->ryz + vec.z*This->rzz + This->tz );
}
inline
G4ThreeVector G4AffineTransform_TransformAxis(const G4AffineTransform *This, G4ThreeVector axis)
{
  return G4ThreeVector_create(
   axis.x*This->rxx + axis.y*This->ryx + axis.z*This->rzx,
   axis.x*This->rxy + axis.y*This->ryy + axis.z*This->rzy,
   axis.x*This->rxz + axis.y*This->ryz + axis.z*This->rzz );
}
inline
G4AffineTransform G4AffineTransform_Inverse(const G4AffineTransform *This)
{
        return G4AffineTransform_create_elements(
    This->rxx, This->ryx, This->rzx,
    This->rxy, This->ryy, This->rzy,
    This->rxz, This->ryz, This->rzz,
    -This->tx*This->rxx - This->ty*This->rxy - This->tz*This->rxz,
    -This->tx*This->ryx - This->ty*This->ryy - This->tz*This->ryz,
    -This->tx*This->rzx - This->ty*This->rzy - This->tz*This->rzz );
}
inline
G4AffineTransform G4AffineTransform_Invert(G4AffineTransform *This)
{
        G4double v1 = -This->tx*This->rxx - This->ty*This->rxy - This->tz*This->rxz;
        G4double v2 = -This->tx*This->ryx - This->ty*This->ryy - This->tz*This->ryz;
        G4double v3 = -This->tx*This->rzx - This->ty*This->rzy - This->tz*This->rzz;
        This->tx=v1; This->ty=v2; This->tz=v3;
        G4double tmp1=This->ryx; This->ryx=This->rxy; This->rxy=tmp1;
        G4double tmp2=This->rzx; This->rzx=This->rxz; This->rxz=tmp2;
        G4double tmp3=This->rzy; This->rzy=This->ryz; This->ryz=tmp3;
        return *This;
}
inline
G4ThreeVector G4AffineTransform_NetTranslation(const G4AffineTransform *This)
{
        return G4ThreeVector_create(This->tx,This->ty,This->tz);
}
inline
G4bool G4AffineTransform_IsRotated(const G4AffineTransform *This)
{
        return (This->rxx==1.0 && This->ryy==1.0 && This->rzz==1.0) ? false : true;
}
typedef struct
{
 G4double property;
}
StubMaterial;
struct G4SmartVoxelProxy;
typedef struct
{
 G4double fmaxExtent;
 G4double fminExtent;
 struct G4SmartVoxelProxy* * fslices;
 G4int fNumSlices;
 G4int fminEquivalent;
 G4int fmaxEquivalent;
 EAxis faxis;
 EAxis fparamAxis;
}
G4SmartVoxelHeader;
typedef struct
{
 G4int *fcontents;
 G4int fminEquivalent;
 G4int fmaxEquivalent;
 G4int fNumContents;
}
G4SmartVoxelNode;
typedef struct G4SmartVoxelProxy
{
 G4SmartVoxelHeader* fHeader;
    G4SmartVoxelNode* fNode;
}
G4SmartVoxelProxy;
inline
void G4VoxelNode_ctor( G4SmartVoxelNode *This, G4int no )
{
 This->fmaxEquivalent = no;
 This->fminEquivalent = no;
 This->fcontents = __null;
 This->fNumContents = 0;
}
inline G4int
G4VoxelNode_GetNoContained( const G4SmartVoxelNode *This)
{
 return This->fNumContents;
}
inline G4int
G4VoxelNode_GetVolume(
 const G4SmartVoxelNode *This, G4int contentNo)
{
 ((contentNo >= 0 && contentNo < This->fNumContents) ? static_cast<void> (0) : __assert_fail ("contentNo >= 0 && contentNo < This->fNumContents", "G4Voxels.c", 35, __PRETTY_FUNCTION__));
 return This->fcontents[contentNo];
}
inline G4int
G4VoxelNode_GetMaxEquivalentSliceNo(
 const G4SmartVoxelNode *This )
{
 return This->fmaxEquivalent;
}
inline G4int
G4VoxelNode_GetMinEquivalentSliceNo(
 const G4SmartVoxelNode *This )
{
 return This->fminEquivalent;
}
inline G4int
G4VoxelHeader_GetMaxEquivalentSliceNo(
 const G4SmartVoxelHeader *This )
{
 return This->fmaxEquivalent;
}
inline G4int
G4VoxelHeader_GetMinEquivalentSliceNo(
 const G4SmartVoxelHeader *This )
{
 return This->fminEquivalent;
}
inline EAxis
G4VoxelHeader_GetAxis( const G4SmartVoxelHeader *This )
{
 return This->faxis;
}
inline G4int
G4VoxelHeader_GetNoSlices( const G4SmartVoxelHeader *This )
{
 return This->fNumSlices;
}
inline G4double
G4VoxelHeader_GetMinExtent( const G4SmartVoxelHeader *This )
{
 return This->fminExtent;
}
inline G4double
G4VoxelHeader_GetMaxExtent( const G4SmartVoxelHeader *This )
{
 return This->fmaxExtent;
}
inline G4SmartVoxelProxy*
G4VoxelHeader_GetSlice( const G4SmartVoxelHeader *This, G4int n )
{
 ((n >= 0 && n < This->fNumSlices) ? static_cast<void> (0) : __assert_fail ("n >= 0 && n < This->fNumSlices", "G4Voxels.c", 94, __PRETTY_FUNCTION__));
 return This->fslices[n];
}
inline G4bool
G4VoxelProxy_IsNode( const G4SmartVoxelProxy *This )
{
 return This->fNode != (( void*)0);
}
inline G4bool
G4VoxelProxy_IsHeader( const G4SmartVoxelProxy *This )
{
 return This->fHeader != (( void*)0);
}
inline G4SmartVoxelNode*
G4VoxelProxy_GetNode( const G4SmartVoxelProxy *This )
{
 return This->fNode;
}
inline G4SmartVoxelHeader*
G4VoxelProxy_GetHeader( const G4SmartVoxelProxy *This )
{
 return This->fHeader;
}
struct G4VPhysicalVolume;
struct G4VSolid;
typedef struct
{
 G4int fNoDaughters;
 struct G4VPhysicalVolume * *fDaughters;
   int check;
   StubMaterial* fMaterial;
 struct G4VSolid* fSolid;
 G4SmartVoxelHeader *fVoxel;
 int align;
}
G4LogicalVolume;
typedef struct G4VSolid
{
 ESolid type;
}
G4VSolid;
inline
EInside G4VSolid_Inside( const G4VSolid *This, G4ThreeVector p);
inline
G4ThreeVector G4VSolid_SurfaceNormal( const G4VSolid *This, G4ThreeVector p);
inline
G4double G4VSolid_DistanceToIn_full(
    const G4VSolid *This,
    G4ThreeVector p,
    G4ThreeVector v);
inline
G4double G4VSolid_DistanceToIn( const G4VSolid *This, G4ThreeVector p);
inline
G4double G4VSolid_DistanceToOut_full(
      const G4VSolid *This,
      G4ThreeVector p,
      G4ThreeVector v,
      const G4bool calcNorm,
      G4bool *validNorm,
      G4ThreeVector *n);
inline
G4double G4VSolid_DistanceToOut( const G4VSolid *This, G4ThreeVector p);
typedef struct
{
 G4VSolid solid;
    G4double fDx,fDy,fDz;
}
G4Box;
extern "C" {
inline EInside G4Box_Inside( const G4Box *This, G4ThreeVector p);
inline G4ThreeVector G4Box_SurfaceNormal( const G4Box *This, G4ThreeVector p);
inline G4double G4Box_DistanceToIn_full(
    const G4Box *This,
    G4ThreeVector p,
    G4ThreeVector v);
inline G4double G4Box_DistanceToIn( const G4Box *This, G4ThreeVector p);
inline G4double G4Box_DistanceToOut_full(
      const G4Box *This,
      G4ThreeVector p,
      G4ThreeVector v,
      const G4bool calcNorm,
      G4bool *validNorm,
      G4ThreeVector *n);
inline G4double G4Box_DistanceToOut( const G4Box *This, G4ThreeVector p);
inline
G4ThreeVector G4Box_ApproxSurfaceNormal( const G4Box *This, G4ThreeVector p )
{
  G4double distx, disty, distz ;
  G4ThreeVector norm ;
  distx = fabs(fabs(p.x) - This->fDx) ;
  disty = fabs(fabs(p.y) - This->fDy) ;
  distz = fabs(fabs(p.z) - This->fDz) ;
  if ( distx <= disty )
  {
    if ( distx <= distz )
    {
      if ( p.x < 0 ) norm = G4ThreeVector_create(-1.0,0,0) ;
      else norm = G4ThreeVector_create( 1.0,0,0) ;
    }
    else
    {
      if ( p.z < 0 ) norm = G4ThreeVector_create(0,0,-1.0) ;
      else norm = G4ThreeVector_create(0,0, 1.0) ;
    }
  }
  else
  {
    if ( disty <= distz )
    {
      if ( p.y < 0 ) norm = G4ThreeVector_create(0,-1.0,0) ;
      else norm = G4ThreeVector_create(0, 1.0,0) ;
    }
    else
    {
      if ( p.z < 0 ) norm = G4ThreeVector_create(0,0,-1.0) ;
      else norm = G4ThreeVector_create(0,0, 1.0) ;
    }
  }
  return norm;
}
inline
G4ThreeVector G4Box_SurfaceNormal( const G4Box *This, G4ThreeVector p)
{
  G4double distx, disty, distz ;
  G4ThreeVector norm ;
  const G4double kCarTolerance = 1E-3;
  distx = fabs(fabs(p.x) - This->fDx) ;
  disty = fabs(fabs(p.y) - This->fDy) ;
  distz = fabs(fabs(p.z) - This->fDz) ;
  const G4double delta = 0.5*kCarTolerance;
  const G4ThreeVector nX = G4ThreeVector_create( 1.0, 0,0 );
  const G4ThreeVector nmX = G4ThreeVector_create(-1.0, 0,0 );
  const G4ThreeVector nY = G4ThreeVector_create( 0, 1.0,0 );
  const G4ThreeVector nmY = G4ThreeVector_create( 0,-1.0,0 );
  const G4ThreeVector nZ = G4ThreeVector_create( 0, 0, 1.0);
  const G4ThreeVector nmZ = G4ThreeVector_create( 0, 0,- 1.0);
  G4ThreeVector
 normX = G4ThreeVector_create(0.,0.,0.),
 normY = G4ThreeVector_create(0.,0.,0.),
 normZ = G4ThreeVector_create(0.,0.,0.);
  G4ThreeVector sumnorm = G4ThreeVector_create(0., 0., 0.);
  G4int noSurfaces=0;
  if (distx <= delta)
  {
    noSurfaces ++;
    if ( p.x >= 0.){
      normX= nX ;
    }else{
      normX= nmX;
    }
    sumnorm= normX;
  }
  if (disty <= delta)
  {
    noSurfaces ++;
    if ( p.y >= 0.){
      normY= nY;
    }else{
      normY = nmY;
    }
    G4ThreeVector_sum_assign( &sumnorm, normY );
  }
  if (distz <= delta)
  {
    noSurfaces ++;
    if ( p.z >= 0.){
      normZ= nZ;
    }else{
      normZ = nmZ;
    }
    G4ThreeVector_sum_assign( &sumnorm, normZ );
  }
  const G4double invSqrt2 = 1.0 / sqrt( 2.0);
  const G4double invSqrt3 = 1.0 / sqrt( 3.0);
  norm= G4ThreeVector_create( 0., 0., 0.);
  if( noSurfaces > 0 )
  {
    if( noSurfaces == 1 ){
      norm= sumnorm;
    }else{
      if( noSurfaces == 2 ) {
        norm = G4ThreeVector_mult(sumnorm, invSqrt2);
      } else {
        norm = G4ThreeVector_mult(sumnorm, invSqrt3);
      }
    }
  }else{
     norm = G4Box_ApproxSurfaceNormal(This, p);
  }
  return norm;
}
inline
G4double G4Box_DistanceToIn_full( const G4Box *This, G4ThreeVector p,G4ThreeVector v)
{
  G4double safx, safy, safz ;
  G4double smin=0.0, sminy, sminz ;
  G4double smax=kInfinity, smaxy, smaxz ;
  G4double stmp ;
  G4double sOut=kInfinity, sOuty=kInfinity, sOutz=kInfinity ;
  const G4double kCarTolerance = 1E-3;
  safx = fabs(p.x) - This->fDx ;
  safy = fabs(p.y) - This->fDy ;
  safz = fabs(p.z) - This->fDz ;
  if ( ((p.x*v.x >= 0.0) && safx > -kCarTolerance*0.5)
       || ((p.y*v.y >= 0.0) && safy > -kCarTolerance*0.5)
       || ((p.z*v.z >= 0.0) && safz > -kCarTolerance*0.5) )
  {
    return kInfinity ;
  }
  if ( v.x)
  {
    stmp = 1.0/fabs(v.x) ;
    if (safx >= 0.0)
    {
      smin = safx*stmp ;
      smax = (This->fDx+fabs(p.x))*stmp ;
    }
    else
    {
      if (v.x > 0) sOut = (This->fDx - p.x)*stmp ;
      if (v.x < 0) sOut = (This->fDx + p.x)*stmp ;
    }
  }
  if ( v.y)
  {
    stmp = 1.0/fabs(v.y) ;
    if (safy >= 0.0)
    {
      sminy = safy*stmp ;
      smaxy = (This->fDy+fabs(p.y))*stmp ;
      if (sminy > smin) smin=sminy ;
      if (smaxy < smax) smax=smaxy ;
      if (smin >= smax-kCarTolerance*0.5)
      {
        return kInfinity ;
      }
    }
    else
    {
      if (v.y > 0) sOuty = (This->fDy - p.y)*stmp ;
      if (v.y < 0) sOuty = (This->fDy + p.y)*stmp ;
      if( sOuty < sOut ) sOut = sOuty ;
    }
  }
  if ( v.z )
  {
    stmp = 1.0/fabs(v.z) ;
    if ( safz >= 0.0)
    {
      sminz = safz*stmp ;
      smaxz = (This->fDz+fabs(p.z))*stmp ;
      if (sminz > smin) smin = sminz ;
      if (smaxz < smax) smax = smaxz ;
      if (smin >= smax-kCarTolerance*0.5)
      {
        return kInfinity ;
      }
    }
    else
    {
      if (v.z > 0) sOutz = (This->fDz - p.z)*stmp ;
      if (v.z < 0) sOutz = (This->fDz + p.z)*stmp ;
      if( sOutz < sOut ) sOut = sOutz ;
    }
  }
  if ( sOut <= smin + 0.5*kCarTolerance)
  {
    return kInfinity ;
  }
  if (smin < 0.5*kCarTolerance) smin = 0.0 ;
  return smin ;
}
inline
G4double G4Box_DistanceToIn( const G4Box *This, G4ThreeVector p)
{
  G4double safex, safey, safez, safe = 0.0 ;
  safex = fabs(p.x) - This->fDx ;
  safey = fabs(p.y) - This->fDy ;
  safez = fabs(p.z) - This->fDz ;
  if (safex > safe) safe = safex ;
  if (safey > safe) safe = safey ;
  if (safez > safe) safe = safez ;
  return safe ;
}
inline
G4double G4Box_DistanceToOut_full( const G4Box *This, G4ThreeVector p,G4ThreeVector v,
                               const G4bool calcNorm,
                                G4bool *validNorm,G4ThreeVector *n)
{
  const G4double kCarTolerance = 1E-3;
  enum {kBoxUndefined,kPX,kMX,kPY,kMY,kPZ,kMZ} side = kBoxUndefined ;
  G4double pdist,stmp,snxt;
  if (calcNorm) *validNorm = true ;
  if (v.x > 0)
  {
    pdist = This->fDx - p.x ;
    if (pdist > kCarTolerance*0.5)
    {
      snxt = pdist/v.x ;
      side = kPX ;
    }
    else
    {
      if (calcNorm) *n = G4ThreeVector_create(1,0,0) ;
      return snxt = 0 ;
    }
  }
  else if (v.x < 0)
  {
    pdist = This->fDx + p.x ;
    if (pdist > kCarTolerance*0.5)
    {
      snxt = -pdist/v.x ;
      side = kMX ;
    }
    else
    {
      if (calcNorm) *n = G4ThreeVector_create(-1,0,0) ;
      return snxt = 0 ;
    }
  }
  else snxt = kInfinity ;
  if ( v.y > 0 )
  {
    pdist=This->fDy-p.y;
    if (pdist>kCarTolerance*0.5)
    {
      stmp=pdist/v.y;
      if (stmp<snxt)
      {
        snxt=stmp;
        side=kPY;
      }
    }
    else
    {
      if (calcNorm) *n = G4ThreeVector_create(0,1,0) ;
      return snxt = 0 ;
    }
  }
  else if ( v.y < 0 )
  {
    pdist = This->fDy + p.y ;
    if (pdist > kCarTolerance*0.5)
    {
      stmp=-pdist/v.y;
      if (stmp<snxt)
      {
        snxt=stmp;
        side=kMY;
      }
    }
    else
    {
      if (calcNorm) *n = G4ThreeVector_create(0,-1,0) ;
      return snxt = 0 ;
    }
  }
  if (v.z>0)
  {
    pdist=This->fDz-p.z;
    if (pdist > kCarTolerance*0.5)
    {
      stmp=pdist/v.z;
      if (stmp < snxt)
      {
        snxt=stmp;
        side=kPZ;
      }
    }
    else
    {
      if (calcNorm) *n = G4ThreeVector_create(0,0,1) ;
      return snxt = 0 ;
    }
  }
  else if (v.z<0)
  {
    pdist = This->fDz + p.z ;
    if (pdist > kCarTolerance*0.5)
    {
      stmp=-pdist/v.z;
      if (stmp < snxt)
      {
        snxt=stmp;
        side=kMZ;
      }
    }
    else
    {
      if (calcNorm) *n = G4ThreeVector_create(0,0,-1) ;
      return snxt = 0 ;
    }
  }
  if (calcNorm)
  {
    switch (side)
    {
      case kPX:
        *n=G4ThreeVector_create(1,0,0);
        break;
      case kMX:
        *n=G4ThreeVector_create(-1,0,0);
        break;
      case kPY:
        *n=G4ThreeVector_create(0,1,0);
        break;
      case kMY:
        *n=G4ThreeVector_create(0,-1,0);
        break;
      case kPZ:
        *n=G4ThreeVector_create(0,0,1);
        break;
      case kMZ:
        *n=G4ThreeVector_create(0,0,-1);
        break;
      default:
        break;
    }
  }
  return snxt;
}
inline
G4double G4Box_DistanceToOut( const G4Box *This, G4ThreeVector p )
{
  G4double safx1,safx2,safy1,safy2,safz1,safz2,safe=0.0;
  safx1 = This->fDx - p.x ;
  safx2 = This->fDx + p.x ;
  safy1 = This->fDy - p.y ;
  safy2 = This->fDy + p.y ;
  safz1 = This->fDz - p.z ;
  safz2 = This->fDz + p.z ;
  if (safx2 < safx1) safe = safx2 ;
  else safe = safx1 ;
  if (safy1 < safe) safe = safy1 ;
  if (safy2 < safe) safe = safy2 ;
  if (safz1 < safe) safe = safz1 ;
  if (safz2 < safe) safe = safz2 ;
  if (safe < 0) safe = 0 ;
  return safe ;
}
inline
EInside G4Box_Inside( const G4Box *This, G4ThreeVector p)
{
  const G4double kCarTolerance = 1E-3;
  EInside in = kOutside ;
  if ( fabs(p.x) <= This->fDx - kCarTolerance*0.5 )
  {
    if (fabs(p.y) <= This->fDy - kCarTolerance*0.5 )
    {
      if (fabs(p.z) <= This->fDz - kCarTolerance*0.5 ) in = kInside ;
      else if (fabs(p.z) <= This->fDz + kCarTolerance*0.5 ) in = kSurface ;
    }
    else if (fabs(p.y) <= This->fDy + kCarTolerance*0.5 )
    {
      if (fabs(p.z) <= This->fDz + kCarTolerance*0.5 ) in = kSurface ;
    }
  }
  else if (fabs(p.x) <= This->fDx + kCarTolerance*0.5 )
  {
    if (fabs(p.y) <= This->fDy + kCarTolerance*0.5 )
    {
      if (fabs(p.z) <= This->fDz + kCarTolerance*0.5) in = kSurface ;
    }
  }
  return in ;
}
}
typedef struct
{
 G4VSolid solid;
    G4double fRmax;
    G4double fRmaxTolerance;
 G4double align;
}
G4Orb;
extern "C" {
inline EInside G4Orb_Inside( const G4Orb *This, G4ThreeVector p);
inline G4ThreeVector G4Orb_SurfaceNormal( const G4Orb *This, G4ThreeVector p);
inline G4double G4Orb_DistanceToIn_full(
    const G4Orb *This,
    G4ThreeVector p,
    G4ThreeVector v);
inline G4double G4Orb_DistanceToIn( const G4Orb *This, G4ThreeVector p);
inline G4double G4Orb_DistanceToOut_full(
      const G4Orb *This,
      G4ThreeVector p,
      G4ThreeVector v,
      const G4bool calcNorm,
      G4bool *validNorm,
      G4ThreeVector *n);
inline G4double G4Orb_DistanceToOut( const G4Orb *This, G4ThreeVector p);
inline
EInside G4Orb_Inside( const G4Orb *This, G4ThreeVector p)
{
  G4double rad2,tolRMax;
  EInside in;
  rad2 = G4ThreeVector_mag2(p);
  G4double rad = sqrt(rad2);
  tolRMax = This->fRmax - This->fRmaxTolerance*0.5 ;
  if ( rad <= tolRMax ) { in = kInside ; }
  else
  {
    tolRMax = This->fRmax + This->fRmaxTolerance*0.5 ;
    if ( rad <= tolRMax ) { in = kSurface ; }
    else { in = kOutside ; }
  }
  return in;
}
inline
G4ThreeVector G4Orb_SurfaceNormal( const G4Orb *This, G4ThreeVector p)
{
  (void)This;
  return G4ThreeVector_unit(p);
}
inline
G4double G4Orb_DistanceToIn_full( const G4Orb *This, G4ThreeVector p,G4ThreeVector v)
{
  G4double snxt = kInfinity ;
  G4double rad2, pDotV3d;
  G4double c, d2, s = kInfinity ;
  rad2 = G4ThreeVector_mag2(p);
  pDotV3d = G4ThreeVector_dot(p,v);
  G4double rad = sqrt(rad2);
  c = (rad - This->fRmax)*(rad + This->fRmax);
  if ( c > This->fRmaxTolerance*This->fRmax )
  {
    d2 = pDotV3d*pDotV3d - c ;
    if ( d2 >= 0 )
    {
      s = -pDotV3d - sqrt(d2) ;
      if ( s >= 0 )
      {
        return snxt = s;
      }
    }
    else
    {
      return snxt = kInfinity;
    }
  }
  else
  {
    if ( c > -This->fRmaxTolerance*This->fRmax )
    {
      d2 = pDotV3d*pDotV3d - c ;
      if ( (d2 < This->fRmaxTolerance*This->fRmax) || (pDotV3d >= 0) )
      {
        return snxt = kInfinity;
      }
      else
      {
        return snxt = 0.;
      }
    }
  }
  return snxt;
}
inline
G4double G4Orb_DistanceToIn( const G4Orb *This, G4ThreeVector p)
{
  G4double safe = 0.0,
           rad = G4ThreeVector_mag(p);
  safe = rad - This->fRmax;
  if( safe < 0 ) { safe = 0.; }
  return safe;
}
inline
G4double G4Orb_DistanceToOut_full( const G4Orb *This, G4ThreeVector p,G4ThreeVector v,
                               const G4bool calcNorm,
                                G4bool *validNorm,G4ThreeVector *n)
{
 G4double snxt = kInfinity;
  enum {kNull,kRMax} side = kNull;
  G4double rad2,pDotV3d;
  G4ThreeVector ipoint;
  G4double c,d2;
  rad2 = G4ThreeVector_mag2(p);
  pDotV3d = G4ThreeVector_dot(p,v);
  const G4double Rmax_plus = This->fRmax + This->fRmaxTolerance*0.5;
  G4double rad = sqrt(rad2);
  if ( rad <= Rmax_plus )
  {
    c = (rad - This->fRmax)*(rad + This->fRmax);
    if ( c < This->fRmaxTolerance*This->fRmax )
    {
      d2 = pDotV3d*pDotV3d - c;
      if( ( c > -This->fRmaxTolerance*This->fRmax) &&
          ( ( pDotV3d >= 0 ) || ( d2 < 0 )) )
      {
        if(calcNorm)
        {
          *validNorm = true ;
          *n = G4ThreeVector_create(p.x/This->fRmax,p.y/This->fRmax,p.z/This->fRmax) ;
        }
        return snxt = 0;
      }
      else
      {
        snxt = -pDotV3d + sqrt(d2);
        side = kRMax ;
      }
    }
  }
  else
  {
  }
  if (calcNorm)
  {
    switch( side )
    {
      case kRMax:
  ipoint = G4ThreeVector_saxpy(snxt,v,p);
  *n=G4ThreeVector_mult(ipoint,1.0/This->fRmax);
        *validNorm=true;
        break;
      default:
        break;
    }
  }
  return snxt;
}
inline
G4double G4Orb_DistanceToOut( const G4Orb *This, G4ThreeVector p )
{
   G4double safe=0.0,rad = G4ThreeVector_mag(p);
  safe = This->fRmax - rad;
  if ( safe < 0. ) safe = 0.;
  return safe;
}
}
inline
EInside G4VSolid_Inside( const G4VSolid *This, G4ThreeVector p)
{
 switch(This->type)
 {
  case kBox:
   return G4Box_Inside(( const G4Box*)This,p);
  case kOrb:
   return G4Orb_Inside(( const G4Orb*)This,p);
  default:
   ((false) ? static_cast<void> (0) : __assert_fail ("false", "G4VSolid_inline.c", 64, __PRETTY_FUNCTION__));
   return kOutside;
 }
}
inline
G4ThreeVector G4VSolid_SurfaceNormal( const G4VSolid *This, G4ThreeVector p)
{
 switch(This->type)
 {
  case kBox:
   return G4Box_SurfaceNormal(( const G4Box*)This,p);
  case kOrb:
   return G4Orb_SurfaceNormal(( const G4Orb*)This,p);
  default:
   ((false) ? static_cast<void> (0) : __assert_fail ("false", "G4VSolid_inline.c", 95, __PRETTY_FUNCTION__));
   return G4ThreeVector_create(0,0,0);
 }
}
inline
G4double G4VSolid_DistanceToIn_full(
    const G4VSolid *This,
    G4ThreeVector p,
    G4ThreeVector v)
{
 switch(This->type)
 {
  case kBox:
   return G4Box_DistanceToIn_full(( const G4Box*)This,p,v);
  case kOrb:
   return G4Orb_DistanceToIn_full(( const G4Orb*)This,p,v);
  default:
   ((false) ? static_cast<void> (0) : __assert_fail ("false", "G4VSolid_inline.c", 129, __PRETTY_FUNCTION__));
   return 0;
 }
}
inline
G4double G4VSolid_DistanceToIn( const G4VSolid *This, G4ThreeVector p)
{
 switch(This->type)
 {
  case kBox:
   return G4Box_DistanceToIn(( const G4Box*)This,p);
  case kOrb:
   return G4Orb_DistanceToIn(( const G4Orb*)This,p);
  default:
   ((false) ? static_cast<void> (0) : __assert_fail ("false", "G4VSolid_inline.c", 160, __PRETTY_FUNCTION__));
   return 0;
 }
}
inline
G4double G4VSolid_DistanceToOut_full(
      const G4VSolid *This,
      G4ThreeVector p,
      G4ThreeVector v,
      const G4bool calcNorm,
      G4bool *validNorm,
      G4ThreeVector *n)
{
 switch(This->type)
 {
  case kBox:
   return G4Box_DistanceToOut_full(( const G4Box*)This,p,v,calcNorm,validNorm,n);
  case kOrb:
   return G4Orb_DistanceToOut_full(( const G4Orb*)This,p,v,calcNorm,validNorm,n);
  default:
   ((false) ? static_cast<void> (0) : __assert_fail ("false", "G4VSolid_inline.c", 197, __PRETTY_FUNCTION__));
   return 0;
 }
}
inline
G4double G4VSolid_DistanceToOut( const G4VSolid *This, G4ThreeVector p)
{
 switch(This->type)
 {
  case kBox:
   return G4Box_DistanceToOut(( const G4Box*)This,p);
  case kOrb:
   return G4Orb_DistanceToOut(( const G4Orb*)This,p);
  default:
   ((false) ? static_cast<void> (0) : __assert_fail ("false", "G4VSolid_inline.c", 228, __PRETTY_FUNCTION__));
   return 0;
 }
}
inline
 G4SmartVoxelHeader * G4LogicalVolume_GetVoxelHeader( const G4LogicalVolume* This)
{
 return This->fVoxel;
}
inline
G4int G4LogicalVolume_GetNoDaughters( const G4LogicalVolume* This)
{
  return This->fNoDaughters;
}
inline
 struct G4VPhysicalVolume* G4LogicalVolume_GetDaughter( const G4LogicalVolume* This, const G4int i)
{
  return This->fDaughters[i];
}
inline
 struct G4VSolid* G4LogicalVolume_GetSolid( const G4LogicalVolume* This)
{
  return This->fSolid;
}
inline
 StubMaterial* G4LogicalVolume_GetMaterial( const G4LogicalVolume* This)
{
  return This->fMaterial;
}
typedef struct G4VPhysicalVolume
{
    G4RotationMatrix frot;
    G4ThreeVector ftrans;
 int guard1;
    G4LogicalVolume *flogical;
 int guard2;
 G4LogicalVolume *flmother;
 int guard3;
 int count;
 int counter_shadow;
}
G4VPhysicalVolume;
inline
G4ThreeVector G4VPhysicalVolume_GetTranslation( const G4VPhysicalVolume *This)
{
  return This->ftrans;
}
inline
 G4LogicalVolume* G4VPhysicalVolume_GetLogicalVolume( const G4VPhysicalVolume *This)
{
  return This->flogical;
}
inline
 G4LogicalVolume* G4VPhysicalVolume_GetMotherLogical( const G4VPhysicalVolume *This)
{
  return This->flmother;
}
inline
G4RotationMatrix G4VPhysicalVolume_GetObjectRotationValue( const G4VPhysicalVolume *This)
{
  return This->frot;
}
inline
G4ThreeVector G4VPhysicalVolume_GetObjectTranslation( const G4VPhysicalVolume *This)
{
 return This->ftrans;
}
typedef struct
{
   G4AffineTransform fTransform;
   G4VPhysicalVolume* fPhysicalVolumePtr;
   EVolume fVolumeType;
}
G4NavigationLevel;
typedef struct
{
 G4NavigationLevel fNavHistory[16];
 G4int fStackDepth;
 int align;
}
G4NavigationHistory;
inline
void G4NavigationLevel_ctor(
   G4NavigationLevel *This,
   G4VPhysicalVolume* pPhysVol,
   G4AffineTransform afTransform,
   EVolume volTp )
{
 This->fTransform = afTransform;
 This->fPhysicalVolumePtr = pPhysVol;
 This->fVolumeType = volTp;
}
inline
void G4NavigationLevel_ctor_relative(
   G4NavigationLevel *This,
   G4VPhysicalVolume* pPhysVol,
   G4AffineTransform levelAbove,
   G4AffineTransform relativeCurrent,
   EVolume volTp )
{
 This->fPhysicalVolumePtr = pPhysVol;
 This->fVolumeType = volTp;
 G4AffineTransform_InverseProduct(&(This->fTransform), &levelAbove, &relativeCurrent );
}
inline
G4NavigationLevel G4NavigationLevel_create(
   G4VPhysicalVolume* pPhysVol,
   G4AffineTransform afTransform,
   EVolume volTp )
{
 G4NavigationLevel lev;
 G4NavigationLevel_ctor( &lev, pPhysVol, afTransform, volTp );
 return lev;
}
inline
G4NavigationLevel G4NavigationLevel_create_relative(
 G4VPhysicalVolume* pPhysVol,
 G4AffineTransform levelAbove,
 G4AffineTransform relativeCurrent,
 EVolume volTp)
{
 G4NavigationLevel lev;
 G4NavigationLevel_ctor_relative( &lev, pPhysVol, levelAbove, relativeCurrent, volTp );
 return lev;
}
inline
 G4VPhysicalVolume* G4NavigationLevel_GetPhysicalVolume(
 const G4NavigationLevel *This )
{
  return This->fPhysicalVolumePtr;
}
inline
G4AffineTransform G4NavigationLevel_GetTransform(
 const G4NavigationLevel *This )
{
  return This->fTransform;
}
inline
const G4AffineTransform* G4NavigationLevel_GetPtrTransform(
 const G4NavigationLevel *This )
{
  return &(This->fTransform);
}
inline
EVolume G4NavigationLevel_GetVolumeType(
 const G4NavigationLevel *This )
{
  return This->fVolumeType;
}
inline
void G4NavigationHistory_Reset( G4NavigationHistory *This )
{
 This->fStackDepth = 0;
}
inline
void G4NavigationHistory_Clear( G4NavigationHistory *This )
{
  G4AffineTransform origin = G4AffineTransform_create_vector(G4ThreeVector_create(0.,0.,0.));
  G4NavigationLevel tmpNavLevel = G4NavigationLevel_create(0, origin, kNormal) ;
  G4NavigationHistory_Reset( This );
  for (G4int ilev=16 -1; ilev>=0; ilev--)
  {
     This->fNavHistory[ilev] = tmpNavLevel;
  }
}
inline
void G4NavigationHistory_ctor( G4NavigationHistory *This )
{
 This->fStackDepth = 0;
 G4NavigationHistory_Clear( This );
}
inline
void G4NavigationHistory_dtor( G4NavigationHistory *This )
{
 (void)This;
}
inline
void G4NavigationHistory_SetFirstEntry(
 G4NavigationHistory *This, G4VPhysicalVolume* pVol)
{
  G4ThreeVector translation = G4ThreeVector_create(0.,0.,0.);
  if( pVol!=(( void*)0) )
  {
    translation = G4VPhysicalVolume_GetTranslation( pVol );
  }
  This->fNavHistory[0] =
    G4NavigationLevel_create( pVol, G4AffineTransform_create_vector(translation), kNormal );
}
inline
const G4AffineTransform* G4NavigationHistory_GetPtrTopTransform(
 const G4NavigationHistory *This )
{
  return G4NavigationLevel_GetPtrTransform( &(This->fNavHistory[This->fStackDepth]) );
}
inline
G4AffineTransform G4NavigationHistory_GetTopTransform(
 const G4NavigationHistory *This )
{
  return G4NavigationLevel_GetTransform( &(This->fNavHistory[This->fStackDepth]) );
}
inline
EVolume G4NavigationHistory_GetTopVolumeType(
 const G4NavigationHistory *This )
{
  return G4NavigationLevel_GetVolumeType( &(This->fNavHistory[This->fStackDepth]) );
}
inline
 G4VPhysicalVolume* G4NavigationHistory_GetTopVolume(
 const G4NavigationHistory *This )
{
  return G4NavigationLevel_GetPhysicalVolume( &(This->fNavHistory[This->fStackDepth]) );
}
inline
G4int G4NavigationHistory_GetDepth(
 const G4NavigationHistory *This )
{
  return This->fStackDepth;
}
inline
G4AffineTransform
G4NavigationHistory_GetTransform(
 const G4NavigationHistory *This, G4int n )
{
  return G4NavigationLevel_GetTransform( &(This->fNavHistory[n]) );
}
inline
EVolume G4NavigationHistory_GetVolumeType(
 const G4NavigationHistory *This, G4int n )
{
  return G4NavigationLevel_GetVolumeType( &(This->fNavHistory[n]) );
}
inline
 G4VPhysicalVolume* G4NavigationHistory_GetVolume(
 const G4NavigationHistory *This, G4int n )
{
  return G4NavigationLevel_GetPhysicalVolume( &(This->fNavHistory[n]) );
}
inline
G4int G4NavigationHistory_GetMaxDepth(
 const G4NavigationHistory *This )
{
 (void)This;
 return 16;
}
inline
void G4NavigationHistory_BackLevel( G4NavigationHistory *This )
{
  ((This->fStackDepth>0) ? static_cast<void> (0) : __assert_fail ("This->fStackDepth>0", "G4NavigationHistory_inline.c", 207, __PRETTY_FUNCTION__));
  This->fStackDepth--;
}
inline
void G4NavigationHistory_NewLevel(
  G4NavigationHistory *This,
  G4VPhysicalVolume *pNewMother,
  EVolume vType )
{
  This->fStackDepth++;
  ((This->fStackDepth < 16) ? static_cast<void> (0) : __assert_fail ("This->fStackDepth < 16", "G4NavigationHistory_inline.c", 244, __PRETTY_FUNCTION__));
  This->fNavHistory[This->fStackDepth] =
    G4NavigationLevel_create_relative(
   pNewMother,
   G4NavigationLevel_GetTransform( &(This->fNavHistory[This->fStackDepth-1]) ),
   G4AffineTransform_create_full(
    G4VPhysicalVolume_GetObjectRotationValue( pNewMother ),
    G4VPhysicalVolume_GetTranslation( pNewMother )),
   vType );
}
typedef struct{
  G4VPhysicalVolume * PVolume;
  G4int trackId;
  }SolidInfo;
   typedef struct{
   float safety;
   float step;
   int trackId;
   G4VPhysicalVolume * PVolume;
   }ResultInfo;
   typedef struct{
   float safety;
   float step;
   G4VPhysicalVolume * PVolume;
   }
   FinalResult;
   typedef struct{
   G4ThreeVector Point;
   G4ThreeVector Direction;
   }PointInformation;
   void Find_minimum ( ResultInfo * Result_For_Current_Solid, FinalResult * Compacter_Result, int PrevSum, int size)
{
 int locationId = get_global_id(0);
 int i, loc ;
 float result_step, result_safety, Current_result_step, Current_result_safety;
 float Initial_result_step = (Compacter_Result [ locationId ]).step;
 float Initial_result_safety = (Compacter_Result [ locationId ]).safety;
 Current_result_step = Initial_result_step;
 Current_result_safety = Initial_result_safety;
 for( i = 0; i < size ; i++)
 {
  result_step = Result_For_Current_Solid[ PrevSum + i].step;
  result_safety = Result_For_Current_Solid[ PrevSum + i].safety;
  if ( result_step < Current_result_step)
  {
   loc = PrevSum + i;
   Current_result_step = result_step;
  }
  if ( result_safety < Current_result_safety)
  {
   Current_result_safety = result_safety;
  }
 }
 if( Current_result_step != Initial_result_step)
 {
  FinalResult final = { Current_result_safety, Current_result_step, (Result_For_Current_Solid[ loc ].PVolume)};
  Compacter_Result[ locationId ] = final;
 }
BARRIER_FLEXIBLE;
}
typedef struct
{
 G4double fVoxelSliceWidthStack[4];
 G4SmartVoxelHeader* fVoxelHeaderStack[4];
 G4int fVoxelNodeNoStack[4];
 G4int fVoxelNoSlicesStack[4];
 EAxis fVoxelAxisStack[4];
 G4int fVoxelDepth;
 G4SmartVoxelNode *fVoxelNode;
}
G4VoxelNavigation;
 void G4VoxelNavigation_ctor( G4VoxelNavigation *This );
 G4bool G4VoxelNavigation_LevelLocate(
 G4VoxelNavigation *This,
 G4NavigationHistory *history,
 const G4VPhysicalVolume *blockedVol,
 G4ThreeVector globalPoint,
 const G4ThreeVector* globalDirection,
 const G4bool pLocatedOnEdge,
 G4ThreeVector *localPoint );
 G4SmartVoxelNode* G4VoxelNavigation_VoxelLocate(
 G4VoxelNavigation *This,
 G4SmartVoxelHeader *voxelHeader,
 G4ThreeVector point);

G4double
G4VoxelNavigation_ComputeStep(
   G4VoxelNavigation *This,
   G4ThreeVector localPoint,
   G4ThreeVector localDirection,
   const G4double currentProposedStepLength,
   G4double *newSafety,
   G4NavigationHistory *history,
   G4bool *validExitNormal,
   G4ThreeVector *exitNormal,
   G4bool *exiting,
   G4bool *entering,
   G4VPhysicalVolume *(*pBlockedPhysical)
   , G4double * Result
   );
 G4double G4VoxelNavigation_ComputeSafety(
 G4VoxelNavigation *This,
 G4ThreeVector localPoint,
 const G4NavigationHistory *history);
typedef struct
{
 G4NavigationHistory fHistory;
 G4VoxelNavigation fVoxelNav;
 G4ThreeVector fStepEndPoint;
 G4ThreeVector fLastLocatedPointLocal;
 G4ThreeVector fExitNormal;
 G4ThreeVector fGrandMotherExitNormal;
 G4bool fEnteredDaughter;
 G4bool fExitedMother;
 G4bool fWasLimitedByGeometry;
 G4bool fEntering;
 G4bool fExiting;
 G4bool fLastStepWasZero;
 G4bool fLocatedOnEdge;
 G4bool fLocatedOutsideWorld;
 G4bool fValidExitNormal;
 G4bool fPushed;
 G4int fNumberZeroSteps;
 int align1;
 G4double fPreviousSafety;
 G4VPhysicalVolume *fBlockedPhysicalVolume;
 G4VPhysicalVolume *fTopPhysical;
}
G4Navigator;
 void G4Navigator_ctor( G4Navigator *This );
 void G4Navigator_SetWorldVolume(
 G4Navigator *This,
 G4VPhysicalVolume* pWorld );
 G4VPhysicalVolume* G4Navigator_LocateGlobalPointAndSetup(
  G4Navigator *This,
  G4ThreeVector globalPoint,
  const G4ThreeVector* pGlobalDirection,
  G4bool relativeSearch,
  G4bool ignoreDirection,
  float * Result);

G4double G4Navigator_ComputeStep(
  G4Navigator *This,
  G4ThreeVector pGlobalpoint,
  G4ThreeVector pDirection,
  const G4double pCurrentProposedStepLength,
  G4double *pNewSafety
   , G4bool cur_vol_local
   , G4double * Result
  );
 void G4Navigator_SetGeometricallyLimitedStep( G4Navigator *This );
 G4double G4NormalNavigation_ComputeStep(
 G4ThreeVector localPoint,
 G4ThreeVector localDirection,
 const G4double currentProposedStepLength,
 G4double *newSafety,
 G4NavigationHistory *history,
 G4bool *validExitNormal,
 G4ThreeVector *exitNormal,
 G4bool *exiting,
 G4bool *entering,
 G4VPhysicalVolume *(*pBlockedPhysical));
 G4double G4NormalNavigation_ComputeSafety(
 G4ThreeVector localPoint,
 const G4NavigationHistory *history );
 G4bool G4NormalNavigation_LevelLocate(
 G4NavigationHistory *history,
 const G4VPhysicalVolume *blockedVol,
 G4ThreeVector* globalPoint,
 const G4ThreeVector* globalDirection,
 G4bool pLocatedOnEdge,
 G4ThreeVector* localPoint );
  void G4VoxelNavigation_ctor( G4VoxelNavigation *This );
 void G4Navigator_ResetState( G4Navigator *This )
{
  This->fWasLimitedByGeometry = false;
  This->fEntering = false;
  This->fExiting = false;
  This->fLocatedOnEdge = false;
  This->fLastStepWasZero = false;
  This->fEnteredDaughter = false;
  This->fExitedMother = false;
  This->fPushed = false;
  This->fValidExitNormal = false;
  This->fExitNormal = G4ThreeVector_create(0,0,0);
  This->fPreviousSafety = 0.0;
  This->fNumberZeroSteps = 0;
  This->fBlockedPhysicalVolume = (( void*)0);
  This->fLastLocatedPointLocal = G4ThreeVector_create( 1e37, -1e37, 0.0 );
  This->fLocatedOutsideWorld = false;
}
inline
G4ThreeVector G4Navigator_ComputeLocalAxis( const G4Navigator *This, G4ThreeVector pVec)
{
 G4AffineTransform t =
  G4NavigationHistory_GetTopTransform( &(This->fHistory) );
 return G4AffineTransform_TransformAxis(&t, pVec);
}
inline G4ThreeVector
G4Navigator_ComputeLocalPoint( const G4Navigator *This, G4ThreeVector pGlobalPoint)
{
 G4AffineTransform t =
  G4NavigationHistory_GetTopTransform( &(This->fHistory) );
 return G4AffineTransform_TransformPoint(&t, pGlobalPoint);
}
 void G4Navigator_SetWorldVolume( G4Navigator *This, G4VPhysicalVolume* pWorld )
{
 This->fTopPhysical = pWorld;
 G4NavigationHistory_SetFirstEntry( &(This->fHistory), pWorld );
}
 void G4Navigator_SetGeometricallyLimitedStep( G4Navigator *This )
{
 This->fWasLimitedByGeometry = true;
}
inline
void G4Navigator_ResetStackAndState( G4Navigator *This )
{
 G4NavigationHistory_Reset( &(This->fHistory) );
 G4Navigator_ResetState( This );
}
inline
EVolume G4Navigator_VolumeType( const G4Navigator *This, const G4VPhysicalVolume *pVol )
{
 (void)This;
 (void)pVol;
 return kNormal;
}
 void G4Navigator_ctor( G4Navigator *This )
{
 G4NavigationHistory_ctor( &(This->fHistory) );
 G4VoxelNavigation_ctor( &(This->fVoxelNav ) );
 G4Navigator_ResetStackAndState( This );
 This->fWasLimitedByGeometry = false;
 This->fTopPhysical = (( void*)0);
 This->fPushed = false;
 This->fStepEndPoint = G4ThreeVector_create( kInfinity, kInfinity, kInfinity );
}

 G4VPhysicalVolume*
G4Navigator_LocateGlobalPointAndSetup(
  G4Navigator *This,
  G4ThreeVector globalPoint,
  const G4ThreeVector* pGlobalDirection,
  G4bool relativeSearch,
  G4bool ignoreDirection,
  float * Result
  )
{
  G4bool notKnownContained=true, noResult;
  G4VPhysicalVolume *targetPhysical;
  G4VSolid *targetSolid = (( void*)0);
  G4ThreeVector localPoint = G4ThreeVector_create(0,0,0);
  G4ThreeVector globalDirection = G4ThreeVector_create(0,0,0);
  EInside insideCode;
  G4bool considerDirection = (!ignoreDirection) || This->fLocatedOnEdge;
  if( considerDirection && pGlobalDirection != 0 )
  {
    globalDirection=*pGlobalDirection;
  }
  if ( 1 )
  {
     G4Navigator_ResetStackAndState( This );
  }
  else
  {
    if ( This->fWasLimitedByGeometry )
    {
      This->fWasLimitedByGeometry = false;
      This->fEnteredDaughter = This->fEntering;
      This->fExitedMother = This->fExiting;
      if ( This->fExiting )
      {
        if ( G4NavigationHistory_GetDepth( &(This->fHistory) ) )
        {
          This->fBlockedPhysicalVolume = G4NavigationHistory_GetTopVolume( &(This->fHistory) );
          G4NavigationHistory_BackLevel( &(This->fHistory) );
        }
        else
        {
          This->fLastLocatedPointLocal = localPoint;
          This->fLocatedOutsideWorld = true;
          return (( void*)0);
        }
        if ( This->fLocatedOnEdge )
        {
          This->fExiting= false;
        }
      }
      else
        if ( This->fEntering )
        {
    G4NavigationHistory_NewLevel( &(This->fHistory), This->fBlockedPhysicalVolume, kNormal);
          This->fEntering = false;
          This->fBlockedPhysicalVolume = (( void*)0);
          G4AffineTransform t = G4NavigationHistory_GetTopTransform( &(This->fHistory) );
          localPoint = G4AffineTransform_TransformPoint(&t,globalPoint);
          notKnownContained = false;
        }
    }
    else
    {
      This->fBlockedPhysicalVolume = (( void*)0);
      This->fEntering = false;
      This->fEnteredDaughter = false;
      This->fExiting = false;
      This->fExitedMother = false;
    }
  }
  while (notKnownContained)
  {
 targetSolid =
   G4LogicalVolume_GetSolid(
    G4VPhysicalVolume_GetLogicalVolume(
    G4NavigationHistory_GetTopVolume(&(This->fHistory))));
 G4AffineTransform t = G4NavigationHistory_GetTopTransform( &(This->fHistory) );
 localPoint = G4AffineTransform_TransformPoint(&t,globalPoint);
 insideCode = G4VSolid_Inside(targetSolid,localPoint);
    if ( insideCode==kOutside )
    {
      if ( G4NavigationHistory_GetDepth( &(This->fHistory) ) )
      {
        This->fBlockedPhysicalVolume = G4NavigationHistory_GetTopVolume( &(This->fHistory) );
        G4NavigationHistory_BackLevel( &(This->fHistory) );
        This->fExiting = false;
      }
      else
      {
        This->fLastLocatedPointLocal = localPoint;
        This->fLocatedOutsideWorld = true;
        return (( void*)0);
      }
    }
    else
      if ( insideCode==kSurface )
      {
        G4bool isExiting = This->fExiting;
        if( (!This->fExiting)&&considerDirection )
        {
   G4bool directionExiting = false;
   G4AffineTransform t = G4NavigationHistory_GetTopTransform( &(This->fHistory) );
   G4ThreeVector localDirection =G4AffineTransform_TransformAxis(&t,globalDirection);
   G4ThreeVector normal = G4VSolid_SurfaceNormal(targetSolid, localPoint);
   directionExiting = G4ThreeVector_dot(normal,localDirection) > 0.0;
   isExiting = isExiting || directionExiting;
        }
        if( isExiting )
        {
          if ( G4NavigationHistory_GetDepth( &(This->fHistory) ) )
          {
            This->fBlockedPhysicalVolume = G4NavigationHistory_GetTopVolume( &(This->fHistory) );
            G4NavigationHistory_BackLevel( &(This->fHistory) );
            This->fValidExitNormal = false;
          }
          else
          {
            This->fLastLocatedPointLocal = localPoint;
            This->fLocatedOutsideWorld = true;
            return (( void*)0);
          }
        }
        else
        {
          notKnownContained=false;
        }
      }
      else
      {
        notKnownContained=false;
      }
  }
  noResult = true;
  do
  {
    targetPhysical = G4NavigationHistory_GetTopVolume( &(This->fHistory) );
  G4LogicalVolume *targetLogical = G4VPhysicalVolume_GetLogicalVolume(targetPhysical);
    if ( G4LogicalVolume_GetVoxelHeader( targetLogical ) != (( void*)0) )
    {
  noResult =
   G4VoxelNavigation_LevelLocate(
    &(This->fVoxelNav),
    &(This->fHistory),
    This->fBlockedPhysicalVolume,
    globalPoint,
    pGlobalDirection,
    considerDirection,
    &localPoint);
 }
 else
 {
  noResult = G4NormalNavigation_LevelLocate(
    &(This->fHistory),
    This->fBlockedPhysicalVolume,
    &globalPoint,
    pGlobalDirection,
    considerDirection,
    &localPoint);
 }
    if ( noResult )
    {
      This->fBlockedPhysicalVolume = (( void*)0);
      This->fEntering = false;
      This->fEnteredDaughter = true;
    }
  } while (noResult);
  This->fLastLocatedPointLocal = localPoint;
  This->fLocatedOutsideWorld= false;
  return targetPhysical;
}
 void
G4Navigator_LocateGlobalPointWithinVolume( G4Navigator *This, G4ThreeVector pGlobalpoint)
{
 This->fLastLocatedPointLocal = G4Navigator_ComputeLocalPoint( This, pGlobalpoint );
 G4VPhysicalVolume* motherPhysical = G4NavigationHistory_GetTopVolume( &(This->fHistory) );
 G4LogicalVolume* motherLogical = G4VPhysicalVolume_GetLogicalVolume( motherPhysical );
 G4SmartVoxelHeader* pVoxelHeader = G4LogicalVolume_GetVoxelHeader( motherLogical );
 if ( pVoxelHeader )
 {
  G4VoxelNavigation_VoxelLocate( &(This->fVoxelNav), pVoxelHeader, This->fLastLocatedPointLocal );
 }
 This->fBlockedPhysicalVolume = (( void*)0);
 This->fEntering = false;
 This->fEnteredDaughter = false;
 This->fExiting = false;
 This->fExitedMother = false;
}

G4double G4Navigator_ComputeStep(
  G4Navigator *This,
  G4ThreeVector pGlobalpoint,
  G4ThreeVector pDirection,
  const G4double pCurrentProposedStepLength,
  G4double *pNewSafety
   , G4bool cur_vol_local
   , G4double * Result
  )
{
  G4ThreeVector localDirection = G4Navigator_ComputeLocalAxis(This,pDirection);
  G4double Step = 1e37;
  G4VPhysicalVolume *motherPhysical = G4NavigationHistory_GetTopVolume( &(This->fHistory) );
  const G4double kCarTolerance = 1E-3;
  G4LogicalVolume *motherLogical = G4VPhysicalVolume_GetLogicalVolume(motherPhysical);
  G4ThreeVector newLocalPoint = G4Navigator_ComputeLocalPoint( This, pGlobalpoint);
  if( !G4ThreeVector_equal(newLocalPoint, This->fLastLocatedPointLocal) )
  {
    G4ThreeVector oldLocalPoint = This->fLastLocatedPointLocal;
    G4double moveLenSq = G4ThreeVector_diff2(newLocalPoint,oldLocalPoint);
    if ( moveLenSq >= kCarTolerance*kCarTolerance )
    {
      G4Navigator_LocateGlobalPointWithinVolume( This, pGlobalpoint );
    }
  }
  if ( G4LogicalVolume_GetVoxelHeader(motherLogical) != (( void*)0) )
  {
 if( cur_vol_local )
 Step = G4VoxelNavigation_ComputeStep(
   &(This->fVoxelNav),
   This->fLastLocatedPointLocal,
   localDirection,
   pCurrentProposedStepLength,
   pNewSafety,
   &(This->fHistory),
   &(This->fValidExitNormal),
   &(This->fExitNormal),
   &(This->fExiting),
   &(This->fEntering),
   &(This->fBlockedPhysicalVolume)
   , Result
   );
 else
     return 0;
  }
  else
  {
 Step = G4NormalNavigation_ComputeStep(
   This->fLastLocatedPointLocal,
   localDirection,
   pCurrentProposedStepLength,
   pNewSafety,
   &(This->fHistory),
   &(This->fValidExitNormal),
   &(This->fExitNormal),
   &(This->fExiting),
   &(This->fEntering),
   &(This->fBlockedPhysicalVolume));
  }
  This->fPreviousSafety = *pNewSafety;
  This->fLocatedOnEdge = This->fLastStepWasZero && (Step==0.0);
  This->fLastStepWasZero = (Step==0.0);
  if (This->fPushed) This->fPushed = This->fLastStepWasZero;
  if ( This->fLastStepWasZero )
  {
    This->fNumberZeroSteps++;
    if( This->fNumberZeroSteps > 10 -1 )
    {
       Step += 0.9*kCarTolerance;
       This->fPushed = true;
    }
    if( This->fNumberZeroSteps > 25 -1 )
    {
   abort();
    }
  }
  else
  {
    if (!This->fPushed) This->fNumberZeroSteps = 0;
  }
  This->fEnteredDaughter = This->fEntering;
  This->fExitedMother = This->fExiting;
  if( This->fExiting )
  {
    if(This->fValidExitNormal)
    {
      This->fGrandMotherExitNormal= This->fExitNormal;
    }
    else
    {
      G4ThreeVector finalLocalPoint =
  G4ThreeVector_saxpy( Step, localDirection, This->fLastLocatedPointLocal );
      This->fGrandMotherExitNormal =
  G4VSolid_SurfaceNormal(
   G4LogicalVolume_GetSolid(motherLogical),finalLocalPoint);
      G4RotationMatrix mRot = G4VPhysicalVolume_GetObjectRotationValue(motherPhysical);
      G4RotationMatrix inv = G4RotationMatrix_inverse(&mRot);
      This->fGrandMotherExitNormal
       = G4RotationMatrix_apply(&inv,This->fGrandMotherExitNormal);
    }
  }
  This->fStepEndPoint =
 G4ThreeVector_saxpy(Step, pDirection, pGlobalpoint );
  if( (Step == pCurrentProposedStepLength) && (!This->fExiting) && (!This->fEntering) )
  {
    Step = kInfinity;
  }
  return Step;
}
inline G4bool
G4AuxiliaryNavServices_CheckPointOnSurface(
         const G4VSolid* sampleSolid,
                     G4ThreeVector localPoint,
                     const G4ThreeVector* globalDirection,
                     G4AffineTransform sampleTransform,
                     const G4bool locatedOnEdge)
{
  G4ThreeVector localDirection, sampleNormal;
  G4bool enter = false;
  EInside insideSolid =
 G4VSolid_Inside(sampleSolid, localPoint);
  if ( insideSolid!=kOutside )
  {
    G4bool checkDirection= locatedOnEdge && (globalDirection!=0);
    if( (insideSolid==kSurface) && checkDirection)
    {
      localDirection= G4AffineTransform_TransformAxis(&sampleTransform,*globalDirection);
      sampleNormal = G4VSolid_SurfaceNormal(sampleSolid,localPoint);
      if ( G4ThreeVector_dot(sampleNormal,localDirection) <= 0 )
      {
        if( G4ThreeVector_dot(sampleNormal,localDirection) == 0 )
        {
          G4double distanceToIn =
   G4VSolid_DistanceToIn_full( sampleSolid, localPoint, localDirection );
          if( distanceToIn != kInfinity )
          {
            enter = true;
          }
        }
        else
        {
          enter = true;
        }
      }
    }
    else
    {
      enter = true;
    }
  }
  return enter;
}
 G4bool
G4NormalNavigation_LevelLocate(
 G4NavigationHistory *history,
 const G4VPhysicalVolume *blockedVol,
 G4ThreeVector* globalPoint,
 const G4ThreeVector* globalDirection,
 G4bool pLocatedOnEdge,
 G4ThreeVector* localPoint )
{
  G4VPhysicalVolume *targetPhysical, *samplePhysical;
  G4LogicalVolume *targetLogical;
  G4VSolid *sampleSolid;
  G4ThreeVector samplePoint;
  G4int targetNoDaughters;
  targetPhysical = G4NavigationHistory_GetTopVolume(history);
  targetLogical = G4VPhysicalVolume_GetLogicalVolume(targetPhysical);
  targetNoDaughters = G4LogicalVolume_GetNoDaughters(targetLogical);
  if (targetNoDaughters == 0) return false;
  for ( int sampleNo=targetNoDaughters-1; sampleNo>=0; sampleNo-- )
  {
   samplePhysical =
  G4LogicalVolume_GetDaughter(targetLogical,sampleNo);
   if ( samplePhysical!=blockedVol )
   {
  G4NavigationHistory_NewLevel(history, samplePhysical, kNormal );
  sampleSolid =
   G4LogicalVolume_GetSolid(
    G4VPhysicalVolume_GetLogicalVolume(samplePhysical));
  G4AffineTransform tf =
   G4NavigationHistory_GetTopTransform(history);
  samplePoint =
   G4AffineTransform_TransformPoint( &tf, *globalPoint );
  if( G4AuxiliaryNavServices_CheckPointOnSurface(
   sampleSolid, samplePoint, globalDirection,
   tf, pLocatedOnEdge) )
  {
    *localPoint = samplePoint;
    return true;
  }
  else
  {
   G4NavigationHistory_BackLevel(history);
  }
   }
  }
  return false;
}

G4double
G4NormalNavigation_ComputeStep(
 G4ThreeVector localPoint,
 G4ThreeVector localDirection,
 const G4double currentProposedStepLength,
 G4double *newSafety,
 G4NavigationHistory *history,
 G4bool *validExitNormal,
 G4ThreeVector *exitNormal,
 G4bool *exiting,
 G4bool *entering,
 G4VPhysicalVolume *(*pBlockedPhysical))
{
  G4VPhysicalVolume *motherPhysical, *samplePhysical, *blockedExitedVol=0;
  G4LogicalVolume *motherLogical;
  G4VSolid *motherSolid;
  G4ThreeVector sampleDirection;
  G4double ourStep=currentProposedStepLength, motherSafety, ourSafety;
  G4int localNoDaughters, sampleNo;
  motherPhysical = G4NavigationHistory_GetTopVolume(history);
  motherLogical = G4VPhysicalVolume_GetLogicalVolume(motherPhysical);
  motherSolid = G4LogicalVolume_GetSolid(motherLogical);
  motherSafety = G4VSolid_DistanceToOut(motherSolid,localPoint);
  ourSafety = motherSafety;
  if ( *exiting && *validExitNormal )
  {
    if ( G4ThreeVector_dot(localDirection,*exitNormal)>=kMinExitingNormalCosine )
    {
      blockedExitedVol =* pBlockedPhysical;
      ourSafety = 0;
    }
  }
  *exiting = false;
  *entering = false;
  localNoDaughters = G4LogicalVolume_GetNoDaughters(motherLogical);
  for ( sampleNo=localNoDaughters-1; sampleNo>=0; sampleNo--)
  {
    samplePhysical = G4LogicalVolume_GetDaughter(motherLogical,sampleNo);
    if ( samplePhysical!=blockedExitedVol )
    {
      G4AffineTransform sampleTf =
       G4AffineTransform_create_full(
  G4VPhysicalVolume_GetObjectRotationValue(samplePhysical),
  G4VPhysicalVolume_GetTranslation(samplePhysical));
   G4AffineTransform_Invert(&sampleTf);
      const G4ThreeVector samplePoint =
   G4AffineTransform_TransformPoint(&sampleTf, localPoint);
      const G4VSolid *sampleSolid =
  G4LogicalVolume_GetSolid(
   G4VPhysicalVolume_GetLogicalVolume( samplePhysical ));
      const G4double sampleSafety =
  G4VSolid_DistanceToIn(sampleSolid,samplePoint);
      if ( sampleSafety<ourSafety )
      {
        ourSafety=sampleSafety;
      }
      if ( sampleSafety<=ourStep )
      {
        sampleDirection = G4AffineTransform_TransformAxis(&sampleTf, localDirection);
        const G4double sampleStep =
   G4VSolid_DistanceToIn_full(sampleSolid,samplePoint,sampleDirection);
        if ( sampleStep<=ourStep )
        {
          ourStep = sampleStep;
          *entering = true;
          *exiting = false;
          *pBlockedPhysical = samplePhysical;
        }
      }
    }
  }
  if ( currentProposedStepLength<ourSafety )
  {
    *entering = false;
    *exiting = false;
    *pBlockedPhysical = (( void*)0);
    ourStep = kInfinity;
  }
  else
  {
    if ( motherSafety<=ourStep )
    {
      G4double motherStep =
  G4VSolid_DistanceToOut_full(
   motherSolid,
   localPoint,
   localDirection,
   true,
   validExitNormal,
   exitNormal);
      if ( motherStep<=ourStep )
      {
        ourStep = motherStep;
        *exiting = true;
        *entering = false;
        if ( *validExitNormal )
        {
          G4RotationMatrix rot = G4VPhysicalVolume_GetObjectRotationValue(motherPhysical);
    G4RotationMatrix inv = G4RotationMatrix_inverse(&rot);
          *exitNormal = G4RotationMatrix_apply(&inv, *exitNormal);
        }
      }
      else
      {
        *validExitNormal = false;
      }
    }
  }
  *newSafety = ourSafety;
  return ourStep;
}
inline G4bool
G4AuxiliaryNavServices_CheckPointOnSurface(
         const G4VSolid* sampleSolid,
                     G4ThreeVector localPoint,
                     const G4ThreeVector* globalDirection,
                     G4AffineTransform sampleTransform,
                     const G4bool locatedOnEdge);
inline G4bool
G4AuxiliaryNavServices_CheckPointExiting(
       const G4VSolid* sampleSolid,
                   G4ThreeVector localPoint,
                   const G4ThreeVector* globalDirection,
                   G4AffineTransform sampleTransform );
inline
 G4SmartVoxelNode*
G4VoxelNavigation_VoxelLocate(
   G4VoxelNavigation *This,
   G4SmartVoxelHeader* pHead,
   G4ThreeVector localPoint )
{
  G4SmartVoxelHeader *targetVoxelHeader=pHead;
  G4SmartVoxelNode *targetVoxelNode = (( void*)0);
  const G4SmartVoxelProxy *sampleProxy;
  EAxis targetHeaderAxis;
  G4double targetHeaderMin, targetHeaderNodeWidth;
  G4int targetHeaderNoSlices, targetNodeNo;
  This->fVoxelDepth = 0;
  while ( targetVoxelNode == (( void*)0) )
  {
    targetHeaderAxis = G4VoxelHeader_GetAxis(targetVoxelHeader);
    targetHeaderNoSlices = G4VoxelHeader_GetNoSlices(targetVoxelHeader);
    targetHeaderMin = G4VoxelHeader_GetMinExtent(targetVoxelHeader);
    targetHeaderNodeWidth =
  (G4VoxelHeader_GetMaxExtent(targetVoxelHeader)-targetHeaderMin)
                          / targetHeaderNoSlices;
    targetNodeNo = (G4int)(
  (G4ThreeVector_coord(localPoint,targetHeaderAxis)-targetHeaderMin)
                          / targetHeaderNodeWidth);
    if ( targetNodeNo<0 )
    {
  targetNodeNo = 0;
    }
    else if ( targetNodeNo>=targetHeaderNoSlices )
 {
  targetNodeNo = targetHeaderNoSlices-1;
 }
    This->fVoxelAxisStack[This->fVoxelDepth] = targetHeaderAxis;
    This->fVoxelNoSlicesStack[This->fVoxelDepth] = targetHeaderNoSlices;
    This->fVoxelSliceWidthStack[This->fVoxelDepth] = targetHeaderNodeWidth;
    This->fVoxelNodeNoStack[This->fVoxelDepth] = targetNodeNo;
    This->fVoxelHeaderStack[This->fVoxelDepth] = targetVoxelHeader;
    sampleProxy = G4VoxelHeader_GetSlice(targetVoxelHeader, targetNodeNo);
    if ( G4VoxelProxy_IsNode(sampleProxy) )
    {
      targetVoxelNode = G4VoxelProxy_GetNode(sampleProxy);
    }
    else
    {
      targetVoxelHeader = G4VoxelProxy_GetHeader(sampleProxy);
      This->fVoxelDepth++;
      ((This->fVoxelDepth < 4) ? static_cast<void> (0) : __assert_fail ("This->fVoxelDepth < 4", "G4VoxelNavigation.c", 91, __PRETTY_FUNCTION__));
    }
  }
  This->fVoxelNode = targetVoxelNode;
  return targetVoxelNode;
}

G4bool
G4VoxelNavigation_LocateNextVoxel(
   G4VoxelNavigation *This,
   G4ThreeVector localPoint,
   G4ThreeVector localDirection,
   const G4double currentStep )
{
  G4SmartVoxelHeader *workHeader=(( void*)0), *newHeader=(( void*)0);
  G4SmartVoxelProxy *newProxy=(( void*)0);
  G4SmartVoxelNode *newVoxelNode= (( void*)0);
  G4ThreeVector targetPoint, voxelPoint;
  G4double workNodeWidth, workMinExtent, workCoord;
  G4double minVal, maxVal, newDistance=0.;
  G4double newHeaderMin, newHeaderNodeWidth;
  G4int depth=0, newDepth=0, workNodeNo=0, newNodeNo=0, newHeaderNoSlices=0;
  EAxis workHeaderAxis, newHeaderAxis;
  G4bool isNewVoxel=false;
  G4double currentDistance = currentStep;
  for (depth=0; depth<This->fVoxelDepth; depth++)
  {
    targetPoint =
  G4ThreeVector_saxpy(currentDistance,localDirection,localPoint);
    newDistance = currentDistance;
    workHeader = This->fVoxelHeaderStack[depth];
    workHeaderAxis = This->fVoxelAxisStack[depth];
    workNodeNo = This->fVoxelNodeNoStack[depth];
    workNodeWidth = This->fVoxelSliceWidthStack[depth];
    workMinExtent = G4VoxelHeader_GetMinExtent(workHeader);
    workCoord = G4ThreeVector_coord(targetPoint,workHeaderAxis);
    minVal = workMinExtent+workNodeNo*workNodeWidth;
    if ( minVal<=workCoord+1E-3*0.5 )
    {
      maxVal = minVal+workNodeWidth;
      if ( maxVal<=workCoord-1E-3*0.5 )
      {
        newNodeNo = workNodeNo+1;
        newHeader = workHeader;
        newDistance = (maxVal-G4ThreeVector_coord(localPoint,workHeaderAxis))
                    / G4ThreeVector_coord(localDirection,workHeaderAxis);
        isNewVoxel = true;
        newDepth = depth;
      }
    }
    else
    {
      newNodeNo = workNodeNo-1;
      newHeader = workHeader;
      newDistance = (minVal-G4ThreeVector_coord(localPoint,workHeaderAxis))
                  / G4ThreeVector_coord(localDirection,workHeaderAxis);
      isNewVoxel = true;
      newDepth = depth;
    }
    currentDistance = newDistance;
  }
  targetPoint =
 G4ThreeVector_saxpy(currentDistance,localDirection,localPoint);
  depth = This->fVoxelDepth;
  {
    workHeader = This->fVoxelHeaderStack[depth];
    workHeaderAxis = This->fVoxelAxisStack[depth];
    workNodeNo = This->fVoxelNodeNoStack[depth];
    workNodeWidth = This->fVoxelSliceWidthStack[depth];
    workMinExtent = G4VoxelHeader_GetMinExtent(workHeader);
    workCoord = G4ThreeVector_coord(targetPoint,workHeaderAxis);
    minVal = workMinExtent+G4VoxelNode_GetMinEquivalentSliceNo(This->fVoxelNode)*workNodeWidth;
    if ( minVal<=workCoord+1E-3*0.5 )
    {
      maxVal = workMinExtent+(G4VoxelNode_GetMaxEquivalentSliceNo(This->fVoxelNode)+1)
                            *workNodeWidth;
      if ( maxVal<=workCoord-1E-3*0.5 )
      {
        newNodeNo = G4VoxelNode_GetMaxEquivalentSliceNo(This->fVoxelNode)+1;
        newHeader = workHeader;
        newDistance = (maxVal-G4ThreeVector_coord(localPoint,workHeaderAxis))
                    / G4ThreeVector_coord(localDirection,workHeaderAxis);
        isNewVoxel = true;
        newDepth = depth;
      }
    }
    else
    {
      newNodeNo = G4VoxelNode_GetMinEquivalentSliceNo(This->fVoxelNode)-1;
      newHeader = workHeader;
      newDistance = (minVal-G4ThreeVector_coord(localPoint,workHeaderAxis))
                  / G4ThreeVector_coord(localDirection,workHeaderAxis);
      isNewVoxel = true;
      newDepth = depth;
    }
    currentDistance = newDistance;
  }
  if (isNewVoxel)
  {
    if ( (newNodeNo<0) || (newNodeNo>=G4VoxelHeader_GetNoSlices(newHeader)))
    {
      isNewVoxel = false;
    }
    else
    {
      voxelPoint = G4ThreeVector_saxpy(newDistance,localDirection,localPoint);
      ((newDepth < 4) ? static_cast<void> (0) : __assert_fail ("newDepth < 4", "G4VoxelNavigation.c", 237, __PRETTY_FUNCTION__));
      This->fVoxelNodeNoStack[newDepth] = newNodeNo;
      This->fVoxelDepth = newDepth;
      newVoxelNode = 0;
      while ( newVoxelNode == (( void*)0) )
      {
        newProxy = G4VoxelHeader_GetSlice(newHeader,newNodeNo);
        if ( G4VoxelProxy_IsNode(newProxy) )
        {
          newVoxelNode = G4VoxelProxy_GetNode(newProxy);
        }
        else
        {
          This->fVoxelDepth++;
          ((This->fVoxelDepth < 4) ? static_cast<void> (0) : __assert_fail ("This->fVoxelDepth < 4", "G4VoxelNavigation.c", 251, __PRETTY_FUNCTION__));
          newHeader = G4VoxelProxy_GetHeader(newProxy);
          newHeaderAxis = G4VoxelHeader_GetAxis(newHeader);
          newHeaderNoSlices = G4VoxelHeader_GetNoSlices(newHeader);
          newHeaderMin = G4VoxelHeader_GetMinExtent(newHeader);
          newHeaderNodeWidth =
   (G4VoxelHeader_GetMaxExtent(newHeader)-newHeaderMin)
                             / newHeaderNoSlices;
          newNodeNo = (G4int)(
   (G4ThreeVector_coord(voxelPoint,newHeaderAxis)-newHeaderMin)
                             / newHeaderNodeWidth );
          if ( newNodeNo<0 )
          {
            newNodeNo=0;
          }
          else if ( newNodeNo>=newHeaderNoSlices )
               {
                 newNodeNo = newHeaderNoSlices-1;
               }
          This->fVoxelAxisStack[This->fVoxelDepth] = newHeaderAxis;
          This->fVoxelNoSlicesStack[This->fVoxelDepth] = newHeaderNoSlices;
          This->fVoxelSliceWidthStack[This->fVoxelDepth] = newHeaderNodeWidth;
          This->fVoxelNodeNoStack[This->fVoxelDepth] = newNodeNo;
          This->fVoxelHeaderStack[This->fVoxelDepth] = newHeader;
        }
      }
      This->fVoxelNode = newVoxelNode;
    }
  }
  return isNewVoxel;
}

G4double
G4VoxelNavigation_ComputeVoxelSafety(
   const G4VoxelNavigation *This,
   G4ThreeVector localPoint)
{
  G4SmartVoxelHeader *curHeader;
  G4double voxelSafety, curNodeWidth;
  G4double curNodeOffset, minCurCommonDelta, maxCurCommonDelta;
  G4int minCurNodeNoDelta, maxCurNodeNoDelta;
  G4int localVoxelDepth, curNodeNo;
  EAxis curHeaderAxis;
  localVoxelDepth = This->fVoxelDepth;
  curHeader = This->fVoxelHeaderStack[localVoxelDepth];
  curHeaderAxis = This->fVoxelAxisStack[localVoxelDepth];
  curNodeNo = This->fVoxelNodeNoStack[localVoxelDepth];
  curNodeWidth = This->fVoxelSliceWidthStack[localVoxelDepth];
  curNodeOffset = curNodeNo*curNodeWidth;
  maxCurNodeNoDelta = G4VoxelNode_GetMaxEquivalentSliceNo(This->fVoxelNode)-curNodeNo;
  minCurNodeNoDelta = curNodeNo-G4VoxelNode_GetMaxEquivalentSliceNo(This->fVoxelNode);
  minCurCommonDelta = G4ThreeVector_coord(localPoint,curHeaderAxis)
 - G4VoxelHeader_GetMinExtent(curHeader) - curNodeOffset;
  maxCurCommonDelta = curNodeWidth-minCurCommonDelta;
  if ( minCurNodeNoDelta<maxCurNodeNoDelta )
  {
    voxelSafety = minCurNodeNoDelta*curNodeWidth;
    voxelSafety += minCurCommonDelta;
  }
  else if (maxCurNodeNoDelta < minCurNodeNoDelta)
       {
         voxelSafety = maxCurNodeNoDelta*curNodeWidth;
         voxelSafety += maxCurCommonDelta;
        }
        else
        {
          voxelSafety = minCurNodeNoDelta*curNodeWidth;
          voxelSafety += (((minCurCommonDelta)<(maxCurCommonDelta))?(minCurCommonDelta):(maxCurCommonDelta));
        }
  while ( (localVoxelDepth>0) && (voxelSafety>0) )
  {
    localVoxelDepth--;
    curHeader = This->fVoxelHeaderStack[localVoxelDepth];
    curHeaderAxis = This->fVoxelAxisStack[localVoxelDepth];
    curNodeNo = This->fVoxelNodeNoStack[localVoxelDepth];
    curNodeWidth = This->fVoxelSliceWidthStack[localVoxelDepth];
    curNodeOffset = curNodeNo*curNodeWidth;
    minCurCommonDelta = G4ThreeVector_coord(localPoint,curHeaderAxis)
                        - G4VoxelHeader_GetMinExtent(curHeader) - curNodeOffset;
    maxCurCommonDelta = curNodeWidth-minCurCommonDelta;
    if ( minCurCommonDelta<voxelSafety )
    {
      voxelSafety = minCurCommonDelta;
    }
    if ( maxCurCommonDelta<voxelSafety )
    {
      voxelSafety = maxCurCommonDelta;
    }
  }
  if ( voxelSafety<0 )
  {
    voxelSafety = 0;
  }
  return voxelSafety;
}

void G4VoxelNavigation_ctor( G4VoxelNavigation *This )
{
 This->fVoxelDepth = -1;
 This->fVoxelNode = (( void*)0);
}
inline
G4bool
G4VoxelNavigation_LevelLocate(
   G4VoxelNavigation *This,
   G4NavigationHistory* history,
   const G4VPhysicalVolume* blockedVol,
   G4ThreeVector globalPoint,
   const G4ThreeVector* globalDirection,
   const G4bool pLocatedOnEdge,
   G4ThreeVector *localPoint )
{
  G4SmartVoxelHeader *targetVoxelHeader;
  G4SmartVoxelNode *targetVoxelNode;
  G4VPhysicalVolume *targetPhysical, *samplePhysical;
  G4LogicalVolume *targetLogical;
  G4VSolid *sampleSolid;
  G4ThreeVector samplePoint;
  G4int targetNoDaughters;
  targetPhysical = G4NavigationHistory_GetTopVolume(history);
  targetLogical = G4VPhysicalVolume_GetLogicalVolume(targetPhysical);
  targetVoxelHeader = G4LogicalVolume_GetVoxelHeader(targetLogical);
  targetVoxelNode =
 G4VoxelNavigation_VoxelLocate(This,targetVoxelHeader,*localPoint);
  targetNoDaughters=G4VoxelNode_GetNoContained(targetVoxelNode);
  if ( targetNoDaughters==0 ) return false;
  for ( int sampleNo=targetNoDaughters-1; sampleNo>=0; sampleNo-- )
  {
    samplePhysical =
  G4LogicalVolume_GetDaughter( targetLogical,
   G4VoxelNode_GetVolume(targetVoxelNode,sampleNo));
    if ( samplePhysical!=blockedVol )
    {
      G4NavigationHistory_NewLevel(history, samplePhysical, kNormal);
      sampleSolid =
  G4LogicalVolume_GetSolid(
   G4VPhysicalVolume_GetLogicalVolume( samplePhysical ));
   G4AffineTransform tf = G4NavigationHistory_GetTopTransform( history );
      samplePoint =
  G4AffineTransform_TransformPoint( &tf, globalPoint );
      if( G4AuxiliaryNavServices_CheckPointOnSurface(
   sampleSolid, samplePoint, globalDirection,
   tf, pLocatedOnEdge) )
      {
        *localPoint = samplePoint;
        return true;
      }
      else
      {
    G4NavigationHistory_BackLevel( history );
      }
    }
  }
  return false;
}

G4double
G4VoxelNavigation_ComputeStep(
   G4VoxelNavigation *This,
   G4ThreeVector localPoint,
   G4ThreeVector localDirection,
   const G4double currentProposedStepLength,
   G4double *newSafety,
   G4NavigationHistory *history,
   G4bool *validExitNormal,
   G4ThreeVector *exitNormal,
   G4bool *exiting,
   G4bool *entering,
   G4VPhysicalVolume *(*pBlockedPhysical)
    , G4double * Result
   )
{
  G4VPhysicalVolume *motherPhysical, *samplePhysical,
 *blockedExitedVol = (( void*)0);
  G4LogicalVolume *motherLogical;
  G4VSolid *motherSolid;
  G4ThreeVector sampleDirection;
  G4double ourStep=currentProposedStepLength, motherSafety, ourSafety;
  G4int sampleNo;
  G4bool initialNode, noStep;
  const G4SmartVoxelNode *curVoxelNode;
  G4int curNoVolumes, contentNo;
  G4double voxelSafety;
  motherPhysical = G4NavigationHistory_GetTopVolume( history );
  motherLogical = G4VPhysicalVolume_GetLogicalVolume(motherPhysical);
  motherSolid = G4LogicalVolume_GetSolid(motherLogical);
  motherSafety = G4VSolid_DistanceToOut(motherSolid, localPoint);
  ourSafety = motherSafety;
  if ( *exiting && *validExitNormal )
  {
    if ( G4ThreeVector_dot(localDirection,*exitNormal)>=kMinExitingNormalCosine )
    {
      blockedExitedVol = *pBlockedPhysical;
      ourSafety = 0;
    }
  }
  *exiting = false;
  *entering = false;
  initialNode = true;
  noStep = true;
  while ( noStep )
  {
    curVoxelNode = This->fVoxelNode;
    curNoVolumes = G4VoxelNode_GetNoContained(curVoxelNode);
    for (contentNo=curNoVolumes-1; contentNo>=0; contentNo--)
    {
      sampleNo = G4VoxelNode_GetVolume( curVoxelNode, contentNo);
        samplePhysical = G4LogicalVolume_GetDaughter(motherLogical,sampleNo);
        if ( samplePhysical!=blockedExitedVol )
        {
    G4AffineTransform sampleTf =
   G4AffineTransform_create_full(
    G4VPhysicalVolume_GetObjectRotationValue(samplePhysical),
    G4VPhysicalVolume_GetTranslation(samplePhysical));
          G4AffineTransform_Invert(&sampleTf);
          const G4ThreeVector samplePoint =
    G4AffineTransform_TransformPoint(&sampleTf,localPoint);
          const G4VSolid *sampleSolid =
    G4LogicalVolume_GetSolid(
     G4VPhysicalVolume_GetLogicalVolume(
      samplePhysical ));
          const G4double sampleSafety =
   G4VSolid_DistanceToIn(sampleSolid,samplePoint);
          if ( sampleSafety<ourSafety )
          {
            ourSafety = sampleSafety;
          }
          if ( sampleSafety<=ourStep )
          {
            sampleDirection =
    G4AffineTransform_TransformAxis( &sampleTf, localDirection );
            G4double sampleStep =
    G4VSolid_DistanceToIn_full(sampleSolid, samplePoint, sampleDirection);
            if ( sampleStep<=ourStep )
            {
              ourStep = sampleStep;
              *entering = true;
              *exiting = false;
              *pBlockedPhysical = samplePhysical;
            }
        }
      }
    }
    if (initialNode)
    {
      initialNode = false;
      voxelSafety = G4VoxelNavigation_ComputeVoxelSafety(This,localPoint);
      if ( voxelSafety<ourSafety )
      {
        ourSafety = voxelSafety;
      }
      if ( currentProposedStepLength<ourSafety )
      {
        noStep = false;
        *entering = false;
        *exiting = false;
        *pBlockedPhysical = (( void*)0);
        ourStep = kInfinity;
      }
      else
      {
        if ( motherSafety<=ourStep )
        {
          G4double motherStep =
   G4VSolid_DistanceToOut_full( motherSolid, localPoint, localDirection,
                                         true, validExitNormal, exitNormal);
          if ( motherStep<=ourStep )
          {
            ourStep = motherStep;
            *exiting = true;
            *entering = false;
            if ( *validExitNormal )
            {
    G4RotationMatrix rot = G4VPhysicalVolume_GetObjectRotationValue(motherPhysical);
    G4RotationMatrix inv = G4RotationMatrix_inverse(&rot);
    *exitNormal = G4RotationMatrix_apply( &inv, *exitNormal );
            }
          }
          else
          {
            *validExitNormal = false;
          }
        }
      }
      *newSafety = ourSafety;
    }
    if (noStep)
    {
      noStep = G4VoxelNavigation_LocateNextVoxel(This, localPoint, localDirection, ourStep);
    }
  }
  int locationId = get_global_id(0);
  return ourStep;
}
 void trace(
 Particle *input,
 G4double *output,
 G4VPhysicalVolume *worldVolumeAndGeomBuffer,
 G4double phys_step,
 int totalSize
 , G4double * Result
 , SolidInfo * Solids
 , ResultInfo * Result_For_Current_Solid
 , FinalResult * Compacter_Result,
 G4SmartVoxelNode * nullVNode
 )
{
 const unsigned globalIdx = get_global_id(0);
 const unsigned localIdx = get_local_id(0);
 const unsigned locationId = globalIdx;
 if (globalIdx >= totalSize ) return;
 SHAREDMEM int Numbers_Of_Solid[ BlockSize ];
 SHAREDMEM int Sum_Of_Solids[ BlockSize ];
 SHAREDMEM bool noStepArray [ BlockSize ];
 SHAREDMEM PointInformation LocationArray[ BlockSize ];
 SHAREDMEM G4VPhysicalVolume * info[ BlockSize ];
 G4VoxelNode_ctor( nullVNode ,1 );
 SHAREDMEM bool Cur_Vol_Store [ BlockSize ];
 G4Navigator navi;
 G4Navigator *nav = &navi;
 G4Navigator_ctor(nav);
 G4Navigator_SetWorldVolume( nav, worldVolumeAndGeomBuffer );
 Particle p = input[globalIdx];
 if( globalIdx == 0)
 {
 }
    const G4VPhysicalVolume * cur_vol =
  G4Navigator_LocateGlobalPointAndSetup(
   nav, p.pos, __null, false, true, Result );
 G4bool cur_vol_local = true, cur_vol_all = true;
 G4double step, safety = 0.1;
 G4double integratedDensity = 0;
 int temp = 0;
 while ( cur_vol_all )
 {
  {
  const G4double curDensity =
    G4LogicalVolume_GetMaterial( G4VPhysicalVolume_GetLogicalVolume( cur_vol ))->property;
  PointInformation NewPoint = { p.pos, p.dir };
  LocationArray[ locationId ] = NewPoint;
  if( temp == 1)
  {
   Result[ locationId ] = step;
  }
  step = G4Navigator_ComputeStep( nav, p.pos, p.dir, phys_step, &safety
       , cur_vol_local
       , Result
       );
  if ( step == kInfinity ) step = phys_step;
  const G4double nextStepIntegratedD = curDensity * step;
  int locationId = get_global_id(0);
  integratedDensity += nextStepIntegratedD;
  G4ThreeVector_sum_assign( &(p.pos), G4ThreeVector_mult( p.dir, step ) );
  G4Navigator_SetGeometricallyLimitedStep( nav );
  if( globalIdx == 0 ){
  }
  cur_vol =
   G4Navigator_LocateGlobalPointAndSetup(
    nav, p.pos, &(p.dir), true, false, Result );
  if ( !cur_vol )
   cur_vol_local = false;
  }
  Cur_Vol_Store[ locationId ] = cur_vol_local;
  BARRIER_FLEXIBLE;
  cur_vol_all = NoStepReduction( Cur_Vol_Store, BlockSize );
  BARRIER_FLEXIBLE;
  temp++;
}
 output[globalIdx] = integratedDensity;
}
 void relocate ( int * ptr, void * buf, int size )
{
  typedef unsigned char byte;
  const unsigned globalidx = get_global_id(0);
  if(globalidx>=size) return;
  int destoffs, targoffs;
  destoffs = *(ptr + 2*globalidx);
  targoffs = *(ptr + 2*globalidx+ 1);
  *((byte*)buf+destoffs) =
    (byte*)buf + targoffs;
}
 void check( G4VPhysicalVolume *worldVolumeAndGeomBuffer, unsigned long * result)
{
 unsigned int hope = ( unsigned int )worldVolumeAndGeomBuffer;
 *result = hope;
}
__kernel void test ( bool * output
 , bool * input
 )
{
 int tid = get_global_id(0);
 int offset = 1;
 G4bool result;
 if( tid == 0)
 {
 input[ 0] = true;
 input[ 1] = true;
 input[ 2] = true;
 input[ 3] = true;
 input[ 4] = true;
 input[ 5] = true;
    input[ 6] = false;
    input[ 7] = true;
 }
 BARRIER_ALL;
}
 void checkgeom( G4VPhysicalVolume *worldVolumeAndGeomBuffer, int * result, int number_of_increments)
{
 const unsigned globalid = get_global_id(0);
 if(globalid>=1) return;
 __private int i=0;
 G4Navigator navi;
 G4Navigator *nav = &navi;
 G4Navigator_ctor(nav);
 G4Navigator_SetWorldVolume( nav, worldVolumeAndGeomBuffer );
 G4ThreeVector pos = G4ThreeVector_create( 0.0, 0.0, 0.0);
 const G4VPhysicalVolume * cur_vol;
 unsigned int geom_start = ( unsigned int )worldVolumeAndGeomBuffer;
 pos = G4ThreeVector_create( 0.7, 1.0, 0.7);
 float x_increment = 0.2, y_increment = 0.2, z_increment = 0.2;
 for( i=0; i < number_of_increments*3 ; i+=3)
 {
  result[i] = ( int ) cur_vol->count;
  result[i + 1] = (( unsigned int )(cur_vol->flogical) - geom_start);
  result[i + 2] = ( int ) G4LogicalVolume_GetMaterial( G4VPhysicalVolume_GetLogicalVolume( cur_vol ))->property;
  pos.x+=x_increment;
  pos.y+=y_increment;
  pos.z+=z_increment;
 }
}
struct CameraParameters
{
 double
  heading,
  pitch,
  roll,
  dist,
  yfov,
  target_x,
  target_y,
  target_z;
 CameraParameters()
 :
  heading(0), pitch(0), roll(0), dist(1),
  yfov(90), target_x(0), target_y(0), target_z(0)
 {}
};
struct EventOrigin
{
 double x,y,z;
};
class Geometry
{
public:
 typedef unsigned char byte;
 virtual ~Geometry() {}
 virtual void create() = 0;
 virtual void relocate( void *newbegin ) = 0;
 virtual int size() const = 0;
 virtual int ptrs_size() const=0;
 virtual void *getBuffer() = 0;
 virtual double getScale() const = 0;
 virtual CameraParameters getCamera() const = 0;
 virtual EventOrigin getEvent() const
 {
  EventOrigin e = { 0,0,0 };
  return e;
 }
 virtual int getNumVoxelNodes() const { return 0; }
};
typedef struct { const char *err, *fn; int line, errcode; } my_cuda_err;
typedef struct { int secs; int usecs; } mytimet;
extern "C"
{
 void myprint( const char *chr );
 void myprint1( const char *chr, int n );
 mytimet mytimer();
 void myprinttdiff(mytimet a, mytimet b);
 void mysleep(int n);
}
static inline int ceilDiv( int a, int d )
{
 return a/d + ((a%d)?1:0);
}
Particle *gpuInput;
G4double *gpuOutput;
Geometry::byte *gpuGeom;
int numInput, numOutput, numInputPerRound;
const int WARP_SIZE = 32;
void createGrid( int numInput, dim3* grid, dim3* block )
{
 const int MAXSIZE = 10000000;
 const int NUMCORES = 448;
 const int NUMMULTIPROC = 14;
 const int BLOCKS_PER_MULTIPROC = 8;
 const int MAX_WARPS_PER_MULTIPROC = 48;
 const int MAX_DATA_PER_MULTIPROC = MAX_WARPS_PER_MULTIPROC*WARP_SIZE;
 int size = numInput;
 if (size > MAXSIZE) size = MAXSIZE;
 int dataPerMultiproc = ceilDiv(size,NUMMULTIPROC);
 if ( dataPerMultiproc > MAX_DATA_PER_MULTIPROC )
  dataPerMultiproc = MAX_DATA_PER_MULTIPROC;
 int blockSize = ceilDiv(dataPerMultiproc,BLOCKS_PER_MULTIPROC);
 const int MAX_BLOCK_SIZE = 1024;
 if (blockSize > MAX_BLOCK_SIZE) blockSize = MAX_BLOCK_SIZE;
 int numBlocks = ceilDiv(size,blockSize);
 int numWarps = ceilDiv(blockSize,WARP_SIZE) * numBlocks;
 if (numWarps > NUMCORES)
 {
  blockSize = ceilDiv(blockSize,WARP_SIZE)*WARP_SIZE;
  dataPerMultiproc = blockSize * BLOCKS_PER_MULTIPROC;
  if ( dataPerMultiproc > MAX_DATA_PER_MULTIPROC )
   blockSize -= WARP_SIZE;
 }
 size = blockSize*ceilDiv(size,blockSize);
 if (size > MAXSIZE) size = MAXSIZE;
 block->x = blockSize;
 block->y = block->z = 1;
 grid->x = size/blockSize;
 grid->y = 1;
 grid->z = 1;
}
my_cuda_err cudainit( Geometry *geom, int N )
{
 const mytimet t0 = mytimer();
 numOutput = numInput = numInputPerRound = N;
 do { hipError_t errc = hipSetDeviceFlags(0); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 157, errc }; return r; } } while(0);
 do { hipError_t errc = hipMalloc( (void**)&gpuInput, sizeof(Particle)*numInput ); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 159, errc }; return r; } } while(0);
 do { hipError_t errc = hipMalloc( (void**)&gpuOutput, sizeof(G4double)*numOutput ); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 160, errc }; return r; } } while(0);
 do { hipError_t errc = hipMalloc( (void**)&gpuGeom, geom->size() ); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 161, errc }; return r; } } while(0);
 geom->relocate( gpuGeom );
 hipFuncSetCacheConfig(reinterpret_cast<const void*>(trace), hipFuncCachePreferL1);
 do { hipError_t errc = hipMemcpy( gpuGeom, geom->getBuffer(), geom->size(), hipMemcpyHostToDevice ); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 197, errc }; return r; } } while(0);
 const mytimet t1 = mytimer();
 myprint("Initialization: ");
 myprinttdiff(t0, t1);
 my_cuda_err ok = { __null, __null, 0, hipSuccess }; return ok;
}
my_cuda_err cudaexec( G4double phys_step, int totalInput, Particle *input, G4double *output )
{
   for ( int i = 0; i < totalInput; i += numInput )
   {
 if ( i + numInput > totalInput ) numInput = totalInput-i;
 do { hipError_t errc = hipMemcpy( gpuInput, input+i, sizeof(Particle)*numInput, hipMemcpyHostToDevice ); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 333, errc }; return r; } } while(0);
 dim3 grid, block;
 createGrid( numInput, &grid, &block );
 trace <<< grid, block >>>( gpuInput, gpuOutput, (G4VPhysicalVolume*)gpuGeom, phys_step, numInput );
 do { hipError_t errc = hipGetLastError(); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 340, errc }; return r; } } while(0);
 do { hipError_t errc = hipMemcpy( output+i, gpuOutput, sizeof(G4double)*numOutput, hipMemcpyDeviceToHost ); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 343, errc }; return r; } } while(0);
   }
   my_cuda_err ok = { __null, __null, 0, hipSuccess }; return ok;
}
my_cuda_err cudafinish()
{
 const mytimet t0 = mytimer();
 do { hipError_t errc = hipFree( gpuInput ); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 356, errc }; return r; } } while(0);
 do { hipError_t errc = hipFree( gpuOutput ); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 357, errc }; return r; } } while(0);
 do { hipError_t errc = hipFree( gpuGeom ); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 358, errc }; return r; } } while(0);
 do { hipError_t errc = hipDeviceReset(); if (errc != hipSuccess) { my_cuda_err r = { hipGetErrorString(errc), "cuda.cpp", 375, errc }; return r; } } while(0);
 const mytimet t1 = mytimer();
 myprint("Finalization: ");
 myprinttdiff(t0, t1);
 my_cuda_err ok = { __null, __null, 0, hipSuccess }; return ok;
}
